#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <math.h>
#include <stddef.h>
#include <stdlib.h>
#include <stdio.h>
#include "caml/alloc.h"
#include "caml/bigarray.h"
#include "caml/memory.h"
#include "caml/mlvalues.h"
#include "gpu-utils.cu"
#include "hipblas.h"
enum tensor_state {STATE_OK, STATE_CPU_INVALID, STATE_GPU_INVALID, STATE_RETURNED};
__managed__ enum tensor_state (*t_state);
typedef struct Tensor {int64_t id; float (*data); int64_t dims[3]; int64_t rank; int64_t offset; int64_t size;} Tensor;
typedef struct Seq {int64_t (*seq); int64_t len;} Seq;
typedef struct Tensor1 {int64_t id; int64_t (*data); int64_t dims[3]; int64_t rank; int64_t offset; int64_t size;} Tensor1;
typedef struct Rec {Tensor b; Tensor w; int64_t ty; Tensor b_grads; Tensor w_grads; Tensor in_grads; Tensor out_bufs; Tensor softmax_bufs;} Rec;
typedef struct Rec1 {int64_t ty; Tensor in_grads; Tensor out_bufs; Tensor softmax_bufs;} Rec1;
typedef struct Seq1 {Rec (*seq); int64_t len;} Seq1;
typedef struct Rec2 {Rec1 lossfn; Seq1 components;} Rec2;
typedef struct Rec3 {Tensor inputs; Tensor1 correct_linear_outidxs;} Rec3;
typedef struct Seq2 {Rec3 (*seq); int64_t len;} Seq2;
typedef struct Rec4 {int64_t epochs; int64_t batchsize; float init_alpha; float decay_alpha; float init_lambda; char printStatus; float decay_lambda; char evaluateBetweenEpochs; char evaluateBeforeFirstEpoch;} Rec4;
typedef struct Rec5 {float _0; float _1;} Rec5;

static hipblasHandle_t _cublas_handle;

__host__ __device__ int64_t cartesian_to_linear_index0(int64_t dims1[3], int64_t rank1) {
  {
    int64_t t;
    (t = 0);
    return t;
  }
}
__host__ __device__ int64_t cartesian_to_linear_index1(int64_t dims1[3], int64_t rank1, int64_t i) {
  if ((rank1 == 3)) {
    {
      int64_t t1;
      (t1 = (((dims1[2]) * (dims1[1])) * i));
      return t1;
    }
  } else {
    if ((rank1 == 2)) {
      {
        int64_t t2;
        (t2 = ((dims1[1]) * i));
        return t2;
      }
    } else {
      {
        int64_t t3;
        (t3 = i);
        return t3;
      }
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index2(int64_t dims1[3], int64_t rank1, int64_t i1, int64_t i2) {
  if ((rank1 == 3)) {
    {
      int64_t t4;
      (t4 = ((((dims1[2]) * (dims1[1])) * i1) + ((dims1[2]) * i2)));
      return t4;
    }
  } else {
    if ((rank1 == 2)) {
      {
        int64_t t5;
        (t5 = (((dims1[1]) * i1) + i2));
        return t5;
      }
    } else {
      printf("Accessed tensor of rank %ld using 2 indices\n", rank1);
      {
        int64_t t6;
        (t6 = -1);
        return t6;
      }
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index3(int64_t dims1[3], int64_t rank1, int64_t i3, int64_t i4, int64_t i5) {
  if ((rank1 == 3)) {
    {
      int64_t t7;
      (t7 = (((((dims1[2]) * (dims1[1])) * i3) + ((dims1[2]) * i4)) + i5));
      return t7;
    }
  } else {
    printf("Accessed tensor of rank %ld using 3 indices\n", rank1);
    {
      int64_t t8;
      (t8 = -1);
      return t8;
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index(int64_t dims1[3], int64_t rank1, Seq cartesian_idx) {
  if (((cartesian_idx.len) == 1)) {
    {
      int64_t t9;
      (t9 = cartesian_to_linear_index1(dims1, rank1, ((cartesian_idx.seq)[0])));
      return t9;
    }
  } else {
    if (((cartesian_idx.len) == 2)) {
      {
        int64_t t10;
        (t10 = cartesian_to_linear_index2(dims1, rank1, ((cartesian_idx.seq)[0]), ((cartesian_idx.seq)[1])));
        return t10;
      }
    } else {
      if (((cartesian_idx.len) == 3)) {
        {
          int64_t t11;
          (t11 = cartesian_to_linear_index3(dims1, rank1, ((cartesian_idx.seq)[0]), ((cartesian_idx.seq)[1]), ((cartesian_idx.seq)[2])));
          return t11;
        }
      } else {
        {
          int64_t t12;
          (t12 = cartesian_to_linear_index0(dims1, rank1));
          return t12;
        }
      }
    }
  }
}
__host__ __device__ Seq tensor_shape(int64_t dims2[3], int64_t rank2) {
  Seq s;
  ((s.seq) = dims2);
  ((s.len) = rank2);
  {
    Seq t13;
    (t13 = s);
    return t13;
  }
}
extern double exp(double);
__host__ __device__ float exp1(float x) {
  {
    float t14;
    (t14 = exp(x));
    return t14;
  }
}
__host__ __device__ int64_t t15(int64_t acc, int64_t e) {
  {
    int64_t t16;
    (t16 = (acc * e));
    return t16;
  }
}
__host__ __device__ int64_t foldl(int64_t acc_init, Seq s1) {
  int64_t acc1 = acc_init;
  int64_t i6 = 0;
  while ((i6 < (s1.len))) {
    (acc1 = t15(acc1, ((s1.seq)[i6])));
    (i6 = (i6 + 1));
  }
  return acc1;
}
__host__ __device__ int64_t tensorSize(Tensor t17) {
  Seq t18;
  (t18 = tensor_shape((t17.dims), (t17.rank)));
  {
    int64_t t19;
    (t19 = foldl(1, t18));
    return t19;
  }
}
__host__ __device__ float t20(Tensor w1, Tensor x1, int64_t m, int64_t x_offset, int64_t i7, float acc2, int64_t j) {
  int64_t t21;
  (t21 = (m * j));
  int64_t t22;
  (t22 = (i7 + t21));
  float t23;
  (t23 = ((w1.data)[(t22 + (w1.offset))]));
  int64_t t24;
  (t24 = (x_offset + j));
  float t25;
  (t25 = ((x1.data)[(t24 + (x1.offset))]));
  float t26;
  (t26 = (t23 * t25));
  {
    float t27;
    (t27 = (acc2 + t26));
    return t27;
  }
}
__host__ __device__ void iterfun(Tensor w1, Tensor x1, Tensor b1, Tensor z, int64_t m, int64_t n, int64_t i8) {
  int64_t s_idx;
  (s_idx = (i8 / m));
  int64_t x_offset;
  (x_offset = (s_idx * n));
  int64_t z_idx;
  (z_idx = i8);
  int64_t i7;
  (i7 = (i8 % m));
  float acc_init1;
  (acc_init1 = ((b1.data)[(i7 + (b1.offset))]));
  float v;
  {
    int64_t i9 = 0;
    float acc3 = acc_init1;
    while ((i9 < n)) {
      (acc3 = t20(w1, x1, m, x_offset, i7, acc3, i9));
      (i9 = (i9 + 1));
    }
    (v = acc3);
  }
  (((z.data)[(z_idx + (z.offset))]) = v);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel(int64_t n1, Tensor w1, Tensor x1, Tensor b1, Tensor z, int64_t m, int64_t n) {
  int64_t idx = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride = (gridDim.x * blockDim.x);
  while ((idx < n1)) {
    iterfun(w1, x1, b1, z, m, n, idx);
    (idx = (idx + stride));
  }
}
__host__ void tensorOpExn__z___Wx_B(int64_t s_max, Tensor w1, Tensor x1, Tensor b1, Tensor z) {
  Seq w_shape;
  (w_shape = tensor_shape((w1.dims), (w1.rank)));
  int64_t m;
  (m = ((w_shape.seq)[1]));
  int64_t n;
  (n = ((w_shape.seq)[0]));
  int64_t m_x_n = m * n;

  float alpha = 1.0;
  float beta = 1.0;
  for (int64_t s = 0; s < s_max; ++s) {
    float *w_data = w1.data + (s * m_x_n);
    float *b_data = b1.data + (s * m);
    hipblasSgemv(
      _cublas_handle,
      HIPBLAS_OP_N,
      (int) m, (int) n,
      &alpha,
      w1.data, (int) m, /* lda */
      x1.data, 0, /* incx */
      &beta,
      b1.data, 0 /* incy */
    );
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
  hipDeviceSynchronize();
  GPU_UTILS_CHECK_CUDA_ERROR();
  /* OLD CODE:
  int64_t t28;
  (t28 = (s_max * m));
  {
    int64_t niterations = t28;
    int64_t tpb = 256;
    int64_t nblocks = (((niterations + (tpb * 10)) - 1) / (tpb * 10));
    loopKernel<<<nblocks, tpb>>>(t28, w1, x1, b1, z, m, n);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
  */
}
__host__ __device__ int64_t t29(Tensor x2, Tensor z1, int64_t x_offset1, float y_val, int64_t z_idx1, int64_t row) {
  int64_t t30;
  (t30 = (x_offset1 + row));
  float t31;
  (t31 = ((x2.data)[(t30 + (x2.offset))]));
  float t32;
  (t32 = (y_val * t31));
  (((z1.data)[(z_idx1 + (z1.offset))]) = t32);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z1.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z1.id)]) = STATE_GPU_INVALID);
  #endif
  {
    int64_t t33;
    (t33 = (z_idx1 + 1));
    return t33;
  }
}
__host__ __device__ void iterfun1(Tensor x2, Tensor y, Tensor z1, int64_t m1, int64_t n2, int64_t m_x_n, int64_t i10) {
  int64_t s_idx1;
  (s_idx1 = (i10 / n2));
  int64_t col;
  (col = (i10 % n2));
  int64_t t34;
  (t34 = (s_idx1 * m_x_n));
  int64_t t35;
  (t35 = (m1 * col));
  int64_t z_offset;
  (z_offset = (t34 + t35));
  int64_t x_offset1;
  (x_offset1 = (s_idx1 * m1));
  int64_t y_offset;
  (y_offset = (s_idx1 * n2));
  int64_t t36;
  (t36 = (y_offset + col));
  float y_val;
  (y_val = ((y.data)[(t36 + (y.offset))]));
  int64_t _;
  {
    int64_t i11 = 0;
    int64_t acc4 = z_offset;
    while ((i11 < m1)) {
      (acc4 = t29(x2, z1, x_offset1, y_val, acc4, i11));
      (i11 = (i11 + 1));
    }
    (_ = acc4);
  }
  ;
}
__global__ void loopKernel1(int64_t n3, Tensor x2, Tensor y, Tensor z1, int64_t m1, int64_t n2, int64_t m_x_n) {
  int64_t idx1 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride1 = (gridDim.x * blockDim.x);
  while ((idx1 < n3)) {
    iterfun1(x2, y, z1, m1, n2, m_x_n, idx1);
    (idx1 = (idx1 + stride1));
  }
}
__host__ void tensorOpExn__z___x___y_T(int64_t s_max1, Tensor x2, Tensor y, Tensor z1) {
  Seq z_shape;
  (z_shape = tensor_shape((z1.dims), (z1.rank)));
  int64_t m1;
  (m1 = ((z_shape.seq)[2]));
  int64_t n2;
  (n2 = ((z_shape.seq)[1]));
  int64_t m_x_n;
  (m_x_n = (m1 * n2));
  int64_t t37;
  (t37 = (s_max1 * n2));
  {
    int64_t niterations1 = t37;
    int64_t tpb1 = 256;
    int64_t nblocks1 = (((niterations1 + (tpb1 * 10)) - 1) / (tpb1 * 10));
    loopKernel1<<<nblocks1, tpb1>>>(t37, x2, y, z1, m1, n2, m_x_n);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ float t38(Tensor x3, Tensor w2, int64_t m2, int64_t x_offset2, int64_t j1, float acc5, int64_t i12) {
  int64_t t39;
  (t39 = (m2 * j1));
  int64_t t40;
  (t40 = (i12 + t39));
  float t41;
  (t41 = ((w2.data)[(t40 + (w2.offset))]));
  int64_t t42;
  (t42 = (x_offset2 + i12));
  float t43;
  (t43 = ((x3.data)[(t42 + (x3.offset))]));
  float t44;
  (t44 = (t41 * t43));
  {
    float t45;
    (t45 = (acc5 + t44));
    return t45;
  }
}
__host__ __device__ void iterfun2(Tensor x3, Tensor w2, Tensor z2, int64_t m2, int64_t n4, int64_t j2) {
  int64_t s_idx2;
  (s_idx2 = (j2 / n4));
  int64_t n_idx;
  (n_idx = (j2 % n4));
  int64_t z_idx2;
  (z_idx2 = j2);
  int64_t x_offset2;
  (x_offset2 = (s_idx2 * m2));
  int64_t j1;
  (j1 = n_idx);
  float v1;
  {
    int64_t i13 = 0;
    float acc6 = 0.;
    while ((i13 < m2)) {
      (acc6 = t38(x3, w2, m2, x_offset2, j1, acc6, i13));
      (i13 = (i13 + 1));
    }
    (v1 = acc6);
  }
  (((z2.data)[(z_idx2 + (z2.offset))]) = v1);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z2.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z2.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel2(int64_t n5, Tensor x3, Tensor w2, Tensor z2, int64_t m2, int64_t n4) {
  int64_t idx2 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride2 = (gridDim.x * blockDim.x);
  while ((idx2 < n5)) {
    iterfun2(x3, w2, z2, m2, n4, idx2);
    (idx2 = (idx2 + stride2));
  }
}
__host__ void tensorOpExn__z____x_T___W__T(int64_t s_max2, Tensor x3, Tensor w2, Tensor z2) {
  Seq w_shape1;
  (w_shape1 = tensor_shape((w2.dims), (w2.rank)));
  int64_t m2;
  (m2 = ((w_shape1.seq)[1]));
  int64_t n4;
  (n4 = ((w_shape1.seq)[0]));
  int64_t t46;
  (t46 = (s_max2 * n4));
  {
    int64_t niterations2 = t46;
    int64_t tpb2 = 256;
    int64_t nblocks2 = (((niterations2 + (tpb2 * 10)) - 1) / (tpb2 * 10));
    loopKernel2<<<nblocks2, tpb2>>>(t46, x3, w2, z2, m2, n4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun3(Tensor x4, Tensor z3, int64_t i14) {
  float x_i;
  (x_i = ((x4.data)[(i14 + (x4.offset))]));
  char t47;
  (t47 = (x_i > 0.));
  float t48;
  if ((t47 == 1)) {
    (t48 = x_i);
  } else {
    (t48 = 0.);
  }
  (((z3.data)[(i14 + (z3.offset))]) = t48);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z3.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z3.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel3(int64_t n6, Tensor x4, Tensor z3) {
  int64_t idx3 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride3 = (gridDim.x * blockDim.x);
  while ((idx3 < n6)) {
    iterfun3(x4, z3, idx3);
    (idx3 = (idx3 + stride3));
  }
}
__host__ void tensorOpExn__z___ReLU_x_(int64_t s_max3, Tensor x4, Tensor z3) {
  Seq t49;
  (t49 = tensor_shape((x4.dims), (x4.rank)));
  int64_t s2;
  (s2 = ((t49.seq)[0]));
  int64_t t50;
  (t50 = tensorSize(x4));
  int64_t m3;
  (m3 = (t50 / s2));
  int64_t t51;
  (t51 = (s_max3 * m3));
  {
    int64_t niterations3 = t51;
    int64_t tpb3 = 256;
    int64_t nblocks3 = (((niterations3 + (tpb3 * 10)) - 1) / (tpb3 * 10));
    loopKernel3<<<nblocks3, tpb3>>>(t51, x4, z3);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun4(Tensor x5, Tensor z4, int64_t i15) {
  float x_i1;
  (x_i1 = ((x5.data)[(i15 + (x5.offset))]));
  float t52;
  (t52 = exp1(x_i1));
  (((z4.data)[(i15 + (z4.offset))]) = t52);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z4.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z4.id)]) = STATE_GPU_INVALID);
  #endif
}
__host__ __device__ float t53(Tensor z4, int64_t offset1, float acc7, int64_t i16) {
  int64_t t54;
  (t54 = (offset1 + i16));
  float t55;
  (t55 = ((z4.data)[(t54 + (z4.offset))]));
  {
    float t56;
    (t56 = (acc7 + t55));
    return t56;
  }
}
__host__ __device__ void iterfunSummarize(Tensor expsumbuf, Tensor z4, int64_t m4, int64_t s_idx3) {
  int64_t offset1;
  (offset1 = (s_idx3 * m4));
  float expsum;
  {
    int64_t i17 = 0;
    float acc8 = 0.;
    while ((i17 < m4)) {
      (acc8 = t53(z4, offset1, acc8, i17));
      (i17 = (i17 + 1));
    }
    (expsum = acc8);
  }
  (((expsumbuf.data)[(s_idx3 + (expsumbuf.offset))]) = expsum);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(expsumbuf.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(expsumbuf.id)]) = STATE_GPU_INVALID);
  #endif
}
__host__ __device__ void iterfunRegularize(Tensor expsumbuf, Tensor z4, int64_t m4, int64_t i18) {
  int64_t s_idx4;
  (s_idx4 = (i18 / m4));
  float expsum1;
  (expsum1 = ((expsumbuf.data)[(s_idx4 + (expsumbuf.offset))]));
  float z_i;
  (z_i = ((z4.data)[(i18 + (z4.offset))]));
  float t57;
  (t57 = (z_i / expsum1));
  (((z4.data)[(i18 + (z4.offset))]) = t57);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z4.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z4.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel4(int64_t n7, Tensor expsumbuf, Tensor z4, int64_t m4) {
  int64_t idx4 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride4 = (gridDim.x * blockDim.x);
  while ((idx4 < n7)) {
    iterfunRegularize(expsumbuf, z4, m4, idx4);
    (idx4 = (idx4 + stride4));
  }
}
__global__ void loopKernel5(int64_t n8, Tensor expsumbuf, Tensor z4, int64_t m4) {
  int64_t idx5 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride5 = (gridDim.x * blockDim.x);
  while ((idx5 < n8)) {
    iterfunSummarize(expsumbuf, z4, m4, idx5);
    (idx5 = (idx5 + stride5));
  }
}
__global__ void loopKernel6(int64_t n9, Tensor x5, Tensor z4) {
  int64_t idx6 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride6 = (gridDim.x * blockDim.x);
  while ((idx6 < n9)) {
    iterfun4(x5, z4, idx6);
    (idx6 = (idx6 + stride6));
  }
}
__host__ void tensorOpExn__z___SoftMax_x_(int64_t s_max4, Tensor x5, Tensor expsumbuf, Tensor z4) {
  Seq t58;
  (t58 = tensor_shape((x5.dims), (x5.rank)));
  int64_t s3;
  (s3 = ((t58.seq)[0]));
  int64_t t59;
  (t59 = tensorSize(x5));
  int64_t m4;
  (m4 = (t59 / s3));
  int64_t t60;
  (t60 = (s_max4 * m4));
  {
    int64_t niterations4 = t60;
    int64_t tpb4 = 256;
    int64_t nblocks4 = (((niterations4 + (tpb4 * 10)) - 1) / (tpb4 * 10));
    loopKernel6<<<nblocks4, tpb4>>>(t60, x5, z4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
  {
    int64_t niterations5 = s_max4;
    int64_t tpb5 = 256;
    int64_t nblocks5 = (((niterations5 + (tpb5 * 10)) - 1) / (tpb5 * 10));
    loopKernel5<<<nblocks5, tpb5>>>(s_max4, expsumbuf, z4, m4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
  int64_t t61;
  (t61 = (s_max4 * m4));
  {
    int64_t niterations6 = t61;
    int64_t tpb6 = 256;
    int64_t nblocks6 = (((niterations6 + (tpb6 * 10)) - 1) / (tpb6 * 10));
    loopKernel4<<<nblocks6, tpb6>>>(t61, expsumbuf, z4, m4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun5(Tensor h, Tensor dldh, Tensor z5, int64_t i19) {
  float t62;
  (t62 = ((h.data)[(i19 + (h.offset))]));
  char t63;
  (t63 = (t62 > 0.));
  float dhds_ii;
  if ((t63 == 1)) {
    (dhds_ii = 1.);
  } else {
    (dhds_ii = 0.);
  }
  float dldh_i;
  (dldh_i = ((dldh.data)[(i19 + (dldh.offset))]));
  float t64;
  (t64 = (dhds_ii * dldh_i));
  (((z5.data)[(i19 + (z5.offset))]) = t64);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z5.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z5.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel7(int64_t n10, Tensor h, Tensor dldh, Tensor z5) {
  int64_t idx7 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride7 = (gridDim.x * blockDim.x);
  while ((idx7 < n10)) {
    iterfun5(h, dldh, z5, idx7);
    (idx7 = (idx7 + stride7));
  }
}
__host__ void tensorOpExn__z___d_dx_l_ReLU_x__(int64_t s_max5, Tensor h, Tensor dldh, Tensor z5) {
  Seq t65;
  (t65 = tensor_shape((h.dims), (h.rank)));
  int64_t s4;
  (s4 = ((t65.seq)[0]));
  int64_t t66;
  (t66 = tensorSize(h));
  int64_t m5;
  (m5 = (t66 / s4));
  int64_t t67;
  (t67 = (s_max5 * m5));
  {
    int64_t niterations7 = t67;
    int64_t tpb7 = 256;
    int64_t nblocks7 = (((niterations7 + (tpb7 * 10)) - 1) / (tpb7 * 10));
    loopKernel7<<<nblocks7, tpb7>>>(t67, h, dldh, z5);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ float t68(Tensor p, Tensor dldp, int64_t s_offset, int64_t i20, float p_i, float acc9, int64_t j3) {
  char t69;
  (t69 = (j3 == i20));
  float s_ij;
  if ((t69 == 1)) {
    float t70;
    (t70 = (p_i * p_i));
    (s_ij = (p_i - t70));
  } else {
    int64_t t71;
    (t71 = (s_offset + j3));
    float p_j;
    (p_j = ((p.data)[(t71 + (p.offset))]));
    float t72;
    (t72 = (p_i * p_j));
    (s_ij = (-t72));
  }
  int64_t t73;
  (t73 = (s_offset + j3));
  float dldp_j;
  (dldp_j = ((dldp.data)[(t73 + (dldp.offset))]));
  float t74;
  (t74 = (dldp_j * s_ij));
  {
    float t75;
    (t75 = (acc9 + t74));
    return t75;
  }
}
__host__ __device__ void iterfun6(Tensor p, Tensor dldp, Tensor z6, int64_t m6, int64_t i21) {
  int64_t s_idx5;
  (s_idx5 = (i21 / m6));
  int64_t s_offset;
  (s_offset = (s_idx5 * m6));
  int64_t i20;
  (i20 = (i21 % m6));
  int64_t t76;
  (t76 = (s_offset + i20));
  float p_i;
  (p_i = ((p.data)[(t76 + (p.offset))]));
  float v2;
  {
    int64_t i22 = 0;
    float acc10 = 0.;
    while ((i22 < m6)) {
      (acc10 = t68(p, dldp, s_offset, i20, p_i, acc10, i22));
      (i22 = (i22 + 1));
    }
    (v2 = acc10);
  }
  int64_t t77;
  (t77 = (s_offset + i20));
  (((z6.data)[(t77 + (z6.offset))]) = v2);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z6.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z6.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel8(int64_t n11, Tensor p, Tensor dldp, Tensor z6, int64_t m6) {
  int64_t idx8 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride8 = (gridDim.x * blockDim.x);
  while ((idx8 < n11)) {
    iterfun6(p, dldp, z6, m6, idx8);
    (idx8 = (idx8 + stride8));
  }
}
__host__ void tensorOpExn__z___d_dx_l_SoftMax_x___(int64_t s_max6, Tensor p, Tensor dldp, Tensor z6) {
  Seq t78;
  (t78 = tensor_shape((p.dims), (p.rank)));
  int64_t s5;
  (s5 = ((t78.seq)[0]));
  int64_t t79;
  (t79 = tensorSize(p));
  int64_t m6;
  (m6 = (t79 / s5));
  int64_t t80;
  (t80 = (s_max6 * m6));
  {
    int64_t niterations8 = t80;
    int64_t tpb8 = 256;
    int64_t nblocks8 = (((niterations8 + (tpb8 * 10)) - 1) / (tpb8 * 10));
    loopKernel8<<<nblocks8, tpb8>>>(t80, p, dldp, z6, m6);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun7(Tensor x6, Tensor z7, int64_t i23) {
  float t81;
  (t81 = ((x6.data)[(i23 + (x6.offset))]));
  (((z7.data)[(i23 + (z7.offset))]) = t81);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z7.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z7.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel9(int64_t n12, Tensor x6, Tensor z7) {
  int64_t idx9 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride9 = (gridDim.x * blockDim.x);
  while ((idx9 < n12)) {
    iterfun7(x6, z7, idx9);
    (idx9 = (idx9 + stride9));
  }
}
__host__ void tensorOpExn__z___x(int64_t s_max7, Tensor x6, Tensor z7) {
  Seq t82;
  (t82 = tensor_shape((x6.dims), (x6.rank)));
  int64_t s6;
  (s6 = ((t82.seq)[0]));
  int64_t t83;
  (t83 = tensorSize(x6));
  int64_t m7;
  (m7 = (t83 / s6));
  int64_t t84;
  (t84 = (s_max7 * m7));
  {
    int64_t niterations9 = t84;
    int64_t tpb9 = 256;
    int64_t nblocks9 = (((niterations9 + (tpb9 * 10)) - 1) / (tpb9 * 10));
    loopKernel9<<<nblocks9, tpb9>>>(t84, x6, z7);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun8(float c, Tensor z8, int64_t i24) {
  float t85;
  (t85 = ((z8.data)[(i24 + (z8.offset))]));
  float t86;
  (t86 = (t85 * c));
  (((z8.data)[(i24 + (z8.offset))]) = t86);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z8.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z8.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel10(int64_t n13, float c, Tensor z8) {
  int64_t idx10 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride10 = (gridDim.x * blockDim.x);
  while ((idx10 < n13)) {
    iterfun8(c, z8, idx10);
    (idx10 = (idx10 + stride10));
  }
}
__host__ void tensorOpExn__z____scalar_c_(int64_t s_max8, float c, Tensor z8) {
  Seq t87;
  (t87 = tensor_shape((z8.dims), (z8.rank)));
  int64_t s7;
  (s7 = ((t87.seq)[0]));
  int64_t t88;
  (t88 = tensorSize(z8));
  int64_t m8;
  (m8 = (t88 / s7));
  int64_t t89;
  (t89 = (s_max8 * m8));
  {
    int64_t niterations10 = t89;
    int64_t tpb10 = 256;
    int64_t nblocks10 = (((niterations10 + (tpb10 * 10)) - 1) / (tpb10 * 10));
    loopKernel10<<<nblocks10, tpb10>>>(t89, c, z8);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun9(float c1, Tensor z9, int64_t i25) {
  (((z9.data)[(i25 + (z9.offset))]) = c1);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z9.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z9.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel11(int64_t n14, float c1, Tensor z9) {
  int64_t idx11 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride11 = (gridDim.x * blockDim.x);
  while ((idx11 < n14)) {
    iterfun9(c1, z9, idx11);
    (idx11 = (idx11 + stride11));
  }
}
__host__ void tensorOpExn__Z___scalar_c_(float c1, Tensor z9) {
  int64_t m9;
  (m9 = tensorSize(z9));
  {
    int64_t niterations11 = m9;
    int64_t tpb11 = 256;
    int64_t nblocks11 = (((niterations11 + (tpb11 * 10)) - 1) / (tpb11 * 10));
    loopKernel11<<<nblocks11, tpb11>>>(m9, c1, z9);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun10(Tensor x7, float c2, Tensor z10, int64_t x_offset3, int64_t i26) {
  float t90;
  (t90 = ((z10.data)[(i26 + (z10.offset))]));
  int64_t t91;
  (t91 = (i26 + x_offset3));
  float t92;
  (t92 = ((x7.data)[(t91 + (x7.offset))]));
  float t93;
  (t93 = (t92 * c2));
  float t94;
  (t94 = (t90 + t93));
  (((z10.data)[(i26 + (z10.offset))]) = t94);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z10.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z10.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel12(int64_t n15, Tensor x7, float c2, Tensor z10, int64_t x_offset3) {
  int64_t idx12 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride12 = (gridDim.x * blockDim.x);
  while ((idx12 < n15)) {
    iterfun10(x7, c2, z10, x_offset3, idx12);
    (idx12 = (idx12 + stride12));
  }
}
__host__ void tensorOpExn__Z____x___scalar_c_(int64_t s_idx6, Tensor x7, float c2, Tensor z10) {
  int64_t m10;
  (m10 = tensorSize(z10));
  int64_t x_offset3;
  (x_offset3 = (s_idx6 * m10));
  {
    int64_t niterations12 = m10;
    int64_t tpb12 = 256;
    int64_t nblocks12 = (((niterations12 + (tpb12 * 10)) - 1) / (tpb12 * 10));
    loopKernel12<<<nblocks12, tpb12>>>(m10, x7, c2, z10, x_offset3);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun11(Tensor x8, float c3, Tensor z11, int64_t z_offset1, int64_t i27) {
  int64_t z_idx3;
  (z_idx3 = (i27 + z_offset1));
  float t95;
  (t95 = ((z11.data)[(z_idx3 + (z11.offset))]));
  float t96;
  (t96 = ((x8.data)[(i27 + (x8.offset))]));
  float t97;
  (t97 = (t96 * c3));
  float t98;
  (t98 = (t95 + t97));
  (((z11.data)[(z_idx3 + (z11.offset))]) = t98);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z11.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z11.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel13(int64_t n16, Tensor x8, float c3, Tensor z11, int64_t z_offset1) {
  int64_t idx13 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride13 = (gridDim.x * blockDim.x);
  while ((idx13 < n16)) {
    iterfun11(x8, c3, z11, z_offset1, idx13);
    (idx13 = (idx13 + stride13));
  }
}
__host__ void tensorOpExn__z____X___scalar_c_(int64_t s_idx7, Tensor x8, float c3, Tensor z11) {
  int64_t m11;
  (m11 = tensorSize(x8));
  int64_t z_offset1;
  (z_offset1 = (s_idx7 * m11));
  {
    int64_t niterations13 = m11;
    int64_t tpb13 = 256;
    int64_t nblocks13 = (((niterations13 + (tpb13 * 10)) - 1) / (tpb13 * 10));
    loopKernel13<<<nblocks13, tpb13>>>(m11, x8, c3, z11, z_offset1);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun12(Tensor1 y1, float c4, Tensor z12, int64_t m12, int64_t i28) {
  int64_t idx14;
  (idx14 = ((y1.data)[(i28 + (y1.offset))]));
  int64_t offset2;
  (offset2 = (i28 * m12));
  int64_t z_idx4;
  (z_idx4 = (idx14 + offset2));
  float t99;
  (t99 = ((z12.data)[(z_idx4 + (z12.offset))]));
  float t100;
  (t100 = (t99 + c4));
  (((z12.data)[(z_idx4 + (z12.offset))]) = t100);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z12.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z12.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel14(int64_t n17, Tensor1 y1, float c4, Tensor z12, int64_t m12) {
  int64_t idx15 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride14 = (gridDim.x * blockDim.x);
  while ((idx15 < n17)) {
    iterfun12(y1, c4, z12, m12, idx15);
    (idx15 = (idx15 + stride14));
  }
}
__host__ void tensorOpExp__z____1_Hot_y____scalar_c_(int64_t s_max9, Tensor1 y1, float c4, Tensor z12) {
  Seq t101;
  (t101 = tensor_shape((z12.dims), (z12.rank)));
  int64_t m12;
  (m12 = ((t101.seq)[1]));
  {
    int64_t niterations14 = s_max9;
    int64_t tpb14 = 256;
    int64_t nblocks14 = (((niterations14 + (tpb14 * 10)) - 1) / (tpb14 * 10));
    loopKernel14<<<nblocks14, tpb14>>>(s_max9, y1, c4, z12, m12);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void t102(Tensor z13, int64_t offset3, int64_t idx16, float v3, int64_t j4) {
  char t103;
  (t103 = (j4 == idx16));
  if ((t103 == 1)) {
    int64_t t104;
    (t104 = (j4 + offset3));
    (((z13.data)[(t104 + (z13.offset))]) = v3);
  } else {
    int64_t t105;
    (t105 = (j4 + offset3));
    (((z13.data)[(t105 + (z13.offset))]) = 0.);
  }
}
__host__ __device__ void iterfun13(Tensor1 y2, Tensor x9, Tensor z13, int64_t m13, int64_t i29) {
  int64_t offset3;
  (offset3 = (i29 * m13));
  int64_t idx16;
  (idx16 = ((y2.data)[(i29 + (y2.offset))]));
  int64_t t106;
  (t106 = (idx16 + offset3));
  float t107;
  (t107 = ((x9.data)[(t106 + (x9.offset))]));
  float t108;
  (t108 = (1. / t107));
  float v3;
  (v3 = (-t108));
  {
    int64_t i30 = 0;
    while ((i30 < m13)) {
      t102(z13, offset3, idx16, v3, i30);
      (i30 = (i30 + 1));
    }
  }
}
__global__ void loopKernel15(int64_t n18, Tensor1 y2, Tensor x9, Tensor z13, int64_t m13) {
  int64_t idx17 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride15 = (gridDim.x * blockDim.x);
  while ((idx17 < n18)) {
    iterfun13(y2, x9, z13, m13, idx17);
    (idx17 = (idx17 + stride15));
  }
}
__host__ void tensorOpExn__z___1_Hot_y____scalar__1__x_T___1_Hot_y___(int64_t s_max10, Tensor1 y2, Tensor x9, Tensor z13) {
  Seq t109;
  (t109 = tensor_shape((x9.dims), (x9.rank)));
  int64_t s8;
  (s8 = ((t109.seq)[0]));
  Seq t110;
  (t110 = tensor_shape((x9.dims), (x9.rank)));
  int64_t m13;
  (m13 = ((t110.seq)[1]));
  {
    int64_t niterations15 = s_max10;
    int64_t tpb15 = 256;
    int64_t nblocks15 = (((niterations15 + (tpb15 * 10)) - 1) / (tpb15 * 10));
    loopKernel15<<<nblocks15, tpb15>>>(s_max10, y2, x9, z13, m13);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ float t111(Tensor z14, int64_t m14, int64_t i31, float acc11, int64_t j5) {
  int64_t s_idx8;
  (s_idx8 = (j5 + 1));
  int64_t s_offset1;
  (s_offset1 = (s_idx8 * m14));
  int64_t t112;
  (t112 = (s_offset1 + i31));
  float t113;
  (t113 = ((z14.data)[(t112 + (z14.offset))]));
  {
    float t114;
    (t114 = (acc11 + t113));
    return t114;
  }
}
__host__ __device__ void iterfun14(Tensor z14, int64_t s9, int64_t m14, int64_t i31) {
  int64_t t115;
  (t115 = (s9 - 1));
  float v4;
  {
    int64_t i32 = 0;
    float acc12 = ((z14.data)[(i31 + (z14.offset))]);
    while ((i32 < t115)) {
      (acc12 = t111(z14, m14, i31, acc12, i32));
      (i32 = (i32 + 1));
    }
    (v4 = acc12);
  }
  (((z14.data)[(i31 + (z14.offset))]) = v4);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z14.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z14.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel16(int64_t n19, Tensor z14, int64_t s9, int64_t m14) {
  int64_t idx18 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride16 = (gridDim.x * blockDim.x);
  while ((idx18 < n19)) {
    iterfun14(z14, s9, m14, idx18);
    (idx18 = (idx18 + stride16));
  }
}
__host__ void tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(Tensor z14) {
  Seq t116;
  (t116 = tensor_shape((z14.dims), (z14.rank)));
  int64_t s9;
  (s9 = ((t116.seq)[0]));
  int64_t t117;
  (t117 = tensorSize(z14));
  int64_t m14;
  (m14 = (t117 / s9));
  {
    int64_t niterations16 = m14;
    int64_t tpb16 = 256;
    int64_t nblocks16 = (((niterations16 + (tpb16 * 10)) - 1) / (tpb16 * 10));
    loopKernel16<<<nblocks16, tpb16>>>(m14, z14, s9, m14);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ Tensor nnComponentOutBufs(Rec comp) {
  Tensor X = (comp.out_bufs);
  {
    Tensor t118;
    (t118 = X);
    return t118;
  }
}
__host__ __device__ Tensor nnComponentApplyExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max11, Tensor inputs1, Rec comp1) {
  int64_t ty1;
  int64_t X1 = (comp1.ty);
  (ty1 = X1);
  char t119;
  (t119 = (ty1 == nnCompType_FullyConnected));
  if ((t119 == 1)) {
    Tensor t120;
    Tensor X2 = (comp1.w);
    (t120 = X2);
    Tensor t121;
    Tensor X3 = (comp1.b);
    (t121 = X3);
    Tensor t122;
    Tensor X4 = (comp1.out_bufs);
    (t122 = X4);
    tensorOpExn__z___Wx_B(s_max11, t120, inputs1, t121, t122);
    Tensor X5 = (comp1.out_bufs);
    {
      Tensor t123;
      (t123 = X5);
      return t123;
    }
  } else {
    char t124;
    (t124 = (ty1 == nnCompType_ReLU));
    if ((t124 == 1)) {
      Tensor t125;
      Tensor X6 = (comp1.out_bufs);
      (t125 = X6);
      tensorOpExn__z___ReLU_x_(s_max11, inputs1, t125);
      Tensor X7 = (comp1.out_bufs);
      {
        Tensor t126;
        (t126 = X7);
        return t126;
      }
    } else {
      char t127;
      (t127 = (ty1 == nnCompType_SoftMax));
      if ((t127 == 1)) {
        Tensor t128;
        Tensor X8 = (comp1.softmax_bufs);
        (t128 = X8);
        Tensor t129;
        Tensor X9 = (comp1.out_bufs);
        (t129 = X9);
        tensorOpExn__z___SoftMax_x_(s_max11, inputs1, t128, t129);
        Tensor X10 = (comp1.out_bufs);
        {
          Tensor t130;
          (t130 = X10);
          return t130;
        }
      } else {
        Tensor X11 = (comp1.out_bufs);
        {
          Tensor t131;
          (t131 = X11);
          return t131;
        }
      }
    }
  }
}
__host__ __device__ Tensor nnComponentBackpropExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max12, Tensor comp_inputs, Tensor output_grads, Rec comp2) {
  int64_t ty2;
  int64_t X12 = (comp2.ty);
  (ty2 = X12);
  char t132;
  (t132 = (ty2 == nnCompType_FullyConnected));
  if ((t132 == 1)) {
    Tensor t133;
    Tensor X13 = (comp2.b_grads);
    (t133 = X13);
    tensorOpExn__z___x(s_max12, output_grads, t133);
    Tensor t134;
    Tensor X14 = (comp2.w_grads);
    (t134 = X14);
    tensorOpExn__z___x___y_T(s_max12, output_grads, comp_inputs, t134);
    Tensor t135;
    Tensor X15 = (comp2.w);
    (t135 = X15);
    Tensor t136;
    Tensor X16 = (comp2.in_grads);
    (t136 = X16);
    tensorOpExn__z____x_T___W__T(s_max12, output_grads, t135, t136);
    Tensor X17 = (comp2.in_grads);
    {
      Tensor t137;
      (t137 = X17);
      return t137;
    }
  } else {
    char t138;
    (t138 = (ty2 == nnCompType_ReLU));
    if ((t138 == 1)) {
      Tensor t139;
      Tensor X18 = (comp2.out_bufs);
      (t139 = X18);
      Tensor t140;
      Tensor X19 = (comp2.in_grads);
      (t140 = X19);
      tensorOpExn__z___d_dx_l_ReLU_x__(s_max12, t139, output_grads, t140);
      Tensor X20 = (comp2.in_grads);
      {
        Tensor t141;
        (t141 = X20);
        return t141;
      }
    } else {
      char t142;
      (t142 = (ty2 == nnCompType_SoftMax));
      if ((t142 == 1)) {
        Tensor t143;
        Tensor X21 = (comp2.out_bufs);
        (t143 = X21);
        Tensor t144;
        Tensor X22 = (comp2.in_grads);
        (t144 = X22);
        tensorOpExn__z___d_dx_l_SoftMax_x___(s_max12, t143, output_grads, t144);
        Tensor X23 = (comp2.in_grads);
        {
          Tensor t145;
          (t145 = X23);
          return t145;
        }
      } else {
        Tensor X24 = (comp2.in_grads);
        {
          Tensor t146;
          (t146 = X24);
          return t146;
        }
      }
    }
  }
}
__host__ __device__ void nnComponent_TEMP_SetGradients(int64_t nnCompType_FullyConnected, float scalar, Rec comp3) {
  int64_t ty3;
  int64_t X25 = (comp3.ty);
  (ty3 = X25);
  char t147;
  (t147 = (ty3 == nnCompType_FullyConnected));
  if ((t147 == 1)) {
    Tensor t148;
    Tensor X26 = (comp3.w_grads);
    (t148 = X26);
    tensorOpExn__Z___scalar_c_(scalar, t148);
    Tensor t149;
    Tensor X27 = (comp3.b_grads);
    (t149 = X27);
    tensorOpExn__Z___scalar_c_(scalar, t149);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ReduceGradients(int64_t nnCompType_FullyConnected, Rec comp4) {
  int64_t ty4;
  int64_t X28 = (comp4.ty);
  (ty4 = X28);
  char t150;
  (t150 = (ty4 == nnCompType_FullyConnected));
  if ((t150 == 1)) {
    Tensor t151;
    Tensor X29 = (comp4.w_grads);
    (t151 = X29);
    tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(t151);
    Tensor t152;
    Tensor X30 = (comp4.b_grads);
    (t152 = X30);
    tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(t152);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ScaleGradients(int64_t nnCompType_FullyConnected, float scalar1, Rec comp5) {
  int64_t ty5;
  int64_t X31 = (comp5.ty);
  (ty5 = X31);
  char t153;
  (t153 = (ty5 == nnCompType_FullyConnected));
  if ((t153 == 1)) {
    Tensor t154;
    Tensor X32 = (comp5.w_grads);
    (t154 = X32);
    tensorOpExn__z____scalar_c_(1, scalar1, t154);
    Tensor t155;
    Tensor X33 = (comp5.b_grads);
    (t155 = X33);
    tensorOpExn__z____scalar_c_(1, scalar1, t155);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ApplyGradients(int64_t nnCompType_FullyConnected, float scalar2, Rec comp6) {
  int64_t ty6;
  int64_t X34 = (comp6.ty);
  (ty6 = X34);
  char t156;
  (t156 = (ty6 == nnCompType_FullyConnected));
  if ((t156 == 1)) {
    Tensor t157;
    Tensor X35 = (comp6.w_grads);
    (t157 = X35);
    Tensor t158;
    Tensor X36 = (comp6.w);
    (t158 = X36);
    tensorOpExn__Z____x___scalar_c_(0, t157, scalar2, t158);
    Tensor t159;
    Tensor X37 = (comp6.b_grads);
    (t159 = X37);
    Tensor t160;
    Tensor X38 = (comp6.b);
    (t160 = X38);
    tensorOpExn__Z____x___scalar_c_(0, t159, scalar2, t160);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_L2Regularize(int64_t nnCompType_FullyConnected, float scalar3, Rec comp7) {
  int64_t ty7;
  int64_t X39 = (comp7.ty);
  (ty7 = X39);
  char t161;
  (t161 = (ty7 == nnCompType_FullyConnected));
  if ((t161 == 1)) {
    Tensor t162;
    Tensor X40 = (comp7.w);
    (t162 = X40);
    Tensor t163;
    Tensor X41 = (comp7.w_grads);
    (t163 = X41);
    tensorOpExn__z____X___scalar_c_(0, t162, scalar3, t163);
    Tensor t164;
    Tensor X42 = (comp7.b);
    (t164 = X42);
    Tensor t165;
    Tensor X43 = (comp7.b_grads);
    (t165 = X43);
    tensorOpExn__z____X___scalar_c_(0, t164, scalar3, t165);
  } else {
    ;
  }
}
__host__ __device__ void nnComponentZeroGrad(int64_t nnCompType_FullyConnected, Rec comp8) {
  nnComponent_TEMP_SetGradients(nnCompType_FullyConnected, 0., comp8);
}
__host__ __device__ Tensor nnLossFunctionApplyExn(int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, int64_t s_max13, Tensor inputs2, Rec1 lossfn1) {
  int64_t ty8;
  int64_t X44 = (lossfn1.ty);
  (ty8 = X44);
  char t166;
  (t166 = (ty8 == nnLossfnType_CrossEntropyLoss));
  if ((t166 == 1)) {
    {
      Tensor t167;
      (t167 = inputs2);
      return t167;
    }
  } else {
    char t168;
    (t168 = (ty8 == nnLossfnType_SoftMaxCrossEntropyLoss));
    if ((t168 == 1)) {
      Tensor t169;
      Tensor X45 = (lossfn1.softmax_bufs);
      (t169 = X45);
      Tensor t170;
      Tensor X46 = (lossfn1.in_grads);
      (t170 = X46);
      tensorOpExn__z___SoftMax_x_(s_max13, inputs2, t169, t170);
      Tensor X47 = (lossfn1.in_grads);
      {
        Tensor t171;
        (t171 = X47);
        return t171;
      }
    } else {
      {
        Tensor t172;
        (t172 = inputs2);
        return t172;
      }
    }
  }
}
__host__ __device__ Tensor nnLossFunctionBackpropExn(int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, int64_t s_max14, Tensor inputs3, Tensor1 expecteds, Rec1 lossfn2) {
  int64_t ty9;
  int64_t X48 = (lossfn2.ty);
  (ty9 = X48);
  char t173;
  (t173 = (ty9 == nnLossfnType_CrossEntropyLoss));
  if ((t173 == 1)) {
    Tensor t174;
    Tensor X49 = (lossfn2.in_grads);
    (t174 = X49);
    tensorOpExn__z___1_Hot_y____scalar__1__x_T___1_Hot_y___(s_max14, expecteds, inputs3, t174);
    Tensor X50 = (lossfn2.in_grads);
    {
      Tensor t175;
      (t175 = X50);
      return t175;
    }
  } else {
    char t176;
    (t176 = (ty9 == nnLossfnType_SoftMaxCrossEntropyLoss));
    if ((t176 == 1)) {
      float t177;
      (t177 = (-1.));
      Tensor t178;
      Tensor X51 = (lossfn2.in_grads);
      (t178 = X51);
      tensorOpExp__z____1_Hot_y____scalar_c_(s_max14, expecteds, t177, t178);
      Tensor X52 = (lossfn2.in_grads);
      {
        Tensor t179;
        (t179 = X52);
        return t179;
      }
    } else {
      Tensor X53 = (lossfn2.in_grads);
      {
        Tensor t180;
        (t180 = X53);
        return t180;
      }
    }
  }
}
__host__ __device__ int64_t t181(int64_t nnCompType_FullyConnected, int64_t x10, Rec comp9) {
  nnComponentZeroGrad(nnCompType_FullyConnected, comp9);
  {
    int64_t t182;
    (t182 = 0);
    return t182;
  }
}
__host__ __device__ int64_t foldl1(int64_t acc_init2, Seq1 s10, int64_t nnCompType_FullyConnected) {
  int64_t acc13 = acc_init2;
  int64_t i33 = 0;
  while ((i33 < (s10.len))) {
    (acc13 = t181(nnCompType_FullyConnected, acc13, ((s10.seq)[i33])));
    (i33 = (i33 + 1));
  }
  return acc13;
}
__host__ __device__ void nnZeroGrad(int64_t nnCompType_FullyConnected, Rec2 network) {
  Seq1 t183;
  Seq1 X54 = (network.components);
  (t183 = X54);
  int64_t _2;
  (_2 = foldl1(0, t183, nnCompType_FullyConnected));
  ;
}
__host__ __device__ Tensor t184(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max15, Tensor prevouts, Rec comp10) {
  {
    Tensor t185;
    (t185 = nnComponentApplyExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, prevouts, comp10));
    return t185;
  }
}
__host__ __device__ Tensor foldl2(Tensor acc_init3, Seq1 s11, int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max15) {
  Tensor acc14 = acc_init3;
  int64_t i34 = 0;
  while ((i34 < (s11.len))) {
    (acc14 = t184(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, acc14, ((s11.seq)[i34])));
    (i34 = (i34 + 1));
  }
  return acc14;
}
__host__ __device__ Tensor nnEvalExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network1, Tensor inputs4) {
  Seq t186;
  (t186 = tensor_shape((inputs4.dims), (inputs4.rank)));
  int64_t s_max15;
  (s_max15 = ((t186.seq)[0]));
  Seq1 t187;
  Seq1 X55 = (network1.components);
  (t187 = X55);
  Tensor comp_out;
  (comp_out = foldl2(inputs4, t187, nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15));
  Rec1 t188;
  Rec1 X56 = (network1.lossfn);
  (t188 = X56);
  {
    Tensor t189;
    (t189 = nnLossFunctionApplyExn(nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, s_max15, comp_out, t188));
    return t189;
  }
}
__host__ __device__ Tensor t190(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, Rec2 network2, int64_t s_max16, int64_t n_components, Tensor out_grads, int64_t i35) {
  int64_t t191;
  (t191 = (i35 + 2));
  int64_t idx19;
  (idx19 = (n_components - t191));
  int64_t previdx;
  (previdx = (idx19 - 1));
  Seq1 t192;
  Seq1 X57 = (network2.components);
  (t192 = X57);
  Rec comp11;
  (comp11 = ((t192.seq)[idx19]));
  Seq1 t193;
  Seq1 X58 = (network2.components);
  (t193 = X58);
  Rec t194;
  (t194 = ((t193.seq)[previdx]));
  Tensor in_bufs;
  (in_bufs = nnComponentOutBufs(t194));
  {
    Tensor t195;
    (t195 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, in_bufs, out_grads, comp11));
    return t195;
  }
}
__host__ __device__ void nnBackpropExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network2, Rec3 batch) {
  Tensor t196;
  Tensor X59 = (batch.inputs);
  (t196 = X59);
  Seq t197;
  (t197 = tensor_shape((t196.dims), (t196.rank)));
  int64_t s_max16;
  (s_max16 = ((t197.seq)[0]));
  Tensor t198;
  Tensor X60 = (batch.inputs);
  (t198 = X60);
  Tensor outputs;
  (outputs = nnEvalExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network2, t198));
  Tensor1 t199;
  Tensor1 X61 = (batch.correct_linear_outidxs);
  (t199 = X61);
  Rec1 t200;
  Rec1 X62 = (network2.lossfn);
  (t200 = X62);
  Tensor lossgrads;
  (lossgrads = nnLossFunctionBackpropExn(nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, s_max16, outputs, t199, t200));
  Seq1 t201;
  Seq1 X63 = (network2.components);
  (t201 = X63);
  int64_t n_components;
  (n_components = (t201.len));
  char t202;
  (t202 = (n_components == 0));
  if ((t202 == 1)) {
    ;
  } else {
    char t203;
    (t203 = (n_components == 1));
    if ((t203 == 1)) {
      Seq1 t204;
      Seq1 X64 = (network2.components);
      (t204 = X64);
      Rec lastcomp;
      (lastcomp = ((t204.seq)[0]));
      Tensor t205;
      Tensor X65 = (batch.inputs);
      (t205 = X65);
      Tensor _3;
      (_3 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, t205, lossgrads, lastcomp));
      ;
    } else {
      Seq1 t206;
      Seq1 X66 = (network2.components);
      (t206 = X66);
      int64_t t207;
      (t207 = (n_components - 1));
      Rec lastcomp1;
      (lastcomp1 = ((t206.seq)[t207]));
      Seq1 t208;
      Seq1 X67 = (network2.components);
      (t208 = X67);
      int64_t t209;
      (t209 = (n_components - 2));
      Rec t210;
      (t210 = ((t208.seq)[t209]));
      Tensor lastcomp_in_bufs;
      (lastcomp_in_bufs = nnComponentOutBufs(t210));
      Tensor lastcomp_in_grads;
      (lastcomp_in_grads = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, lastcomp_in_bufs, lossgrads, lastcomp1));
      int64_t t211;
      (t211 = (n_components - 2));
      Tensor firstcomp_out_grads;
      {
        int64_t i36 = 0;
        Tensor acc15 = lastcomp_in_grads;
        while ((i36 < t211)) {
          (acc15 = t190(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, network2, s_max16, n_components, acc15, i36));
          (i36 = (i36 + 1));
        }
        (firstcomp_out_grads = acc15);
      }
      Seq1 t212;
      Seq1 X68 = (network2.components);
      (t212 = X68);
      Rec firstcomp;
      (firstcomp = ((t212.seq)[0]));
      Tensor firstcomp_in_bufs;
      Tensor X69 = (batch.inputs);
      (firstcomp_in_bufs = X69);
      Tensor _4;
      (_4 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, firstcomp_in_bufs, firstcomp_out_grads, firstcomp));
      ;
    }
  }
}
__host__ __device__ int64_t t213(int64_t nnCompType_FullyConnected, float batchsize_regularizer, int64_t x11, Rec comp12) {
  nnComponent_TEMP_ReduceGradients(nnCompType_FullyConnected, comp12);
  nnComponent_TEMP_ScaleGradients(nnCompType_FullyConnected, batchsize_regularizer, comp12);
  {
    int64_t t214;
    (t214 = 0);
    return t214;
  }
}
__host__ __device__ int64_t t215(int64_t nnCompType_FullyConnected, float lambda, int64_t x12, Rec comp13) {
  float t216;
  (t216 = (2. * lambda));
  nnComponent_TEMP_L2Regularize(nnCompType_FullyConnected, t216, comp13);
  {
    int64_t t217;
    (t217 = 0);
    return t217;
  }
}
__host__ __device__ int64_t t218(int64_t nnCompType_FullyConnected, float alpha, int64_t x13, Rec comp14) {
  float t219;
  (t219 = (-alpha));
  nnComponent_TEMP_ApplyGradients(nnCompType_FullyConnected, t219, comp14);
  {
    int64_t t220;
    (t220 = 0);
    return t220;
  }
}
__host__ __device__ int64_t foldl3(int64_t acc_init4, Seq1 s12, int64_t nnCompType_FullyConnected, float alpha) {
  int64_t acc16 = acc_init4;
  int64_t i37 = 0;
  while ((i37 < (s12.len))) {
    (acc16 = t218(nnCompType_FullyConnected, alpha, acc16, ((s12.seq)[i37])));
    (i37 = (i37 + 1));
  }
  return acc16;
}
__host__ __device__ int64_t foldl4(int64_t acc_init5, Seq1 s13, int64_t nnCompType_FullyConnected, float lambda) {
  int64_t acc17 = acc_init5;
  int64_t i38 = 0;
  while ((i38 < (s13.len))) {
    (acc17 = t215(nnCompType_FullyConnected, lambda, acc17, ((s13.seq)[i38])));
    (i38 = (i38 + 1));
  }
  return acc17;
}
__host__ __device__ int64_t foldl5(int64_t acc_init6, Seq1 s14, int64_t nnCompType_FullyConnected, float batchsize_regularizer) {
  int64_t acc18 = acc_init6;
  int64_t i39 = 0;
  while ((i39 < (s14.len))) {
    (acc18 = t213(nnCompType_FullyConnected, batchsize_regularizer, acc18, ((s14.seq)[i39])));
    (i39 = (i39 + 1));
  }
  return acc18;
}
__host__ __device__ void nnGradientDescentExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network3, float alpha, float lambda, Rec3 batch1) {
  nnZeroGrad(nnCompType_FullyConnected, network3);
  Tensor t221;
  Tensor X70 = (batch1.inputs);
  (t221 = X70);
  Seq t222;
  (t222 = tensor_shape((t221.dims), (t221.rank)));
  int64_t batchsize1;
  (batchsize1 = ((t222.seq)[0]));
  nnBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network3, batch1);
  float t223;
  (t223 = (( float ) batchsize1));
  float batchsize_regularizer;
  (batchsize_regularizer = (1. / t223));
  Seq1 t224;
  Seq1 X71 = (network3.components);
  (t224 = X71);
  int64_t _5;
  (_5 = foldl5(0, t224, nnCompType_FullyConnected, batchsize_regularizer));
  char t225;
  (t225 = (lambda == 0.));
  if ((t225 == 1)) {
    ;
  } else {
    Seq1 t226;
    Seq1 X72 = (network3.components);
    (t226 = X72);
    int64_t _6;
    (_6 = foldl4(0, t226, nnCompType_FullyConnected, lambda));
    ;
  }
  Seq1 t227;
  Seq1 X73 = (network3.components);
  (t227 = X73);
  int64_t _7;
  (_7 = foldl3(0, t227, nnCompType_FullyConnected, alpha));
  ;
}
__host__ __device__ int64_t t228(int64_t acc19, int64_t e1) {
  {
    int64_t t229;
    (t229 = (acc19 * e1));
    return t229;
  }
}
__host__ __device__ int64_t foldl6(int64_t acc_init7, Seq s15) {
  int64_t acc20 = acc_init7;
  int64_t i40 = 0;
  while ((i40 < (s15.len))) {
    (acc20 = t228(acc20, ((s15.seq)[i40])));
    (i40 = (i40 + 1));
  }
  return acc20;
}
__host__ __device__ int64_t tensorSize1(Tensor t230) {
  Seq t231;
  (t231 = tensor_shape((t230.dims), (t230.rank)));
  {
    int64_t t232;
    (t232 = foldl6(1, t231));
    return t232;
  }
}
__host__ __device__ int64_t t233(Tensor t234, int64_t s_offset2, int64_t cand_idx, int64_t idx20) {
  int64_t t235;
  (t235 = (idx20 + s_offset2));
  float t236;
  (t236 = ((t234.data)[(t235 + (t234.offset))]));
  int64_t t237;
  (t237 = (cand_idx + s_offset2));
  float t238;
  (t238 = ((t234.data)[(t237 + (t234.offset))]));
  char t239;
  (t239 = (t236 > t238));
  if ((t239 == 1)) {
    {
      int64_t t240;
      (t240 = idx20);
      return t240;
    }
  } else {
    {
      int64_t t241;
      (t241 = cand_idx);
      return t241;
    }
  }
}
__host__ __device__ int64_t tensorLinearMaxIdx(int64_t s_idx9, Tensor t234) {
  Seq t242;
  (t242 = tensor_shape((t234.dims), (t234.rank)));
  int64_t s16;
  (s16 = ((t242.seq)[0]));
  int64_t t243;
  (t243 = tensorSize1(t234));
  int64_t size1;
  (size1 = (t243 / s16));
  int64_t s_offset2;
  (s_offset2 = (s_idx9 * size1));
  int64_t t244;
  (t244 = (size1 - 1));
  {
    int64_t t245;
    {
      int64_t i41 = 0;
      int64_t acc21 = (size1 - 1);
      while ((i41 < t244)) {
        (acc21 = t233(t234, s_offset2, acc21, i41));
        (i41 = (i41 + 1));
      }
      (t245 = acc21);
    }
    return t245;
  }
}
__host__ __device__ int64_t t246(Rec3 batch2, Tensor outputs1, int64_t b_acc, int64_t b_idx) {
  int64_t seqAlloc[1];
  Seq t247;
  int64_t t248;
  (t248 = tensorLinearMaxIdx(b_idx, outputs1));
  Tensor1 t249;
  Tensor1 X74 = (batch2.correct_linear_outidxs);
  (t249 = X74);
  ((seqAlloc[0]) = b_idx);
  ((t247.seq) = seqAlloc);
  ((t247.len) = 1);
  int64_t t250;
  (t250 = ((t249.data)[(cartesian_to_linear_index((t249.dims), (t249.rank), t247) + (t249.offset))]));
  char t251;
  (t251 = (t248 == t250));
  if ((t251 == 1)) {
    {
      int64_t t252;
      (t252 = (b_acc + 1));
      return t252;
    }
  } else {
    {
      int64_t t253;
      (t253 = b_acc);
      return t253;
    }
  }
}
__host__ __device__ int64_t t254(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, char printStatus1, Rec2 network4, Seq2 batches, int64_t acc22, int64_t i42) {
  char (*t255) = "/";
  char (*t256) = "\r";
  if ((printStatus1 == 1)) {
    printf("%s", t256);
    int64_t t257;
    (t257 = (i42 + 1));
    float t258;
    (t258 = (( float ) t257));
    printf("%f", t258);
    printf("%s", t255);
    int64_t t259;
    (t259 = (batches.len));
    float t260;
    (t260 = (( float ) t259));
    printf("%f", t260);
  } else {
    ;
  }
  Rec3 batch2;
  (batch2 = ((batches.seq)[i42]));
  Tensor t261;
  Tensor X75 = (batch2.inputs);
  (t261 = X75);
  Seq t262;
  (t262 = tensor_shape((t261.dims), (t261.rank)));
  int64_t batchsize2;
  (batchsize2 = ((t262.seq)[0]));
  Tensor t263;
  Tensor X76 = (batch2.inputs);
  (t263 = X76);
  Tensor outputs1;
  (outputs1 = nnEvalExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network4, t263));
  int64_t batch_accuracy;
  {
    int64_t i43 = 0;
    int64_t acc23 = 0;
    while ((i43 < batchsize2)) {
      (acc23 = t246(batch2, outputs1, acc23, i43));
      (i43 = (i43 + 1));
    }
    (batch_accuracy = acc23);
  }
  {
    int64_t t264;
    (t264 = (acc22 + batch_accuracy));
    return t264;
  }
}
__host__ __device__ int64_t nnAccuracyDiscrete(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, char printStatus1, Rec2 network4, Seq2 batches) {
  char (*t265) = "\n";
  int64_t t266;
  (t266 = (batches.len));
  int64_t correct_guesses;
  {
    int64_t i44 = 0;
    int64_t acc24 = 0;
    while ((i44 < t266)) {
      (acc24 = t254(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, printStatus1, network4, batches, acc24, i44));
      (i44 = (i44 + 1));
    }
    (correct_guesses = acc24);
  }
  if ((printStatus1 == 1)) {
    printf("%s", t265);
  } else {
    ;
  }
  {
    int64_t t267;
    (t267 = correct_guesses);
    return t267;
  }
}
__host__ __device__ int64_t t268(int64_t acc25, Rec3 batch3) {
  Tensor t269;
  Tensor X77 = (batch3.inputs);
  (t269 = X77);
  Seq t270;
  (t270 = tensor_shape((t269.dims), (t269.rank)));
  int64_t t271;
  (t271 = ((t270.seq)[0]));
  {
    int64_t t272;
    (t272 = (acc25 + t271));
    return t272;
  }
}
__host__ __device__ int64_t foldl7(int64_t acc_init8, Seq2 s17) {
  int64_t acc26 = acc_init8;
  int64_t i45 = 0;
  while ((i45 < (s17.len))) {
    (acc26 = t268(acc26, ((s17.seq)[i45])));
    (i45 = (i45 + 1));
  }
  return acc26;
}
__host__ __device__ float nnAccuracyProportion(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, char printStatus2, Rec2 network5, Seq2 batches1) {
  int64_t correct_guesses1;
  (correct_guesses1 = nnAccuracyDiscrete(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, printStatus2, network5, batches1));
  int64_t datalength;
  (datalength = foldl7(0, batches1));
  float t273;
  (t273 = (( float ) correct_guesses1));
  float t274;
  (t274 = (( float ) datalength));
  {
    float t275;
    (t275 = (t273 / t274));
    return t275;
  }
}
__host__ __device__ void wrappedPrint(char (*s18)) {
  printf("%s", s18);
}
__host__ __device__ void t276(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, float alpha1, float lambda1, int64_t batch_idx) {
  char (*t277) = "/";
  char (*t278) = "\rround ";
  char t279;
  char X78 = (params.printStatus);
  (t279 = X78);
  if ((t279 == 1)) {
    wrappedPrint(t278);
    int64_t t280;
    (t280 = (batch_idx + 1));
    float t281;
    (t281 = (( float ) t280));
    printf("%f", t281);
    wrappedPrint(t277);
    float t282;
    (t282 = (( float ) rounds));
    printf("%f", t282);
  } else {
    ;
  }
  Rec3 t283;
  (t283 = ((training_batches.seq)[batch_idx]));
  nnGradientDescentExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network6, alpha1, lambda1, t283);
}
__host__ __device__ Rec5 t284(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, Seq2 validation_batches, Rec5 acc27, int64_t epoch_idx) {
  Rec5 alloc;
  char (*t285) = "%\n";
  char (*t286) = "Computed accuracy: ";
  char (*t287) = "evalating performance...\n";
  char (*t288) = "\n";
  char (*t289) = "]\n";
  char (*t290) = "[lambda = ";
  char (*t291) = "]\n";
  char (*t292) = "[alpha = ";
  char (*t293) = "]\n";
  char (*t294) = "/";
  char (*t295) = "[Iteration ";
  int64_t epoch;
  (epoch = (epoch_idx + 1));
  float alpha1;
  float X79 = (acc27._0);
  (alpha1 = X79);
  float lambda1;
  float X80 = (acc27._1);
  (lambda1 = X80);
  char t296;
  char X81 = (params.printStatus);
  (t296 = X81);
  if ((t296 == 1)) {
    wrappedPrint(t295);
    float t297;
    (t297 = (( float ) epoch));
    printf("%f", t297);
    wrappedPrint(t294);
    int64_t t298;
    int64_t X82 = (params.epochs);
    (t298 = X82);
    float t299;
    (t299 = (( float ) t298));
    printf("%f", t299);
    printf("%s", t293);
    wrappedPrint(t292);
    printf("%f", alpha1);
    wrappedPrint(t291);
    wrappedPrint(t290);
    printf("%f", lambda1);
    wrappedPrint(t289);
  } else {
    ;
  }
  int64_t t300;
  (t300 = (training_batches.len));
  {
    int64_t i46 = 0;
    while ((i46 < t300)) {
      t276(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, params, network6, rounds, training_batches, alpha1, lambda1, i46);
      (i46 = (i46 + 1));
    }
  }
  wrappedPrint(t288);
  char t301;
  char X83 = (params.evaluateBetweenEpochs);
  (t301 = X83);
  if ((t301 == 1)) {
    char t302;
    char X84 = (params.printStatus);
    (t302 = X84);
    if ((t302 == 1)) {
      wrappedPrint(t287);
    } else {
      ;
    }
    char t303;
    char X85 = (params.printStatus);
    (t303 = X85);
    float accuracy;
    (accuracy = nnAccuracyProportion(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, t303, network6, validation_batches));
    char t304;
    char X86 = (params.printStatus);
    (t304 = X86);
    if ((t304 == 1)) {
      wrappedPrint(t286);
      float t305;
      (t305 = (accuracy * 100.));
      printf("%f", t305);
      wrappedPrint(t285);
    } else {
      ;
    }
  } else {
    ;
  }
  float t306;
  float X87 = (params.decay_alpha);
  (t306 = X87);
  float t307;
  (t307 = (1. - t306));
  float decayed_alpha;
  (decayed_alpha = (alpha1 * t307));
  float t308;
  float X88 = (params.decay_lambda);
  (t308 = X88);
  float t309;
  (t309 = (1. - t308));
  float decayed_lambda;
  (decayed_lambda = (lambda1 * t309));
  ((alloc._0) = decayed_alpha);
  ((alloc._1) = decayed_lambda);
  {
    Rec5 t310;
    (t310 = alloc);
    return t310;
  }
}
void cuda_wrap(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, Seq2 validation_batches) {
  Rec5 t311;
  char (*t312) = "%\n";
  char (*t313) = "Computed accuracy: ";
  char (*t314) = "evalating performance...\n";
  char t315;
  char X89 = (params.evaluateBeforeFirstEpoch);
  (t315 = X89);
  if ((t315 == 1)) {
    char t316;
    char X90 = (params.printStatus);
    (t316 = X90);
    if ((t316 == 1)) {
      wrappedPrint(t314);
    } else {
      ;
    }
    char t317;
    char X91 = (params.printStatus);
    (t317 = X91);
    float accuracy1;
    (accuracy1 = nnAccuracyProportion(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, t317, network6, validation_batches));
    char t318;
    char X92 = (params.printStatus);
    (t318 = X92);
    if ((t318 == 1)) {
      wrappedPrint(t313);
      float t319;
      (t319 = (accuracy1 * 100.));
      printf("%f", t319);
      wrappedPrint(t312);
    } else {
      ;
    }
  } else {
    ;
  }
  float t320;
  float X93 = (params.init_lambda);
  (t320 = X93);
  float t321;
  float X94 = (params.init_alpha);
  (t321 = X94);
  ((t311._0) = t321);
  ((t311._1) = t320);
  int64_t t322;
  int64_t X95 = (params.epochs);
  (t322 = X95);
  Rec5 _8;
  {
    int64_t i47 = 0;
    Rec5 acc28 = t311;
    while ((i47 < t322)) {
      (acc28 = t284(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, params, network6, rounds, training_batches, validation_batches, acc28, i47));
      (i47 = (i47 + 1));
    }
    (_8 = acc28);
  }
  ;
}
extern "C" void vv4F90aVQU5(value nnCompType_FullyConnected, value nnCompType_ReLU, value nnCompType_SoftMax, value nnLossfnType_CrossEntropyLoss, value nnLossfnType_SoftMaxCrossEntropyLoss, value params, value network6, value rounds, value training_batches, value validation_batches) {
  CAMLparam5(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss);
  CAMLxparam5(params, network6, rounds, training_batches, validation_batches);
  // Creating cuBlas context
  hipblasCreate(&_cublas_handle);
  GPU_UTILS_CHECK_CUDA_ERROR();

  int64_t tensor_count = 0;
  int64_t gpu_tmp = Long_val(nnCompType_FullyConnected);
  int64_t gpu_tmp1 = Long_val(nnCompType_ReLU);
  int64_t gpu_tmp2 = Long_val(nnCompType_SoftMax);
  int64_t gpu_tmp3 = Long_val(nnLossfnType_CrossEntropyLoss);
  int64_t gpu_tmp4 = Long_val(nnLossfnType_SoftMaxCrossEntropyLoss);
  Rec4 gpu_tmp5;
  int64_t cuda_rec_field = Long_val(Field(params, 0));
  ((gpu_tmp5.epochs) = cuda_rec_field);
  int64_t cuda_rec_field1 = Long_val(Field(params, 1));
  ((gpu_tmp5.batchsize) = cuda_rec_field1);
  float cuda_rec_field2 = Double_val(Field(params, 2));
  ((gpu_tmp5.init_alpha) = cuda_rec_field2);
  float cuda_rec_field3 = Double_val(Field(params, 3));
  ((gpu_tmp5.decay_alpha) = cuda_rec_field3);
  float cuda_rec_field4 = Double_val(Field(params, 4));
  ((gpu_tmp5.init_lambda) = cuda_rec_field4);
  char cuda_rec_field5 = Int_val(Field(params, 5));
  ((gpu_tmp5.printStatus) = cuda_rec_field5);
  float cuda_rec_field6 = Double_val(Field(params, 6));
  ((gpu_tmp5.decay_lambda) = cuda_rec_field6);
  char cuda_rec_field7 = Int_val(Field(params, 7));
  ((gpu_tmp5.evaluateBetweenEpochs) = cuda_rec_field7);
  char cuda_rec_field8 = Int_val(Field(params, 8));
  ((gpu_tmp5.evaluateBeforeFirstEpoch) = cuda_rec_field8);
  Rec2 gpu_tmp6;
  Rec1 cuda_rec_field9;
  int64_t cuda_rec_field10 = Long_val(Field(Field(network6, 0), 0));
  ((cuda_rec_field9.ty) = cuda_rec_field10);
  Tensor cuda_rec_field11;
  ((cuda_rec_field11.rank) = (Caml_ba_array_val(Field(Field(network6, 0), 1))->num_dims));
  if (((cuda_rec_field11.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field11.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field11.offset) = 0);
  ((cuda_rec_field11.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i48 = 0;
  int64_t n20 = (sizeof(float));
  while ((i48 < (cuda_rec_field11.rank))) {
    (((cuda_rec_field11.dims)[i48]) = ((Caml_ba_array_val(Field(Field(network6, 0), 1))->dim)[i48]));
    (n20 = (n20 * ((Caml_ba_array_val(Field(Field(network6, 0), 1))->dim)[i48])));
    (i48 = (i48 + 1));
  }
  ((cuda_rec_field11.size) = n20);
  float (*t323);
  hipMallocManaged((&t323), n20);
  GPU_UTILS_CHECK_CUDA_ERROR();
  double (*t_ocaml) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 1)));
  int64_t i49 = 0;
  int64_t elems = (n20 / (sizeof(float)));
  while ((i49 < elems)) {
    ((t323[i49]) = (( float ) (t_ocaml[i49])));
    (i49 = (i49 + 1));
  }
  ((cuda_rec_field11.data) = t323);
  ((cuda_rec_field9.in_grads) = cuda_rec_field11);
  Tensor cuda_rec_field12;
  ((cuda_rec_field12.rank) = (Caml_ba_array_val(Field(Field(network6, 0), 2))->num_dims));
  if (((cuda_rec_field12.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field12.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field12.offset) = 0);
  ((cuda_rec_field12.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i50 = 0;
  int64_t n21 = (sizeof(float));
  while ((i50 < (cuda_rec_field12.rank))) {
    (((cuda_rec_field12.dims)[i50]) = ((Caml_ba_array_val(Field(Field(network6, 0), 2))->dim)[i50]));
    (n21 = (n21 * ((Caml_ba_array_val(Field(Field(network6, 0), 2))->dim)[i50])));
    (i50 = (i50 + 1));
  }
  ((cuda_rec_field12.size) = n21);
  float (*t324);
  hipMallocManaged((&t324), n21);
  GPU_UTILS_CHECK_CUDA_ERROR();
  double (*t_ocaml1) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 2)));
  int64_t i51 = 0;
  int64_t elems1 = (n21 / (sizeof(float)));
  while ((i51 < elems1)) {
    ((t324[i51]) = (( float ) (t_ocaml1[i51])));
    (i51 = (i51 + 1));
  }
  ((cuda_rec_field12.data) = t324);
  ((cuda_rec_field9.out_bufs) = cuda_rec_field12);
  Tensor cuda_rec_field13;
  ((cuda_rec_field13.rank) = (Caml_ba_array_val(Field(Field(network6, 0), 3))->num_dims));
  if (((cuda_rec_field13.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field13.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field13.offset) = 0);
  ((cuda_rec_field13.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i52 = 0;
  int64_t n22 = (sizeof(float));
  while ((i52 < (cuda_rec_field13.rank))) {
    (((cuda_rec_field13.dims)[i52]) = ((Caml_ba_array_val(Field(Field(network6, 0), 3))->dim)[i52]));
    (n22 = (n22 * ((Caml_ba_array_val(Field(Field(network6, 0), 3))->dim)[i52])));
    (i52 = (i52 + 1));
  }
  ((cuda_rec_field13.size) = n22);
  float (*t325);
  hipMallocManaged((&t325), n22);
  GPU_UTILS_CHECK_CUDA_ERROR();
  double (*t_ocaml2) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 3)));
  int64_t i53 = 0;
  int64_t elems2 = (n22 / (sizeof(float)));
  while ((i53 < elems2)) {
    ((t325[i53]) = (( float ) (t_ocaml2[i53])));
    (i53 = (i53 + 1));
  }
  ((cuda_rec_field13.data) = t325);
  ((cuda_rec_field9.softmax_bufs) = cuda_rec_field13);
  ((gpu_tmp6.lossfn) = cuda_rec_field9);
  Seq1 cuda_rec_field14;
  ((cuda_rec_field14.len) = Wosize_val(Field(network6, 1)));
  hipMallocManaged((&(cuda_rec_field14.seq)), (Wosize_val(Field(network6, 1)) * (sizeof(Rec))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i54 = 0;
  while ((i54 < (cuda_rec_field14.len))) {
    Rec cuda_seq_temp;
    Tensor cuda_rec_field15;
    ((cuda_rec_field15.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 0))->num_dims));
    if (((cuda_rec_field15.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field15.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field15.offset) = 0);
    ((cuda_rec_field15.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i55 = 0;
    int64_t n23 = (sizeof(float));
    while ((i55 < (cuda_rec_field15.rank))) {
      (((cuda_rec_field15.dims)[i55]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 0))->dim)[i55]));
      (n23 = (n23 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 0))->dim)[i55])));
      (i55 = (i55 + 1));
    }
    ((cuda_rec_field15.size) = n23);
    float (*t326);
    hipMallocManaged((&t326), n23);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml3) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 0)));
    int64_t i56 = 0;
    int64_t elems3 = (n23 / (sizeof(float)));
    while ((i56 < elems3)) {
      ((t326[i56]) = (( float ) (t_ocaml3[i56])));
      (i56 = (i56 + 1));
    }
    ((cuda_rec_field15.data) = t326);
    ((cuda_seq_temp.b) = cuda_rec_field15);
    Tensor cuda_rec_field16;
    ((cuda_rec_field16.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 1))->num_dims));
    if (((cuda_rec_field16.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field16.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field16.offset) = 0);
    ((cuda_rec_field16.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i57 = 0;
    int64_t n24 = (sizeof(float));
    while ((i57 < (cuda_rec_field16.rank))) {
      (((cuda_rec_field16.dims)[i57]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 1))->dim)[i57]));
      (n24 = (n24 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 1))->dim)[i57])));
      (i57 = (i57 + 1));
    }
    ((cuda_rec_field16.size) = n24);
    float (*t327);
    hipMallocManaged((&t327), n24);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml4) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 1)));
    int64_t i58 = 0;
    int64_t elems4 = (n24 / (sizeof(float)));
    while ((i58 < elems4)) {
      ((t327[i58]) = (( float ) (t_ocaml4[i58])));
      (i58 = (i58 + 1));
    }
    ((cuda_rec_field16.data) = t327);
    ((cuda_seq_temp.w) = cuda_rec_field16);
    int64_t cuda_rec_field17 = Long_val(Field(Field(Field(network6, 1), i54), 2));
    ((cuda_seq_temp.ty) = cuda_rec_field17);
    Tensor cuda_rec_field18;
    ((cuda_rec_field18.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 3))->num_dims));
    if (((cuda_rec_field18.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field18.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field18.offset) = 0);
    ((cuda_rec_field18.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i59 = 0;
    int64_t n25 = (sizeof(float));
    while ((i59 < (cuda_rec_field18.rank))) {
      (((cuda_rec_field18.dims)[i59]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 3))->dim)[i59]));
      (n25 = (n25 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 3))->dim)[i59])));
      (i59 = (i59 + 1));
    }
    ((cuda_rec_field18.size) = n25);
    float (*t328);
    hipMallocManaged((&t328), n25);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml5) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 3)));
    int64_t i60 = 0;
    int64_t elems5 = (n25 / (sizeof(float)));
    while ((i60 < elems5)) {
      ((t328[i60]) = (( float ) (t_ocaml5[i60])));
      (i60 = (i60 + 1));
    }
    ((cuda_rec_field18.data) = t328);
    ((cuda_seq_temp.b_grads) = cuda_rec_field18);
    Tensor cuda_rec_field19;
    ((cuda_rec_field19.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 4))->num_dims));
    if (((cuda_rec_field19.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field19.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field19.offset) = 0);
    ((cuda_rec_field19.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i61 = 0;
    int64_t n26 = (sizeof(float));
    while ((i61 < (cuda_rec_field19.rank))) {
      (((cuda_rec_field19.dims)[i61]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 4))->dim)[i61]));
      (n26 = (n26 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 4))->dim)[i61])));
      (i61 = (i61 + 1));
    }
    ((cuda_rec_field19.size) = n26);
    float (*t329);
    hipMallocManaged((&t329), n26);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml6) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 4)));
    int64_t i62 = 0;
    int64_t elems6 = (n26 / (sizeof(float)));
    while ((i62 < elems6)) {
      ((t329[i62]) = (( float ) (t_ocaml6[i62])));
      (i62 = (i62 + 1));
    }
    ((cuda_rec_field19.data) = t329);
    ((cuda_seq_temp.w_grads) = cuda_rec_field19);
    Tensor cuda_rec_field20;
    ((cuda_rec_field20.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 5))->num_dims));
    if (((cuda_rec_field20.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field20.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field20.offset) = 0);
    ((cuda_rec_field20.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i63 = 0;
    int64_t n27 = (sizeof(float));
    while ((i63 < (cuda_rec_field20.rank))) {
      (((cuda_rec_field20.dims)[i63]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 5))->dim)[i63]));
      (n27 = (n27 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 5))->dim)[i63])));
      (i63 = (i63 + 1));
    }
    ((cuda_rec_field20.size) = n27);
    float (*t330);
    hipMallocManaged((&t330), n27);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml7) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 5)));
    int64_t i64 = 0;
    int64_t elems7 = (n27 / (sizeof(float)));
    while ((i64 < elems7)) {
      ((t330[i64]) = (( float ) (t_ocaml7[i64])));
      (i64 = (i64 + 1));
    }
    ((cuda_rec_field20.data) = t330);
    ((cuda_seq_temp.in_grads) = cuda_rec_field20);
    Tensor cuda_rec_field21;
    ((cuda_rec_field21.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 6))->num_dims));
    if (((cuda_rec_field21.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field21.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field21.offset) = 0);
    ((cuda_rec_field21.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i65 = 0;
    int64_t n28 = (sizeof(float));
    while ((i65 < (cuda_rec_field21.rank))) {
      (((cuda_rec_field21.dims)[i65]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 6))->dim)[i65]));
      (n28 = (n28 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 6))->dim)[i65])));
      (i65 = (i65 + 1));
    }
    ((cuda_rec_field21.size) = n28);
    float (*t331);
    hipMallocManaged((&t331), n28);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml8) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 6)));
    int64_t i66 = 0;
    int64_t elems8 = (n28 / (sizeof(float)));
    while ((i66 < elems8)) {
      ((t331[i66]) = (( float ) (t_ocaml8[i66])));
      (i66 = (i66 + 1));
    }
    ((cuda_rec_field21.data) = t331);
    ((cuda_seq_temp.out_bufs) = cuda_rec_field21);
    Tensor cuda_rec_field22;
    ((cuda_rec_field22.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 7))->num_dims));
    if (((cuda_rec_field22.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field22.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field22.offset) = 0);
    ((cuda_rec_field22.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i67 = 0;
    int64_t n29 = (sizeof(float));
    while ((i67 < (cuda_rec_field22.rank))) {
      (((cuda_rec_field22.dims)[i67]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 7))->dim)[i67]));
      (n29 = (n29 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 7))->dim)[i67])));
      (i67 = (i67 + 1));
    }
    ((cuda_rec_field22.size) = n29);
    float (*t332);
    hipMallocManaged((&t332), n29);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml9) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 7)));
    int64_t i68 = 0;
    int64_t elems9 = (n29 / (sizeof(float)));
    while ((i68 < elems9)) {
      ((t332[i68]) = (( float ) (t_ocaml9[i68])));
      (i68 = (i68 + 1));
    }
    ((cuda_rec_field22.data) = t332);
    ((cuda_seq_temp.softmax_bufs) = cuda_rec_field22);
    (((cuda_rec_field14.seq)[i54]) = cuda_seq_temp);
    (i54 = (i54 + 1));
  }
  ((gpu_tmp6.components) = cuda_rec_field14);
  int64_t gpu_tmp7 = Long_val(rounds);
  Seq2 gpu_tmp8;
  ((gpu_tmp8.len) = Wosize_val(training_batches));
  hipMallocManaged((&(gpu_tmp8.seq)), (Wosize_val(training_batches) * (sizeof(Rec3))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i69 = 0;
  while ((i69 < (gpu_tmp8.len))) {
    Rec3 cuda_seq_temp1;
    Tensor cuda_rec_field23;
    ((cuda_rec_field23.rank) = (Caml_ba_array_val(Field(Field(training_batches, i69), 0))->num_dims));
    if (((cuda_rec_field23.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field23.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field23.offset) = 0);
    ((cuda_rec_field23.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i70 = 0;
    int64_t n30 = (sizeof(float));
    while ((i70 < (cuda_rec_field23.rank))) {
      (((cuda_rec_field23.dims)[i70]) = ((Caml_ba_array_val(Field(Field(training_batches, i69), 0))->dim)[i70]));
      (n30 = (n30 * ((Caml_ba_array_val(Field(Field(training_batches, i69), 0))->dim)[i70])));
      (i70 = (i70 + 1));
    }
    ((cuda_rec_field23.size) = n30);
    float (*t333);
    hipMallocManaged((&t333), n30);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml10) = (( double (*) ) Caml_ba_data_val(Field(Field(training_batches, i69), 0)));
    int64_t i71 = 0;
    int64_t elems10 = (n30 / (sizeof(float)));
    while ((i71 < elems10)) {
      ((t333[i71]) = (( float ) (t_ocaml10[i71])));
      (i71 = (i71 + 1));
    }
    ((cuda_rec_field23.data) = t333);
    ((cuda_seq_temp1.inputs) = cuda_rec_field23);
    Tensor1 cuda_rec_field24;
    ((cuda_rec_field24.rank) = (Caml_ba_array_val(Field(Field(training_batches, i69), 1))->num_dims));
    if (((cuda_rec_field24.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field24.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field24.offset) = 0);
    ((cuda_rec_field24.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i72 = 0;
    int64_t n31 = (sizeof(int64_t));
    while ((i72 < (cuda_rec_field24.rank))) {
      (((cuda_rec_field24.dims)[i72]) = ((Caml_ba_array_val(Field(Field(training_batches, i69), 1))->dim)[i72]));
      (n31 = (n31 * ((Caml_ba_array_val(Field(Field(training_batches, i69), 1))->dim)[i72])));
      (i72 = (i72 + 1));
    }
    ((cuda_rec_field24.size) = n31);
    int64_t (*t334);
    hipMallocManaged((&t334), n31);
    GPU_UTILS_CHECK_CUDA_ERROR();
    int64_t (*t_ocaml11) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(training_batches, i69), 1)));
    int64_t i73 = 0;
    int64_t elems11 = (n31 / (sizeof(int64_t)));
    while ((i73 < elems11)) {
      ((t334[i73]) = (( int64_t ) (t_ocaml11[i73])));
      (i73 = (i73 + 1));
    }
    ((cuda_rec_field24.data) = t334);
    ((cuda_seq_temp1.correct_linear_outidxs) = cuda_rec_field24);
    (((gpu_tmp8.seq)[i69]) = cuda_seq_temp1);
    (i69 = (i69 + 1));
  }
  Seq2 gpu_tmp9;
  ((gpu_tmp9.len) = Wosize_val(validation_batches));
  hipMallocManaged((&(gpu_tmp9.seq)), (Wosize_val(validation_batches) * (sizeof(Rec3))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i74 = 0;
  while ((i74 < (gpu_tmp9.len))) {
    Rec3 cuda_seq_temp2;
    Tensor cuda_rec_field25;
    ((cuda_rec_field25.rank) = (Caml_ba_array_val(Field(Field(validation_batches, i74), 0))->num_dims));
    if (((cuda_rec_field25.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field25.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field25.offset) = 0);
    ((cuda_rec_field25.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i75 = 0;
    int64_t n32 = (sizeof(float));
    while ((i75 < (cuda_rec_field25.rank))) {
      (((cuda_rec_field25.dims)[i75]) = ((Caml_ba_array_val(Field(Field(validation_batches, i74), 0))->dim)[i75]));
      (n32 = (n32 * ((Caml_ba_array_val(Field(Field(validation_batches, i74), 0))->dim)[i75])));
      (i75 = (i75 + 1));
    }
    ((cuda_rec_field25.size) = n32);
    float (*t335);
    hipMallocManaged((&t335), n32);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml12) = (( double (*) ) Caml_ba_data_val(Field(Field(validation_batches, i74), 0)));
    int64_t i76 = 0;
    int64_t elems12 = (n32 / (sizeof(float)));
    while ((i76 < elems12)) {
      ((t335[i76]) = (( float ) (t_ocaml12[i76])));
      (i76 = (i76 + 1));
    }
    ((cuda_rec_field25.data) = t335);
    ((cuda_seq_temp2.inputs) = cuda_rec_field25);
    Tensor1 cuda_rec_field26;
    ((cuda_rec_field26.rank) = (Caml_ba_array_val(Field(Field(validation_batches, i74), 1))->num_dims));
    if (((cuda_rec_field26.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field26.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field26.offset) = 0);
    ((cuda_rec_field26.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i77 = 0;
    int64_t n33 = (sizeof(int64_t));
    while ((i77 < (cuda_rec_field26.rank))) {
      (((cuda_rec_field26.dims)[i77]) = ((Caml_ba_array_val(Field(Field(validation_batches, i74), 1))->dim)[i77]));
      (n33 = (n33 * ((Caml_ba_array_val(Field(Field(validation_batches, i74), 1))->dim)[i77])));
      (i77 = (i77 + 1));
    }
    ((cuda_rec_field26.size) = n33);
    int64_t (*t336);
    hipMallocManaged((&t336), n33);
    GPU_UTILS_CHECK_CUDA_ERROR();
    int64_t (*t_ocaml13) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(validation_batches, i74), 1)));
    int64_t i78 = 0;
    int64_t elems13 = (n33 / (sizeof(int64_t)));
    while ((i78 < elems13)) {
      ((t336[i78]) = (( int64_t ) (t_ocaml13[i78])));
      (i78 = (i78 + 1));
    }
    ((cuda_rec_field26.data) = t336);
    ((cuda_seq_temp2.correct_linear_outidxs) = cuda_rec_field26);
    (((gpu_tmp9.seq)[i74]) = cuda_seq_temp2);
    (i74 = (i74 + 1));
  }
  hipMallocManaged((&t_state), (tensor_count * (sizeof(enum tensor_state))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  ((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) = STATE_OK);
  ((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) = STATE_OK);
  ((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) = STATE_OK);
  int64_t i79 = 0;
  while ((i79 < ((gpu_tmp6.components).len))) {
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).b).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).w).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).b_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).w_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).in_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).out_bufs).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).softmax_bufs).id)]) = STATE_OK);
    (i79 = (i79 + 1));
  }
  int64_t i80 = 0;
  while ((i80 < (gpu_tmp8.len))) {
    ((t_state[((((gpu_tmp8.seq)[i80]).inputs).id)]) = STATE_OK);
    ((t_state[((((gpu_tmp8.seq)[i80]).correct_linear_outidxs).id)]) = STATE_OK);
    (i80 = (i80 + 1));
  }
  int64_t i81 = 0;
  while ((i81 < (gpu_tmp9.len))) {
    ((t_state[((((gpu_tmp9.seq)[i81]).inputs).id)]) = STATE_OK);
    ((t_state[((((gpu_tmp9.seq)[i81]).correct_linear_outidxs).id)]) = STATE_OK);
    (i81 = (i81 + 1));
  }
  cuda_wrap(gpu_tmp, gpu_tmp1, gpu_tmp2, gpu_tmp3, gpu_tmp4, gpu_tmp5, gpu_tmp6, gpu_tmp7, gpu_tmp8, gpu_tmp9);
  if (((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) != STATE_OK)) {
    double (*t_ocaml14) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 1)));
    int64_t i82 = 0;
    int64_t elems14 = ((((gpu_tmp6.lossfn).in_grads).size) / (sizeof(float)));
    while ((i82 < elems14)) {
      ((t_ocaml14[i82]) = (( float ) ((((gpu_tmp6.lossfn).in_grads).data)[i82])));
      (i82 = (i82 + 1));
    }
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).in_grads).data));
    GPU_UTILS_CHECK_CUDA_ERROR();
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) != STATE_OK)) {
    double (*t_ocaml15) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 2)));
    int64_t i83 = 0;
    int64_t elems15 = ((((gpu_tmp6.lossfn).out_bufs).size) / (sizeof(float)));
    while ((i83 < elems15)) {
      ((t_ocaml15[i83]) = (( float ) ((((gpu_tmp6.lossfn).out_bufs).data)[i83])));
      (i83 = (i83 + 1));
    }
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).out_bufs).data));
    GPU_UTILS_CHECK_CUDA_ERROR();
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) != STATE_OK)) {
    double (*t_ocaml16) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 3)));
    int64_t i84 = 0;
    int64_t elems16 = ((((gpu_tmp6.lossfn).softmax_bufs).size) / (sizeof(float)));
    while ((i84 < elems16)) {
      ((t_ocaml16[i84]) = (( float ) ((((gpu_tmp6.lossfn).softmax_bufs).data)[i84])));
      (i84 = (i84 + 1));
    }
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).softmax_bufs).data));
    GPU_UTILS_CHECK_CUDA_ERROR();
  } else {
    
  }
  int64_t i85 = 0;
  while ((i85 < ((gpu_tmp6.components).len))) {
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b).id)]) != STATE_OK)) {
      double (*t_ocaml17) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 0)));
      int64_t i86 = 0;
      int64_t elems17 = ((((((gpu_tmp6.components).seq)[i85]).b).size) / (sizeof(float)));
      while ((i86 < elems17)) {
        ((t_ocaml17[i86]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).b).data)[i86])));
        (i86 = (i86 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).b).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w).id)]) != STATE_OK)) {
      double (*t_ocaml18) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 1)));
      int64_t i87 = 0;
      int64_t elems18 = ((((((gpu_tmp6.components).seq)[i85]).w).size) / (sizeof(float)));
      while ((i87 < elems18)) {
        ((t_ocaml18[i87]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).w).data)[i87])));
        (i87 = (i87 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).w).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b_grads).id)]) != STATE_OK)) {
      double (*t_ocaml19) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 3)));
      int64_t i88 = 0;
      int64_t elems19 = ((((((gpu_tmp6.components).seq)[i85]).b_grads).size) / (sizeof(float)));
      while ((i88 < elems19)) {
        ((t_ocaml19[i88]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).b_grads).data)[i88])));
        (i88 = (i88 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).b_grads).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w_grads).id)]) != STATE_OK)) {
      double (*t_ocaml20) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 4)));
      int64_t i89 = 0;
      int64_t elems20 = ((((((gpu_tmp6.components).seq)[i85]).w_grads).size) / (sizeof(float)));
      while ((i89 < elems20)) {
        ((t_ocaml20[i89]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).w_grads).data)[i89])));
        (i89 = (i89 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).w_grads).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).in_grads).id)]) != STATE_OK)) {
      double (*t_ocaml21) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 5)));
      int64_t i90 = 0;
      int64_t elems21 = ((((((gpu_tmp6.components).seq)[i85]).in_grads).size) / (sizeof(float)));
      while ((i90 < elems21)) {
        ((t_ocaml21[i90]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).in_grads).data)[i90])));
        (i90 = (i90 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).in_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).in_grads).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).out_bufs).id)]) != STATE_OK)) {
      double (*t_ocaml22) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 6)));
      int64_t i91 = 0;
      int64_t elems22 = ((((((gpu_tmp6.components).seq)[i85]).out_bufs).size) / (sizeof(float)));
      while ((i91 < elems22)) {
        ((t_ocaml22[i91]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).out_bufs).data)[i91])));
        (i91 = (i91 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).out_bufs).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).out_bufs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).softmax_bufs).id)]) != STATE_OK)) {
      double (*t_ocaml23) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 7)));
      int64_t i92 = 0;
      int64_t elems23 = ((((((gpu_tmp6.components).seq)[i85]).softmax_bufs).size) / (sizeof(float)));
      while ((i92 < elems23)) {
        ((t_ocaml23[i92]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).softmax_bufs).data)[i92])));
        (i92 = (i92 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).softmax_bufs).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).softmax_bufs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    (i85 = (i85 + 1));
  }
  hipFree(((gpu_tmp6.components).seq));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i93 = 0;
  while ((i93 < (gpu_tmp8.len))) {
    if (((t_state[((((gpu_tmp8.seq)[i93]).inputs).id)]) != STATE_OK)) {
      double (*t_ocaml24) = (( double (*) ) Caml_ba_data_val(Field(Field(training_batches, i93), 0)));
      int64_t i94 = 0;
      int64_t elems24 = (((((gpu_tmp8.seq)[i93]).inputs).size) / (sizeof(float)));
      while ((i94 < elems24)) {
        ((t_ocaml24[i94]) = (( float ) (((((gpu_tmp8.seq)[i93]).inputs).data)[i94])));
        (i94 = (i94 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i93]).inputs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp8.seq)[i93]).inputs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).id)]) != STATE_OK)) {
      int64_t (*t_ocaml25) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(training_batches, i93), 1)));
      int64_t i95 = 0;
      int64_t elems25 = (((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).size) / (sizeof(int64_t)));
      while ((i95 < elems25)) {
        ((t_ocaml25[i95]) = (( int64_t ) (((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).data)[i95])));
        (i95 = (i95 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    (i93 = (i93 + 1));
  }
  hipFree((gpu_tmp8.seq));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i96 = 0;
  while ((i96 < (gpu_tmp9.len))) {
    if (((t_state[((((gpu_tmp9.seq)[i96]).inputs).id)]) != STATE_OK)) {
      double (*t_ocaml26) = (( double (*) ) Caml_ba_data_val(Field(Field(validation_batches, i96), 0)));
      int64_t i97 = 0;
      int64_t elems26 = (((((gpu_tmp9.seq)[i96]).inputs).size) / (sizeof(float)));
      while ((i97 < elems26)) {
        ((t_ocaml26[i97]) = (( float ) (((((gpu_tmp9.seq)[i96]).inputs).data)[i97])));
        (i97 = (i97 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i96]).inputs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp9.seq)[i96]).inputs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).id)]) != STATE_OK)) {
      int64_t (*t_ocaml27) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(validation_batches, i96), 1)));
      int64_t i98 = 0;
      int64_t elems27 = (((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).size) / (sizeof(int64_t)));
      while ((i98 < elems27)) {
        ((t_ocaml27[i98]) = (( int64_t ) (((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).data)[i98])));
        (i98 = (i98 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    (i96 = (i96 + 1));
  }
  hipFree((gpu_tmp9.seq));
  GPU_UTILS_CHECK_CUDA_ERROR();
  hipFree(t_state);
  GPU_UTILS_CHECK_CUDA_ERROR();
  hipblasDestroy(_cublas_handle);
  GPU_UTILS_CHECK_CUDA_ERROR();
  CAMLreturn0;
}
extern "C" void vAiCRgHg0yi(value (*args), int argc) {
  return vv4F90aVQU5((args[0]), (args[1]), (args[2]), (args[3]), (args[4]), (args[5]), (args[6]), (args[7]), (args[8]), (args[9]));
}