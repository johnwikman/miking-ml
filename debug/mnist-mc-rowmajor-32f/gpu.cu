#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <math.h>
#include <stddef.h>
#include <stdlib.h>
#include <stdio.h>
#include "caml/alloc.h"
#include "caml/bigarray.h"
#include "caml/memory.h"
#include "caml/mlvalues.h"
#include "gpu-utils.cu"
enum tensor_state {STATE_OK, STATE_CPU_INVALID, STATE_GPU_INVALID, STATE_RETURNED};
__managed__ enum tensor_state (*t_state);
typedef struct Tensor {int64_t id; float (*data); int64_t dims[3]; int64_t rank; int64_t offset; int64_t size;} Tensor;
typedef struct Seq {int64_t (*seq); int64_t len;} Seq;
typedef struct Tensor1 {int64_t id; int64_t (*data); int64_t dims[3]; int64_t rank; int64_t offset; int64_t size;} Tensor1;
typedef struct Rec {Tensor b; Tensor w; int64_t ty; Tensor b_grads; Tensor w_grads; Tensor in_grads; Tensor out_bufs; Tensor softmax_bufs;} Rec;
typedef struct Rec1 {int64_t ty; Tensor in_grads; Tensor out_bufs; Tensor softmax_bufs;} Rec1;
typedef struct Seq1 {Rec (*seq); int64_t len;} Seq1;
typedef struct Rec2 {Rec1 lossfn; Seq1 components;} Rec2;
typedef struct Rec3 {Tensor inputs; Tensor1 correct_linear_outidxs;} Rec3;
typedef struct Seq2 {Rec3 (*seq); int64_t len;} Seq2;
typedef struct Rec4 {int64_t epochs; int64_t batchsize; float init_alpha; float decay_alpha; float init_lambda; char printStatus; float decay_lambda; char evaluateBetweenEpochs; char evaluateBeforeFirstEpoch;} Rec4;
typedef struct Rec5 {float _0; float _1;} Rec5;
__host__ __device__ int64_t cartesian_to_linear_index0(int64_t dims1[3], int64_t rank1) {
  {
    int64_t t;
    (t = 0);
    return t;
  }
}
__host__ __device__ int64_t cartesian_to_linear_index1(int64_t dims1[3], int64_t rank1, int64_t i) {
  if ((rank1 == 3)) {
    {
      int64_t t1;
      (t1 = (((dims1[2]) * (dims1[1])) * i));
      return t1;
    }
  } else {
    if ((rank1 == 2)) {
      {
        int64_t t2;
        (t2 = ((dims1[1]) * i));
        return t2;
      }
    } else {
      {
        int64_t t3;
        (t3 = i);
        return t3;
      }
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index2(int64_t dims1[3], int64_t rank1, int64_t i1, int64_t i2) {
  if ((rank1 == 3)) {
    {
      int64_t t4;
      (t4 = ((((dims1[2]) * (dims1[1])) * i1) + ((dims1[2]) * i2)));
      return t4;
    }
  } else {
    if ((rank1 == 2)) {
      {
        int64_t t5;
        (t5 = (((dims1[1]) * i1) + i2));
        return t5;
      }
    } else {
      printf("Accessed tensor of rank %ld using 2 indices\n", rank1);
      {
        int64_t t6;
        (t6 = -1);
        return t6;
      }
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index3(int64_t dims1[3], int64_t rank1, int64_t i3, int64_t i4, int64_t i5) {
  if ((rank1 == 3)) {
    {
      int64_t t7;
      (t7 = (((((dims1[2]) * (dims1[1])) * i3) + ((dims1[2]) * i4)) + i5));
      return t7;
    }
  } else {
    printf("Accessed tensor of rank %ld using 3 indices\n", rank1);
    {
      int64_t t8;
      (t8 = -1);
      return t8;
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index(int64_t dims1[3], int64_t rank1, Seq cartesian_idx) {
  if (((cartesian_idx.len) == 1)) {
    {
      int64_t t9;
      (t9 = cartesian_to_linear_index1(dims1, rank1, ((cartesian_idx.seq)[0])));
      return t9;
    }
  } else {
    if (((cartesian_idx.len) == 2)) {
      {
        int64_t t10;
        (t10 = cartesian_to_linear_index2(dims1, rank1, ((cartesian_idx.seq)[0]), ((cartesian_idx.seq)[1])));
        return t10;
      }
    } else {
      if (((cartesian_idx.len) == 3)) {
        {
          int64_t t11;
          (t11 = cartesian_to_linear_index3(dims1, rank1, ((cartesian_idx.seq)[0]), ((cartesian_idx.seq)[1]), ((cartesian_idx.seq)[2])));
          return t11;
        }
      } else {
        {
          int64_t t12;
          (t12 = cartesian_to_linear_index0(dims1, rank1));
          return t12;
        }
      }
    }
  }
}
__host__ __device__ Seq tensor_shape(int64_t dims2[3], int64_t rank2) {
  Seq s;
  ((s.seq) = dims2);
  ((s.len) = rank2);
  {
    Seq t13;
    (t13 = s);
    return t13;
  }
}
extern double exp(double);
__host__ __device__ float exp1(float x) {
  {
    float t14;
    (t14 = exp(x));
    return t14;
  }
}
__host__ __device__ int64_t t15(int64_t acc, int64_t e) {
  {
    int64_t t16;
    (t16 = (acc * e));
    return t16;
  }
}
__host__ __device__ int64_t foldl(int64_t acc_init, Seq s1) {
  int64_t acc1 = acc_init;
  int64_t i6 = 0;
  while ((i6 < (s1.len))) {
    (acc1 = t15(acc1, ((s1.seq)[i6])));
    (i6 = (i6 + 1));
  }
  return acc1;
}
__host__ __device__ int64_t tensorSize(Tensor t17) {
  Seq t18;
  (t18 = tensor_shape((t17.dims), (t17.rank)));
  {
    int64_t t19;
    (t19 = foldl(1, t18));
    return t19;
  }
}
__host__ __device__ float t20(Tensor w1, Tensor x1, int64_t n, int64_t x_offset, int64_t i7, float acc2, int64_t j) {
  int64_t t21;
  (t21 = (n * i7));
  int64_t t22;
  (t22 = (t21 + j));
  float t23;
  (t23 = ((w1.data)[(t22 + (w1.offset))]));
  int64_t t24;
  (t24 = (x_offset + j));
  float t25;
  (t25 = ((x1.data)[(t24 + (x1.offset))]));
  float t26;
  (t26 = (t23 * t25));
  {
    float t27;
    (t27 = (acc2 + t26));
    return t27;
  }
}
__host__ __device__ void iterfun(Tensor w1, Tensor x1, Tensor b1, Tensor z, int64_t m, int64_t n, int64_t i8) {
  int64_t s_idx;
  (s_idx = (i8 / m));
  int64_t x_offset;
  (x_offset = (s_idx * n));
  int64_t z_idx;
  (z_idx = i8);
  int64_t i7;
  (i7 = (i8 % m));
  float acc_init1;
  (acc_init1 = ((b1.data)[(i7 + (b1.offset))]));
  float v;
  {
    int64_t i9 = 0;
    float acc3 = acc_init1;
    while ((i9 < n)) {
      (acc3 = t20(w1, x1, n, x_offset, i7, acc3, i9));
      (i9 = (i9 + 1));
    }
    (v = acc3);
  }
  (((z.data)[(z_idx + (z.offset))]) = v);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel(int64_t n1, Tensor w1, Tensor x1, Tensor b1, Tensor z, int64_t m, int64_t n) {
  int64_t idx = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride = (gridDim.x * blockDim.x);
  while ((idx < n1)) {
    iterfun(w1, x1, b1, z, m, n, idx);
    (idx = (idx + stride));
  }
}
__host__ void tensorOpExn__z___Wx_B(int64_t s_max, Tensor w1, Tensor x1, Tensor b1, Tensor z) {
  Seq w_shape;
  (w_shape = tensor_shape((w1.dims), (w1.rank)));
  int64_t m;
  (m = ((w_shape.seq)[0]));
  int64_t n;
  (n = ((w_shape.seq)[1]));
  int64_t t28;
  (t28 = (s_max * m));
  {
    int64_t niterations = t28;
    int64_t tpb = 256;
    int64_t nblocks = (((niterations + (tpb * 10)) - 1) / (tpb * 10));
    loopKernel<<<nblocks, tpb>>>(t28, w1, x1, b1, z, m, n);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ int64_t t29(Tensor x2, Tensor z1, int64_t n2, int64_t x_offset1, float y_val, int64_t z_idx1, int64_t row) {
  int64_t t30;
  (t30 = (x_offset1 + row));
  float t31;
  (t31 = ((x2.data)[(t30 + (x2.offset))]));
  float t32;
  (t32 = (y_val * t31));
  (((z1.data)[(z_idx1 + (z1.offset))]) = t32);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z1.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z1.id)]) = STATE_GPU_INVALID);
  #endif
  {
    int64_t t33;
    (t33 = (z_idx1 + n2));
    return t33;
  }
}
__host__ __device__ void iterfun1(Tensor x2, Tensor y, Tensor z1, int64_t m1, int64_t n2, int64_t m_x_n, int64_t i10) {
  int64_t s_idx1;
  (s_idx1 = (i10 / n2));
  int64_t col;
  (col = (i10 % n2));
  int64_t t34;
  (t34 = (s_idx1 * m_x_n));
  int64_t z_offset;
  (z_offset = (t34 + col));
  int64_t x_offset1;
  (x_offset1 = (s_idx1 * m1));
  int64_t y_offset;
  (y_offset = (s_idx1 * n2));
  int64_t t35;
  (t35 = (y_offset + col));
  float y_val;
  (y_val = ((y.data)[(t35 + (y.offset))]));
  int64_t _;
  {
    int64_t i11 = 0;
    int64_t acc4 = z_offset;
    while ((i11 < m1)) {
      (acc4 = t29(x2, z1, n2, x_offset1, y_val, acc4, i11));
      (i11 = (i11 + 1));
    }
    (_ = acc4);
  }
  ;
}
__global__ void loopKernel1(int64_t n3, Tensor x2, Tensor y, Tensor z1, int64_t m1, int64_t n2, int64_t m_x_n) {
  int64_t idx1 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride1 = (gridDim.x * blockDim.x);
  while ((idx1 < n3)) {
    iterfun1(x2, y, z1, m1, n2, m_x_n, idx1);
    (idx1 = (idx1 + stride1));
  }
}
__host__ void tensorOpExn__z___x___y_T(int64_t s_max1, Tensor x2, Tensor y, Tensor z1) {
  Seq z_shape;
  (z_shape = tensor_shape((z1.dims), (z1.rank)));
  int64_t m1;
  (m1 = ((z_shape.seq)[1]));
  int64_t n2;
  (n2 = ((z_shape.seq)[2]));
  int64_t m_x_n;
  (m_x_n = (m1 * n2));
  int64_t t36;
  (t36 = (s_max1 * n2));
  {
    int64_t niterations1 = t36;
    int64_t tpb1 = 256;
    int64_t nblocks1 = (((niterations1 + (tpb1 * 10)) - 1) / (tpb1 * 10));
    loopKernel1<<<nblocks1, tpb1>>>(t36, x2, y, z1, m1, n2, m_x_n);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ float t37(Tensor x3, Tensor w2, int64_t n4, int64_t x_offset2, int64_t j1, float acc5, int64_t i12) {
  int64_t t38;
  (t38 = (n4 * i12));
  int64_t t39;
  (t39 = (t38 + j1));
  float t40;
  (t40 = ((w2.data)[(t39 + (w2.offset))]));
  int64_t t41;
  (t41 = (x_offset2 + i12));
  float t42;
  (t42 = ((x3.data)[(t41 + (x3.offset))]));
  float t43;
  (t43 = (t40 * t42));
  {
    float t44;
    (t44 = (acc5 + t43));
    return t44;
  }
}
__host__ __device__ void iterfun2(Tensor x3, Tensor w2, Tensor z2, int64_t m2, int64_t n4, int64_t j2) {
  int64_t s_idx2;
  (s_idx2 = (j2 / n4));
  int64_t n_idx;
  (n_idx = (j2 % n4));
  int64_t z_idx2;
  (z_idx2 = j2);
  int64_t x_offset2;
  (x_offset2 = (s_idx2 * m2));
  int64_t j1;
  (j1 = n_idx);
  float v1;
  {
    int64_t i13 = 0;
    float acc6 = 0.;
    while ((i13 < m2)) {
      (acc6 = t37(x3, w2, n4, x_offset2, j1, acc6, i13));
      (i13 = (i13 + 1));
    }
    (v1 = acc6);
  }
  (((z2.data)[(z_idx2 + (z2.offset))]) = v1);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z2.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z2.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel2(int64_t n5, Tensor x3, Tensor w2, Tensor z2, int64_t m2, int64_t n4) {
  int64_t idx2 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride2 = (gridDim.x * blockDim.x);
  while ((idx2 < n5)) {
    iterfun2(x3, w2, z2, m2, n4, idx2);
    (idx2 = (idx2 + stride2));
  }
}
__host__ void tensorOpExn__z____x_T___W__T(int64_t s_max2, Tensor x3, Tensor w2, Tensor z2) {
  Seq w_shape1;
  (w_shape1 = tensor_shape((w2.dims), (w2.rank)));
  int64_t m2;
  (m2 = ((w_shape1.seq)[0]));
  int64_t n4;
  (n4 = ((w_shape1.seq)[1]));
  int64_t t45;
  (t45 = (s_max2 * n4));
  {
    int64_t niterations2 = t45;
    int64_t tpb2 = 256;
    int64_t nblocks2 = (((niterations2 + (tpb2 * 10)) - 1) / (tpb2 * 10));
    loopKernel2<<<nblocks2, tpb2>>>(t45, x3, w2, z2, m2, n4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun3(Tensor x4, Tensor z3, int64_t i14) {
  float x_i;
  (x_i = ((x4.data)[(i14 + (x4.offset))]));
  char t46;
  (t46 = (x_i > 0.));
  float t47;
  if ((t46 == 1)) {
    (t47 = x_i);
  } else {
    (t47 = 0.);
  }
  (((z3.data)[(i14 + (z3.offset))]) = t47);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z3.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z3.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel3(int64_t n6, Tensor x4, Tensor z3) {
  int64_t idx3 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride3 = (gridDim.x * blockDim.x);
  while ((idx3 < n6)) {
    iterfun3(x4, z3, idx3);
    (idx3 = (idx3 + stride3));
  }
}
__host__ void tensorOpExn__z___ReLU_x_(int64_t s_max3, Tensor x4, Tensor z3) {
  Seq t48;
  (t48 = tensor_shape((x4.dims), (x4.rank)));
  int64_t s2;
  (s2 = ((t48.seq)[0]));
  int64_t t49;
  (t49 = tensorSize(x4));
  int64_t m3;
  (m3 = (t49 / s2));
  int64_t t50;
  (t50 = (s_max3 * m3));
  {
    int64_t niterations3 = t50;
    int64_t tpb3 = 256;
    int64_t nblocks3 = (((niterations3 + (tpb3 * 10)) - 1) / (tpb3 * 10));
    loopKernel3<<<nblocks3, tpb3>>>(t50, x4, z3);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun4(Tensor x5, Tensor z4, int64_t i15) {
  float x_i1;
  (x_i1 = ((x5.data)[(i15 + (x5.offset))]));
  float t51;
  (t51 = exp1(x_i1));
  (((z4.data)[(i15 + (z4.offset))]) = t51);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z4.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z4.id)]) = STATE_GPU_INVALID);
  #endif
}
__host__ __device__ float t52(Tensor z4, int64_t offset1, float acc7, int64_t i16) {
  int64_t t53;
  (t53 = (offset1 + i16));
  float t54;
  (t54 = ((z4.data)[(t53 + (z4.offset))]));
  {
    float t55;
    (t55 = (acc7 + t54));
    return t55;
  }
}
__host__ __device__ void iterfunSummarize(Tensor expsumbuf, Tensor z4, int64_t m4, int64_t s_idx3) {
  int64_t offset1;
  (offset1 = (s_idx3 * m4));
  float expsum;
  {
    int64_t i17 = 0;
    float acc8 = 0.;
    while ((i17 < m4)) {
      (acc8 = t52(z4, offset1, acc8, i17));
      (i17 = (i17 + 1));
    }
    (expsum = acc8);
  }
  (((expsumbuf.data)[(s_idx3 + (expsumbuf.offset))]) = expsum);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(expsumbuf.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(expsumbuf.id)]) = STATE_GPU_INVALID);
  #endif
}
__host__ __device__ void iterfunRegularize(Tensor expsumbuf, Tensor z4, int64_t m4, int64_t i18) {
  int64_t s_idx4;
  (s_idx4 = (i18 / m4));
  float expsum1;
  (expsum1 = ((expsumbuf.data)[(s_idx4 + (expsumbuf.offset))]));
  float z_i;
  (z_i = ((z4.data)[(i18 + (z4.offset))]));
  float t56;
  (t56 = (z_i / expsum1));
  (((z4.data)[(i18 + (z4.offset))]) = t56);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z4.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z4.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel4(int64_t n7, Tensor expsumbuf, Tensor z4, int64_t m4) {
  int64_t idx4 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride4 = (gridDim.x * blockDim.x);
  while ((idx4 < n7)) {
    iterfunRegularize(expsumbuf, z4, m4, idx4);
    (idx4 = (idx4 + stride4));
  }
}
__global__ void loopKernel5(int64_t n8, Tensor expsumbuf, Tensor z4, int64_t m4) {
  int64_t idx5 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride5 = (gridDim.x * blockDim.x);
  while ((idx5 < n8)) {
    iterfunSummarize(expsumbuf, z4, m4, idx5);
    (idx5 = (idx5 + stride5));
  }
}
__global__ void loopKernel6(int64_t n9, Tensor x5, Tensor z4) {
  int64_t idx6 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride6 = (gridDim.x * blockDim.x);
  while ((idx6 < n9)) {
    iterfun4(x5, z4, idx6);
    (idx6 = (idx6 + stride6));
  }
}
__host__ void tensorOpExn__z___SoftMax_x_(int64_t s_max4, Tensor x5, Tensor expsumbuf, Tensor z4) {
  Seq t57;
  (t57 = tensor_shape((x5.dims), (x5.rank)));
  int64_t s3;
  (s3 = ((t57.seq)[0]));
  int64_t t58;
  (t58 = tensorSize(x5));
  int64_t m4;
  (m4 = (t58 / s3));
  int64_t t59;
  (t59 = (s_max4 * m4));
  {
    int64_t niterations4 = t59;
    int64_t tpb4 = 256;
    int64_t nblocks4 = (((niterations4 + (tpb4 * 10)) - 1) / (tpb4 * 10));
    loopKernel6<<<nblocks4, tpb4>>>(t59, x5, z4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
  {
    int64_t niterations5 = s_max4;
    int64_t tpb5 = 256;
    int64_t nblocks5 = (((niterations5 + (tpb5 * 10)) - 1) / (tpb5 * 10));
    loopKernel5<<<nblocks5, tpb5>>>(s_max4, expsumbuf, z4, m4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
  int64_t t60;
  (t60 = (s_max4 * m4));
  {
    int64_t niterations6 = t60;
    int64_t tpb6 = 256;
    int64_t nblocks6 = (((niterations6 + (tpb6 * 10)) - 1) / (tpb6 * 10));
    loopKernel4<<<nblocks6, tpb6>>>(t60, expsumbuf, z4, m4);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun5(Tensor h, Tensor dldh, Tensor z5, int64_t i19) {
  float t61;
  (t61 = ((h.data)[(i19 + (h.offset))]));
  char t62;
  (t62 = (t61 > 0.));
  float dhds_ii;
  if ((t62 == 1)) {
    (dhds_ii = 1.);
  } else {
    (dhds_ii = 0.);
  }
  float dldh_i;
  (dldh_i = ((dldh.data)[(i19 + (dldh.offset))]));
  float t63;
  (t63 = (dhds_ii * dldh_i));
  (((z5.data)[(i19 + (z5.offset))]) = t63);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z5.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z5.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel7(int64_t n10, Tensor h, Tensor dldh, Tensor z5) {
  int64_t idx7 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride7 = (gridDim.x * blockDim.x);
  while ((idx7 < n10)) {
    iterfun5(h, dldh, z5, idx7);
    (idx7 = (idx7 + stride7));
  }
}
__host__ void tensorOpExn__z___d_dx_l_ReLU_x__(int64_t s_max5, Tensor h, Tensor dldh, Tensor z5) {
  Seq t64;
  (t64 = tensor_shape((h.dims), (h.rank)));
  int64_t s4;
  (s4 = ((t64.seq)[0]));
  int64_t t65;
  (t65 = tensorSize(h));
  int64_t m5;
  (m5 = (t65 / s4));
  int64_t t66;
  (t66 = (s_max5 * m5));
  {
    int64_t niterations7 = t66;
    int64_t tpb7 = 256;
    int64_t nblocks7 = (((niterations7 + (tpb7 * 10)) - 1) / (tpb7 * 10));
    loopKernel7<<<nblocks7, tpb7>>>(t66, h, dldh, z5);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ float t67(Tensor p, Tensor dldp, int64_t s_offset, int64_t i20, float p_i, float acc9, int64_t j3) {
  char t68;
  (t68 = (j3 == i20));
  float s_ij;
  if ((t68 == 1)) {
    float t69;
    (t69 = (p_i * p_i));
    (s_ij = (p_i - t69));
  } else {
    int64_t t70;
    (t70 = (s_offset + j3));
    float p_j;
    (p_j = ((p.data)[(t70 + (p.offset))]));
    float t71;
    (t71 = (p_i * p_j));
    (s_ij = (-t71));
  }
  int64_t t72;
  (t72 = (s_offset + j3));
  float dldp_j;
  (dldp_j = ((dldp.data)[(t72 + (dldp.offset))]));
  float t73;
  (t73 = (dldp_j * s_ij));
  {
    float t74;
    (t74 = (acc9 + t73));
    return t74;
  }
}
__host__ __device__ void iterfun6(Tensor p, Tensor dldp, Tensor z6, int64_t m6, int64_t i21) {
  int64_t s_idx5;
  (s_idx5 = (i21 / m6));
  int64_t s_offset;
  (s_offset = (s_idx5 * m6));
  int64_t i20;
  (i20 = (i21 % m6));
  int64_t t75;
  (t75 = (s_offset + i20));
  float p_i;
  (p_i = ((p.data)[(t75 + (p.offset))]));
  float v2;
  {
    int64_t i22 = 0;
    float acc10 = 0.;
    while ((i22 < m6)) {
      (acc10 = t67(p, dldp, s_offset, i20, p_i, acc10, i22));
      (i22 = (i22 + 1));
    }
    (v2 = acc10);
  }
  int64_t t76;
  (t76 = (s_offset + i20));
  (((z6.data)[(t76 + (z6.offset))]) = v2);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z6.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z6.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel8(int64_t n11, Tensor p, Tensor dldp, Tensor z6, int64_t m6) {
  int64_t idx8 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride8 = (gridDim.x * blockDim.x);
  while ((idx8 < n11)) {
    iterfun6(p, dldp, z6, m6, idx8);
    (idx8 = (idx8 + stride8));
  }
}
__host__ void tensorOpExn__z___d_dx_l_SoftMax_x___(int64_t s_max6, Tensor p, Tensor dldp, Tensor z6) {
  Seq t77;
  (t77 = tensor_shape((p.dims), (p.rank)));
  int64_t s5;
  (s5 = ((t77.seq)[0]));
  int64_t t78;
  (t78 = tensorSize(p));
  int64_t m6;
  (m6 = (t78 / s5));
  int64_t t79;
  (t79 = (s_max6 * m6));
  {
    int64_t niterations8 = t79;
    int64_t tpb8 = 256;
    int64_t nblocks8 = (((niterations8 + (tpb8 * 10)) - 1) / (tpb8 * 10));
    loopKernel8<<<nblocks8, tpb8>>>(t79, p, dldp, z6, m6);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun7(Tensor x6, Tensor z7, int64_t i23) {
  float t80;
  (t80 = ((x6.data)[(i23 + (x6.offset))]));
  (((z7.data)[(i23 + (z7.offset))]) = t80);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z7.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z7.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel9(int64_t n12, Tensor x6, Tensor z7) {
  int64_t idx9 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride9 = (gridDim.x * blockDim.x);
  while ((idx9 < n12)) {
    iterfun7(x6, z7, idx9);
    (idx9 = (idx9 + stride9));
  }
}
__host__ void tensorOpExn__z___x(int64_t s_max7, Tensor x6, Tensor z7) {
  Seq t81;
  (t81 = tensor_shape((x6.dims), (x6.rank)));
  int64_t s6;
  (s6 = ((t81.seq)[0]));
  int64_t t82;
  (t82 = tensorSize(x6));
  int64_t m7;
  (m7 = (t82 / s6));
  int64_t t83;
  (t83 = (s_max7 * m7));
  {
    int64_t niterations9 = t83;
    int64_t tpb9 = 256;
    int64_t nblocks9 = (((niterations9 + (tpb9 * 10)) - 1) / (tpb9 * 10));
    loopKernel9<<<nblocks9, tpb9>>>(t83, x6, z7);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun8(float c, Tensor z8, int64_t i24) {
  float t84;
  (t84 = ((z8.data)[(i24 + (z8.offset))]));
  float t85;
  (t85 = (t84 * c));
  (((z8.data)[(i24 + (z8.offset))]) = t85);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z8.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z8.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel10(int64_t n13, float c, Tensor z8) {
  int64_t idx10 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride10 = (gridDim.x * blockDim.x);
  while ((idx10 < n13)) {
    iterfun8(c, z8, idx10);
    (idx10 = (idx10 + stride10));
  }
}
__host__ void tensorOpExn__z____scalar_c_(int64_t s_max8, float c, Tensor z8) {
  Seq t86;
  (t86 = tensor_shape((z8.dims), (z8.rank)));
  int64_t s7;
  (s7 = ((t86.seq)[0]));
  int64_t t87;
  (t87 = tensorSize(z8));
  int64_t m8;
  (m8 = (t87 / s7));
  int64_t t88;
  (t88 = (s_max8 * m8));
  {
    int64_t niterations10 = t88;
    int64_t tpb10 = 256;
    int64_t nblocks10 = (((niterations10 + (tpb10 * 10)) - 1) / (tpb10 * 10));
    loopKernel10<<<nblocks10, tpb10>>>(t88, c, z8);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun9(float c1, Tensor z9, int64_t i25) {
  (((z9.data)[(i25 + (z9.offset))]) = c1);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z9.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z9.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel11(int64_t n14, float c1, Tensor z9) {
  int64_t idx11 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride11 = (gridDim.x * blockDim.x);
  while ((idx11 < n14)) {
    iterfun9(c1, z9, idx11);
    (idx11 = (idx11 + stride11));
  }
}
__host__ void tensorOpExn__Z___scalar_c_(float c1, Tensor z9) {
  int64_t m9;
  (m9 = tensorSize(z9));
  {
    int64_t niterations11 = m9;
    int64_t tpb11 = 256;
    int64_t nblocks11 = (((niterations11 + (tpb11 * 10)) - 1) / (tpb11 * 10));
    loopKernel11<<<nblocks11, tpb11>>>(m9, c1, z9);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun10(Tensor x7, float c2, Tensor z10, int64_t x_offset3, int64_t i26) {
  float t89;
  (t89 = ((z10.data)[(i26 + (z10.offset))]));
  int64_t t90;
  (t90 = (i26 + x_offset3));
  float t91;
  (t91 = ((x7.data)[(t90 + (x7.offset))]));
  float t92;
  (t92 = (t91 * c2));
  float t93;
  (t93 = (t89 + t92));
  (((z10.data)[(i26 + (z10.offset))]) = t93);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z10.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z10.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel12(int64_t n15, Tensor x7, float c2, Tensor z10, int64_t x_offset3) {
  int64_t idx12 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride12 = (gridDim.x * blockDim.x);
  while ((idx12 < n15)) {
    iterfun10(x7, c2, z10, x_offset3, idx12);
    (idx12 = (idx12 + stride12));
  }
}
__host__ void tensorOpExn__Z____x___scalar_c_(int64_t s_idx6, Tensor x7, float c2, Tensor z10) {
  int64_t m10;
  (m10 = tensorSize(z10));
  int64_t x_offset3;
  (x_offset3 = (s_idx6 * m10));
  {
    int64_t niterations12 = m10;
    int64_t tpb12 = 256;
    int64_t nblocks12 = (((niterations12 + (tpb12 * 10)) - 1) / (tpb12 * 10));
    loopKernel12<<<nblocks12, tpb12>>>(m10, x7, c2, z10, x_offset3);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun11(Tensor x8, float c3, Tensor z11, int64_t z_offset1, int64_t i27) {
  int64_t z_idx3;
  (z_idx3 = (i27 + z_offset1));
  float t94;
  (t94 = ((z11.data)[(z_idx3 + (z11.offset))]));
  float t95;
  (t95 = ((x8.data)[(i27 + (x8.offset))]));
  float t96;
  (t96 = (t95 * c3));
  float t97;
  (t97 = (t94 + t96));
  (((z11.data)[(z_idx3 + (z11.offset))]) = t97);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z11.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z11.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel13(int64_t n16, Tensor x8, float c3, Tensor z11, int64_t z_offset1) {
  int64_t idx13 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride13 = (gridDim.x * blockDim.x);
  while ((idx13 < n16)) {
    iterfun11(x8, c3, z11, z_offset1, idx13);
    (idx13 = (idx13 + stride13));
  }
}
__host__ void tensorOpExn__z____X___scalar_c_(int64_t s_idx7, Tensor x8, float c3, Tensor z11) {
  int64_t m11;
  (m11 = tensorSize(x8));
  int64_t z_offset1;
  (z_offset1 = (s_idx7 * m11));
  {
    int64_t niterations13 = m11;
    int64_t tpb13 = 256;
    int64_t nblocks13 = (((niterations13 + (tpb13 * 10)) - 1) / (tpb13 * 10));
    loopKernel13<<<nblocks13, tpb13>>>(m11, x8, c3, z11, z_offset1);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void iterfun12(Tensor1 y1, float c4, Tensor z12, int64_t m12, int64_t i28) {
  int64_t idx14;
  (idx14 = ((y1.data)[(i28 + (y1.offset))]));
  int64_t offset2;
  (offset2 = (i28 * m12));
  int64_t z_idx4;
  (z_idx4 = (idx14 + offset2));
  float t98;
  (t98 = ((z12.data)[(z_idx4 + (z12.offset))]));
  float t99;
  (t99 = (t98 + c4));
  (((z12.data)[(z_idx4 + (z12.offset))]) = t99);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z12.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z12.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel14(int64_t n17, Tensor1 y1, float c4, Tensor z12, int64_t m12) {
  int64_t idx15 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride14 = (gridDim.x * blockDim.x);
  while ((idx15 < n17)) {
    iterfun12(y1, c4, z12, m12, idx15);
    (idx15 = (idx15 + stride14));
  }
}
__host__ void tensorOpExp__z____1_Hot_y____scalar_c_(int64_t s_max9, Tensor1 y1, float c4, Tensor z12) {
  Seq t100;
  (t100 = tensor_shape((z12.dims), (z12.rank)));
  int64_t m12;
  (m12 = ((t100.seq)[1]));
  {
    int64_t niterations14 = s_max9;
    int64_t tpb14 = 256;
    int64_t nblocks14 = (((niterations14 + (tpb14 * 10)) - 1) / (tpb14 * 10));
    loopKernel14<<<nblocks14, tpb14>>>(s_max9, y1, c4, z12, m12);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ void t101(Tensor z13, int64_t offset3, int64_t idx16, float v3, int64_t j4) {
  char t102;
  (t102 = (j4 == idx16));
  if ((t102 == 1)) {
    int64_t t103;
    (t103 = (j4 + offset3));
    (((z13.data)[(t103 + (z13.offset))]) = v3);
  } else {
    int64_t t104;
    (t104 = (j4 + offset3));
    (((z13.data)[(t104 + (z13.offset))]) = 0.);
  }
}
__host__ __device__ void iterfun13(Tensor1 y2, Tensor x9, Tensor z13, int64_t m13, int64_t i29) {
  int64_t offset3;
  (offset3 = (i29 * m13));
  int64_t idx16;
  (idx16 = ((y2.data)[(i29 + (y2.offset))]));
  int64_t t105;
  (t105 = (idx16 + offset3));
  float t106;
  (t106 = ((x9.data)[(t105 + (x9.offset))]));
  float t107;
  (t107 = (1. / t106));
  float v3;
  (v3 = (-t107));
  {
    int64_t i30 = 0;
    while ((i30 < m13)) {
      t101(z13, offset3, idx16, v3, i30);
      (i30 = (i30 + 1));
    }
  }
}
__global__ void loopKernel15(int64_t n18, Tensor1 y2, Tensor x9, Tensor z13, int64_t m13) {
  int64_t idx17 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride15 = (gridDim.x * blockDim.x);
  while ((idx17 < n18)) {
    iterfun13(y2, x9, z13, m13, idx17);
    (idx17 = (idx17 + stride15));
  }
}
__host__ void tensorOpExn__z___1_Hot_y____scalar__1__x_T___1_Hot_y___(int64_t s_max10, Tensor1 y2, Tensor x9, Tensor z13) {
  Seq t108;
  (t108 = tensor_shape((x9.dims), (x9.rank)));
  int64_t s8;
  (s8 = ((t108.seq)[0]));
  Seq t109;
  (t109 = tensor_shape((x9.dims), (x9.rank)));
  int64_t m13;
  (m13 = ((t109.seq)[1]));
  {
    int64_t niterations15 = s_max10;
    int64_t tpb15 = 256;
    int64_t nblocks15 = (((niterations15 + (tpb15 * 10)) - 1) / (tpb15 * 10));
    loopKernel15<<<nblocks15, tpb15>>>(s_max10, y2, x9, z13, m13);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ float t110(Tensor z14, int64_t m14, int64_t i31, float acc11, int64_t j5) {
  int64_t s_idx8;
  (s_idx8 = (j5 + 1));
  int64_t s_offset1;
  (s_offset1 = (s_idx8 * m14));
  int64_t t111;
  (t111 = (s_offset1 + i31));
  float t112;
  (t112 = ((z14.data)[(t111 + (z14.offset))]));
  {
    float t113;
    (t113 = (acc11 + t112));
    return t113;
  }
}
__host__ __device__ void iterfun14(Tensor z14, int64_t s9, int64_t m14, int64_t i31) {
  int64_t t114;
  (t114 = (s9 - 1));
  float v4;
  {
    int64_t i32 = 0;
    float acc12 = ((z14.data)[(i31 + (z14.offset))]);
    while ((i32 < t114)) {
      (acc12 = t110(z14, m14, i31, acc12, i32));
      (i32 = (i32 + 1));
    }
    (v4 = acc12);
  }
  (((z14.data)[(i31 + (z14.offset))]) = v4);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z14.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z14.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel16(int64_t n19, Tensor z14, int64_t s9, int64_t m14) {
  int64_t idx18 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride16 = (gridDim.x * blockDim.x);
  while ((idx18 < n19)) {
    iterfun14(z14, s9, m14, idx18);
    (idx18 = (idx18 + stride16));
  }
}
__host__ void tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(Tensor z14) {
  Seq t115;
  (t115 = tensor_shape((z14.dims), (z14.rank)));
  int64_t s9;
  (s9 = ((t115.seq)[0]));
  int64_t t116;
  (t116 = tensorSize(z14));
  int64_t m14;
  (m14 = (t116 / s9));
  {
    int64_t niterations16 = m14;
    int64_t tpb16 = 256;
    int64_t nblocks16 = (((niterations16 + (tpb16 * 10)) - 1) / (tpb16 * 10));
    loopKernel16<<<nblocks16, tpb16>>>(m14, z14, s9, m14);
    GPU_UTILS_CHECK_CUDA_ERROR();
    hipDeviceSynchronize();
    GPU_UTILS_CHECK_CUDA_ERROR();
  }
}
__host__ __device__ Tensor nnComponentOutBufs(Rec comp) {
  Tensor X = (comp.out_bufs);
  {
    Tensor t117;
    (t117 = X);
    return t117;
  }
}
__host__ __device__ Tensor nnComponentApplyExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max11, Tensor inputs1, Rec comp1) {
  int64_t ty1;
  int64_t X1 = (comp1.ty);
  (ty1 = X1);
  char t118;
  (t118 = (ty1 == nnCompType_FullyConnected));
  if ((t118 == 1)) {
    Tensor t119;
    Tensor X2 = (comp1.w);
    (t119 = X2);
    Tensor t120;
    Tensor X3 = (comp1.b);
    (t120 = X3);
    Tensor t121;
    Tensor X4 = (comp1.out_bufs);
    (t121 = X4);
    tensorOpExn__z___Wx_B(s_max11, t119, inputs1, t120, t121);
    Tensor X5 = (comp1.out_bufs);
    {
      Tensor t122;
      (t122 = X5);
      return t122;
    }
  } else {
    char t123;
    (t123 = (ty1 == nnCompType_ReLU));
    if ((t123 == 1)) {
      Tensor t124;
      Tensor X6 = (comp1.out_bufs);
      (t124 = X6);
      tensorOpExn__z___ReLU_x_(s_max11, inputs1, t124);
      Tensor X7 = (comp1.out_bufs);
      {
        Tensor t125;
        (t125 = X7);
        return t125;
      }
    } else {
      char t126;
      (t126 = (ty1 == nnCompType_SoftMax));
      if ((t126 == 1)) {
        Tensor t127;
        Tensor X8 = (comp1.softmax_bufs);
        (t127 = X8);
        Tensor t128;
        Tensor X9 = (comp1.out_bufs);
        (t128 = X9);
        tensorOpExn__z___SoftMax_x_(s_max11, inputs1, t127, t128);
        Tensor X10 = (comp1.out_bufs);
        {
          Tensor t129;
          (t129 = X10);
          return t129;
        }
      } else {
        Tensor X11 = (comp1.out_bufs);
        {
          Tensor t130;
          (t130 = X11);
          return t130;
        }
      }
    }
  }
}
__host__ __device__ Tensor nnComponentBackpropExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max12, Tensor comp_inputs, Tensor output_grads, Rec comp2) {
  int64_t ty2;
  int64_t X12 = (comp2.ty);
  (ty2 = X12);
  char t131;
  (t131 = (ty2 == nnCompType_FullyConnected));
  if ((t131 == 1)) {
    Tensor t132;
    Tensor X13 = (comp2.b_grads);
    (t132 = X13);
    tensorOpExn__z___x(s_max12, output_grads, t132);
    Tensor t133;
    Tensor X14 = (comp2.w_grads);
    (t133 = X14);
    tensorOpExn__z___x___y_T(s_max12, output_grads, comp_inputs, t133);
    Tensor t134;
    Tensor X15 = (comp2.w);
    (t134 = X15);
    Tensor t135;
    Tensor X16 = (comp2.in_grads);
    (t135 = X16);
    tensorOpExn__z____x_T___W__T(s_max12, output_grads, t134, t135);
    Tensor X17 = (comp2.in_grads);
    {
      Tensor t136;
      (t136 = X17);
      return t136;
    }
  } else {
    char t137;
    (t137 = (ty2 == nnCompType_ReLU));
    if ((t137 == 1)) {
      Tensor t138;
      Tensor X18 = (comp2.out_bufs);
      (t138 = X18);
      Tensor t139;
      Tensor X19 = (comp2.in_grads);
      (t139 = X19);
      tensorOpExn__z___d_dx_l_ReLU_x__(s_max12, t138, output_grads, t139);
      Tensor X20 = (comp2.in_grads);
      {
        Tensor t140;
        (t140 = X20);
        return t140;
      }
    } else {
      char t141;
      (t141 = (ty2 == nnCompType_SoftMax));
      if ((t141 == 1)) {
        Tensor t142;
        Tensor X21 = (comp2.out_bufs);
        (t142 = X21);
        Tensor t143;
        Tensor X22 = (comp2.in_grads);
        (t143 = X22);
        tensorOpExn__z___d_dx_l_SoftMax_x___(s_max12, t142, output_grads, t143);
        Tensor X23 = (comp2.in_grads);
        {
          Tensor t144;
          (t144 = X23);
          return t144;
        }
      } else {
        Tensor X24 = (comp2.in_grads);
        {
          Tensor t145;
          (t145 = X24);
          return t145;
        }
      }
    }
  }
}
__host__ __device__ void nnComponent_TEMP_SetGradients(int64_t nnCompType_FullyConnected, float scalar, Rec comp3) {
  int64_t ty3;
  int64_t X25 = (comp3.ty);
  (ty3 = X25);
  char t146;
  (t146 = (ty3 == nnCompType_FullyConnected));
  if ((t146 == 1)) {
    Tensor t147;
    Tensor X26 = (comp3.w_grads);
    (t147 = X26);
    tensorOpExn__Z___scalar_c_(scalar, t147);
    Tensor t148;
    Tensor X27 = (comp3.b_grads);
    (t148 = X27);
    tensorOpExn__Z___scalar_c_(scalar, t148);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ReduceGradients(int64_t nnCompType_FullyConnected, Rec comp4) {
  int64_t ty4;
  int64_t X28 = (comp4.ty);
  (ty4 = X28);
  char t149;
  (t149 = (ty4 == nnCompType_FullyConnected));
  if ((t149 == 1)) {
    Tensor t150;
    Tensor X29 = (comp4.w_grads);
    (t150 = X29);
    tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(t150);
    Tensor t151;
    Tensor X30 = (comp4.b_grads);
    (t151 = X30);
    tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(t151);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ScaleGradients(int64_t nnCompType_FullyConnected, float scalar1, Rec comp5) {
  int64_t ty5;
  int64_t X31 = (comp5.ty);
  (ty5 = X31);
  char t152;
  (t152 = (ty5 == nnCompType_FullyConnected));
  if ((t152 == 1)) {
    Tensor t153;
    Tensor X32 = (comp5.w_grads);
    (t153 = X32);
    tensorOpExn__z____scalar_c_(1, scalar1, t153);
    Tensor t154;
    Tensor X33 = (comp5.b_grads);
    (t154 = X33);
    tensorOpExn__z____scalar_c_(1, scalar1, t154);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ApplyGradients(int64_t nnCompType_FullyConnected, float scalar2, Rec comp6) {
  int64_t ty6;
  int64_t X34 = (comp6.ty);
  (ty6 = X34);
  char t155;
  (t155 = (ty6 == nnCompType_FullyConnected));
  if ((t155 == 1)) {
    Tensor t156;
    Tensor X35 = (comp6.w_grads);
    (t156 = X35);
    Tensor t157;
    Tensor X36 = (comp6.w);
    (t157 = X36);
    tensorOpExn__Z____x___scalar_c_(0, t156, scalar2, t157);
    Tensor t158;
    Tensor X37 = (comp6.b_grads);
    (t158 = X37);
    Tensor t159;
    Tensor X38 = (comp6.b);
    (t159 = X38);
    tensorOpExn__Z____x___scalar_c_(0, t158, scalar2, t159);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_L2Regularize(int64_t nnCompType_FullyConnected, float scalar3, Rec comp7) {
  int64_t ty7;
  int64_t X39 = (comp7.ty);
  (ty7 = X39);
  char t160;
  (t160 = (ty7 == nnCompType_FullyConnected));
  if ((t160 == 1)) {
    Tensor t161;
    Tensor X40 = (comp7.w);
    (t161 = X40);
    Tensor t162;
    Tensor X41 = (comp7.w_grads);
    (t162 = X41);
    tensorOpExn__z____X___scalar_c_(0, t161, scalar3, t162);
    Tensor t163;
    Tensor X42 = (comp7.b);
    (t163 = X42);
    Tensor t164;
    Tensor X43 = (comp7.b_grads);
    (t164 = X43);
    tensorOpExn__z____X___scalar_c_(0, t163, scalar3, t164);
  } else {
    ;
  }
}
__host__ __device__ void nnComponentZeroGrad(int64_t nnCompType_FullyConnected, Rec comp8) {
  nnComponent_TEMP_SetGradients(nnCompType_FullyConnected, 0., comp8);
}
__host__ __device__ Tensor nnLossFunctionApplyExn(int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, int64_t s_max13, Tensor inputs2, Rec1 lossfn1) {
  int64_t ty8;
  int64_t X44 = (lossfn1.ty);
  (ty8 = X44);
  char t165;
  (t165 = (ty8 == nnLossfnType_CrossEntropyLoss));
  if ((t165 == 1)) {
    {
      Tensor t166;
      (t166 = inputs2);
      return t166;
    }
  } else {
    char t167;
    (t167 = (ty8 == nnLossfnType_SoftMaxCrossEntropyLoss));
    if ((t167 == 1)) {
      Tensor t168;
      Tensor X45 = (lossfn1.softmax_bufs);
      (t168 = X45);
      Tensor t169;
      Tensor X46 = (lossfn1.in_grads);
      (t169 = X46);
      tensorOpExn__z___SoftMax_x_(s_max13, inputs2, t168, t169);
      Tensor X47 = (lossfn1.in_grads);
      {
        Tensor t170;
        (t170 = X47);
        return t170;
      }
    } else {
      {
        Tensor t171;
        (t171 = inputs2);
        return t171;
      }
    }
  }
}
__host__ __device__ Tensor nnLossFunctionBackpropExn(int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, int64_t s_max14, Tensor inputs3, Tensor1 expecteds, Rec1 lossfn2) {
  int64_t ty9;
  int64_t X48 = (lossfn2.ty);
  (ty9 = X48);
  char t172;
  (t172 = (ty9 == nnLossfnType_CrossEntropyLoss));
  if ((t172 == 1)) {
    Tensor t173;
    Tensor X49 = (lossfn2.in_grads);
    (t173 = X49);
    tensorOpExn__z___1_Hot_y____scalar__1__x_T___1_Hot_y___(s_max14, expecteds, inputs3, t173);
    Tensor X50 = (lossfn2.in_grads);
    {
      Tensor t174;
      (t174 = X50);
      return t174;
    }
  } else {
    char t175;
    (t175 = (ty9 == nnLossfnType_SoftMaxCrossEntropyLoss));
    if ((t175 == 1)) {
      float t176;
      (t176 = (-1.));
      Tensor t177;
      Tensor X51 = (lossfn2.in_grads);
      (t177 = X51);
      tensorOpExp__z____1_Hot_y____scalar_c_(s_max14, expecteds, t176, t177);
      Tensor X52 = (lossfn2.in_grads);
      {
        Tensor t178;
        (t178 = X52);
        return t178;
      }
    } else {
      Tensor X53 = (lossfn2.in_grads);
      {
        Tensor t179;
        (t179 = X53);
        return t179;
      }
    }
  }
}
__host__ __device__ int64_t t180(int64_t nnCompType_FullyConnected, int64_t x10, Rec comp9) {
  nnComponentZeroGrad(nnCompType_FullyConnected, comp9);
  {
    int64_t t181;
    (t181 = 0);
    return t181;
  }
}
__host__ __device__ int64_t foldl1(int64_t acc_init2, Seq1 s10, int64_t nnCompType_FullyConnected) {
  int64_t acc13 = acc_init2;
  int64_t i33 = 0;
  while ((i33 < (s10.len))) {
    (acc13 = t180(nnCompType_FullyConnected, acc13, ((s10.seq)[i33])));
    (i33 = (i33 + 1));
  }
  return acc13;
}
__host__ __device__ void nnZeroGrad(int64_t nnCompType_FullyConnected, Rec2 network) {
  Seq1 t182;
  Seq1 X54 = (network.components);
  (t182 = X54);
  int64_t _2;
  (_2 = foldl1(0, t182, nnCompType_FullyConnected));
  ;
}
__host__ __device__ Tensor t183(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max15, Tensor prevouts, Rec comp10) {
  {
    Tensor t184;
    (t184 = nnComponentApplyExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, prevouts, comp10));
    return t184;
  }
}
__host__ __device__ Tensor foldl2(Tensor acc_init3, Seq1 s11, int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max15) {
  Tensor acc14 = acc_init3;
  int64_t i34 = 0;
  while ((i34 < (s11.len))) {
    (acc14 = t183(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, acc14, ((s11.seq)[i34])));
    (i34 = (i34 + 1));
  }
  return acc14;
}
__host__ __device__ Tensor nnEvalExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network1, Tensor inputs4) {
  Seq t185;
  (t185 = tensor_shape((inputs4.dims), (inputs4.rank)));
  int64_t s_max15;
  (s_max15 = ((t185.seq)[0]));
  Seq1 t186;
  Seq1 X55 = (network1.components);
  (t186 = X55);
  Tensor comp_out;
  (comp_out = foldl2(inputs4, t186, nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15));
  Rec1 t187;
  Rec1 X56 = (network1.lossfn);
  (t187 = X56);
  {
    Tensor t188;
    (t188 = nnLossFunctionApplyExn(nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, s_max15, comp_out, t187));
    return t188;
  }
}
__host__ __device__ Tensor t189(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, Rec2 network2, int64_t s_max16, int64_t n_components, Tensor out_grads, int64_t i35) {
  int64_t t190;
  (t190 = (i35 + 2));
  int64_t idx19;
  (idx19 = (n_components - t190));
  int64_t previdx;
  (previdx = (idx19 - 1));
  Seq1 t191;
  Seq1 X57 = (network2.components);
  (t191 = X57);
  Rec comp11;
  (comp11 = ((t191.seq)[idx19]));
  Seq1 t192;
  Seq1 X58 = (network2.components);
  (t192 = X58);
  Rec t193;
  (t193 = ((t192.seq)[previdx]));
  Tensor in_bufs;
  (in_bufs = nnComponentOutBufs(t193));
  {
    Tensor t194;
    (t194 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, in_bufs, out_grads, comp11));
    return t194;
  }
}
__host__ __device__ void nnBackpropExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network2, Rec3 batch) {
  Tensor t195;
  Tensor X59 = (batch.inputs);
  (t195 = X59);
  Seq t196;
  (t196 = tensor_shape((t195.dims), (t195.rank)));
  int64_t s_max16;
  (s_max16 = ((t196.seq)[0]));
  Tensor t197;
  Tensor X60 = (batch.inputs);
  (t197 = X60);
  Tensor outputs;
  (outputs = nnEvalExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network2, t197));
  Tensor1 t198;
  Tensor1 X61 = (batch.correct_linear_outidxs);
  (t198 = X61);
  Rec1 t199;
  Rec1 X62 = (network2.lossfn);
  (t199 = X62);
  Tensor lossgrads;
  (lossgrads = nnLossFunctionBackpropExn(nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, s_max16, outputs, t198, t199));
  Seq1 t200;
  Seq1 X63 = (network2.components);
  (t200 = X63);
  int64_t n_components;
  (n_components = (t200.len));
  char t201;
  (t201 = (n_components == 0));
  if ((t201 == 1)) {
    ;
  } else {
    char t202;
    (t202 = (n_components == 1));
    if ((t202 == 1)) {
      Seq1 t203;
      Seq1 X64 = (network2.components);
      (t203 = X64);
      Rec lastcomp;
      (lastcomp = ((t203.seq)[0]));
      Tensor t204;
      Tensor X65 = (batch.inputs);
      (t204 = X65);
      Tensor _3;
      (_3 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, t204, lossgrads, lastcomp));
      ;
    } else {
      Seq1 t205;
      Seq1 X66 = (network2.components);
      (t205 = X66);
      int64_t t206;
      (t206 = (n_components - 1));
      Rec lastcomp1;
      (lastcomp1 = ((t205.seq)[t206]));
      Seq1 t207;
      Seq1 X67 = (network2.components);
      (t207 = X67);
      int64_t t208;
      (t208 = (n_components - 2));
      Rec t209;
      (t209 = ((t207.seq)[t208]));
      Tensor lastcomp_in_bufs;
      (lastcomp_in_bufs = nnComponentOutBufs(t209));
      Tensor lastcomp_in_grads;
      (lastcomp_in_grads = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, lastcomp_in_bufs, lossgrads, lastcomp1));
      int64_t t210;
      (t210 = (n_components - 2));
      Tensor firstcomp_out_grads;
      {
        int64_t i36 = 0;
        Tensor acc15 = lastcomp_in_grads;
        while ((i36 < t210)) {
          (acc15 = t189(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, network2, s_max16, n_components, acc15, i36));
          (i36 = (i36 + 1));
        }
        (firstcomp_out_grads = acc15);
      }
      Seq1 t211;
      Seq1 X68 = (network2.components);
      (t211 = X68);
      Rec firstcomp;
      (firstcomp = ((t211.seq)[0]));
      Tensor firstcomp_in_bufs;
      Tensor X69 = (batch.inputs);
      (firstcomp_in_bufs = X69);
      Tensor _4;
      (_4 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max16, firstcomp_in_bufs, firstcomp_out_grads, firstcomp));
      ;
    }
  }
}
__host__ __device__ int64_t t212(int64_t nnCompType_FullyConnected, float batchsize_regularizer, int64_t x11, Rec comp12) {
  nnComponent_TEMP_ReduceGradients(nnCompType_FullyConnected, comp12);
  nnComponent_TEMP_ScaleGradients(nnCompType_FullyConnected, batchsize_regularizer, comp12);
  {
    int64_t t213;
    (t213 = 0);
    return t213;
  }
}
__host__ __device__ int64_t t214(int64_t nnCompType_FullyConnected, float lambda, int64_t x12, Rec comp13) {
  float t215;
  (t215 = (2. * lambda));
  nnComponent_TEMP_L2Regularize(nnCompType_FullyConnected, t215, comp13);
  {
    int64_t t216;
    (t216 = 0);
    return t216;
  }
}
__host__ __device__ int64_t t217(int64_t nnCompType_FullyConnected, float alpha, int64_t x13, Rec comp14) {
  float t218;
  (t218 = (-alpha));
  nnComponent_TEMP_ApplyGradients(nnCompType_FullyConnected, t218, comp14);
  {
    int64_t t219;
    (t219 = 0);
    return t219;
  }
}
__host__ __device__ int64_t foldl3(int64_t acc_init4, Seq1 s12, int64_t nnCompType_FullyConnected, float alpha) {
  int64_t acc16 = acc_init4;
  int64_t i37 = 0;
  while ((i37 < (s12.len))) {
    (acc16 = t217(nnCompType_FullyConnected, alpha, acc16, ((s12.seq)[i37])));
    (i37 = (i37 + 1));
  }
  return acc16;
}
__host__ __device__ int64_t foldl4(int64_t acc_init5, Seq1 s13, int64_t nnCompType_FullyConnected, float lambda) {
  int64_t acc17 = acc_init5;
  int64_t i38 = 0;
  while ((i38 < (s13.len))) {
    (acc17 = t214(nnCompType_FullyConnected, lambda, acc17, ((s13.seq)[i38])));
    (i38 = (i38 + 1));
  }
  return acc17;
}
__host__ __device__ int64_t foldl5(int64_t acc_init6, Seq1 s14, int64_t nnCompType_FullyConnected, float batchsize_regularizer) {
  int64_t acc18 = acc_init6;
  int64_t i39 = 0;
  while ((i39 < (s14.len))) {
    (acc18 = t212(nnCompType_FullyConnected, batchsize_regularizer, acc18, ((s14.seq)[i39])));
    (i39 = (i39 + 1));
  }
  return acc18;
}
__host__ __device__ void nnGradientDescentExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network3, float alpha, float lambda, Rec3 batch1) {
  nnZeroGrad(nnCompType_FullyConnected, network3);
  Tensor t220;
  Tensor X70 = (batch1.inputs);
  (t220 = X70);
  Seq t221;
  (t221 = tensor_shape((t220.dims), (t220.rank)));
  int64_t batchsize1;
  (batchsize1 = ((t221.seq)[0]));
  nnBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network3, batch1);
  float t222;
  (t222 = (( float ) batchsize1));
  float batchsize_regularizer;
  (batchsize_regularizer = (1. / t222));
  Seq1 t223;
  Seq1 X71 = (network3.components);
  (t223 = X71);
  int64_t _5;
  (_5 = foldl5(0, t223, nnCompType_FullyConnected, batchsize_regularizer));
  char t224;
  (t224 = (lambda == 0.));
  if ((t224 == 1)) {
    ;
  } else {
    Seq1 t225;
    Seq1 X72 = (network3.components);
    (t225 = X72);
    int64_t _6;
    (_6 = foldl4(0, t225, nnCompType_FullyConnected, lambda));
    ;
  }
  Seq1 t226;
  Seq1 X73 = (network3.components);
  (t226 = X73);
  int64_t _7;
  (_7 = foldl3(0, t226, nnCompType_FullyConnected, alpha));
  ;
}
__host__ __device__ int64_t t227(int64_t acc19, int64_t e1) {
  {
    int64_t t228;
    (t228 = (acc19 * e1));
    return t228;
  }
}
__host__ __device__ int64_t foldl6(int64_t acc_init7, Seq s15) {
  int64_t acc20 = acc_init7;
  int64_t i40 = 0;
  while ((i40 < (s15.len))) {
    (acc20 = t227(acc20, ((s15.seq)[i40])));
    (i40 = (i40 + 1));
  }
  return acc20;
}
__host__ __device__ int64_t tensorSize1(Tensor t229) {
  Seq t230;
  (t230 = tensor_shape((t229.dims), (t229.rank)));
  {
    int64_t t231;
    (t231 = foldl6(1, t230));
    return t231;
  }
}
__host__ __device__ int64_t t232(Tensor t233, int64_t s_offset2, int64_t cand_idx, int64_t idx20) {
  int64_t t234;
  (t234 = (idx20 + s_offset2));
  float t235;
  (t235 = ((t233.data)[(t234 + (t233.offset))]));
  int64_t t236;
  (t236 = (cand_idx + s_offset2));
  float t237;
  (t237 = ((t233.data)[(t236 + (t233.offset))]));
  char t238;
  (t238 = (t235 > t237));
  if ((t238 == 1)) {
    {
      int64_t t239;
      (t239 = idx20);
      return t239;
    }
  } else {
    {
      int64_t t240;
      (t240 = cand_idx);
      return t240;
    }
  }
}
__host__ __device__ int64_t tensorLinearMaxIdx(int64_t s_idx9, Tensor t233) {
  Seq t241;
  (t241 = tensor_shape((t233.dims), (t233.rank)));
  int64_t s16;
  (s16 = ((t241.seq)[0]));
  int64_t t242;
  (t242 = tensorSize1(t233));
  int64_t size1;
  (size1 = (t242 / s16));
  int64_t s_offset2;
  (s_offset2 = (s_idx9 * size1));
  int64_t t243;
  (t243 = (size1 - 1));
  {
    int64_t t244;
    {
      int64_t i41 = 0;
      int64_t acc21 = (size1 - 1);
      while ((i41 < t243)) {
        (acc21 = t232(t233, s_offset2, acc21, i41));
        (i41 = (i41 + 1));
      }
      (t244 = acc21);
    }
    return t244;
  }
}
__host__ __device__ int64_t t245(Rec3 batch2, Tensor outputs1, int64_t b_acc, int64_t b_idx) {
  int64_t seqAlloc[1];
  Seq t246;
  int64_t t247;
  (t247 = tensorLinearMaxIdx(b_idx, outputs1));
  Tensor1 t248;
  Tensor1 X74 = (batch2.correct_linear_outidxs);
  (t248 = X74);
  ((seqAlloc[0]) = b_idx);
  ((t246.seq) = seqAlloc);
  ((t246.len) = 1);
  int64_t t249;
  (t249 = ((t248.data)[(cartesian_to_linear_index((t248.dims), (t248.rank), t246) + (t248.offset))]));
  char t250;
  (t250 = (t247 == t249));
  if ((t250 == 1)) {
    {
      int64_t t251;
      (t251 = (b_acc + 1));
      return t251;
    }
  } else {
    {
      int64_t t252;
      (t252 = b_acc);
      return t252;
    }
  }
}
__host__ __device__ int64_t t253(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, char printStatus1, Rec2 network4, Seq2 batches, int64_t acc22, int64_t i42) {
  char (*t254) = "/";
  char (*t255) = "\r";
  if ((printStatus1 == 1)) {
    printf("%s", t255);
    int64_t t256;
    (t256 = (i42 + 1));
    float t257;
    (t257 = (( float ) t256));
    printf("%f", t257);
    printf("%s", t254);
    int64_t t258;
    (t258 = (batches.len));
    float t259;
    (t259 = (( float ) t258));
    printf("%f", t259);
  } else {
    ;
  }
  Rec3 batch2;
  (batch2 = ((batches.seq)[i42]));
  Tensor t260;
  Tensor X75 = (batch2.inputs);
  (t260 = X75);
  Seq t261;
  (t261 = tensor_shape((t260.dims), (t260.rank)));
  int64_t batchsize2;
  (batchsize2 = ((t261.seq)[0]));
  Tensor t262;
  Tensor X76 = (batch2.inputs);
  (t262 = X76);
  Tensor outputs1;
  (outputs1 = nnEvalExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network4, t262));
  int64_t batch_accuracy;
  {
    int64_t i43 = 0;
    int64_t acc23 = 0;
    while ((i43 < batchsize2)) {
      (acc23 = t245(batch2, outputs1, acc23, i43));
      (i43 = (i43 + 1));
    }
    (batch_accuracy = acc23);
  }
  {
    int64_t t263;
    (t263 = (acc22 + batch_accuracy));
    return t263;
  }
}
__host__ __device__ int64_t nnAccuracyDiscrete(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, char printStatus1, Rec2 network4, Seq2 batches) {
  char (*t264) = "\n";
  int64_t t265;
  (t265 = (batches.len));
  int64_t correct_guesses;
  {
    int64_t i44 = 0;
    int64_t acc24 = 0;
    while ((i44 < t265)) {
      (acc24 = t253(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, printStatus1, network4, batches, acc24, i44));
      (i44 = (i44 + 1));
    }
    (correct_guesses = acc24);
  }
  if ((printStatus1 == 1)) {
    printf("%s", t264);
  } else {
    ;
  }
  {
    int64_t t266;
    (t266 = correct_guesses);
    return t266;
  }
}
__host__ __device__ int64_t t267(int64_t acc25, Rec3 batch3) {
  Tensor t268;
  Tensor X77 = (batch3.inputs);
  (t268 = X77);
  Seq t269;
  (t269 = tensor_shape((t268.dims), (t268.rank)));
  int64_t t270;
  (t270 = ((t269.seq)[0]));
  {
    int64_t t271;
    (t271 = (acc25 + t270));
    return t271;
  }
}
__host__ __device__ int64_t foldl7(int64_t acc_init8, Seq2 s17) {
  int64_t acc26 = acc_init8;
  int64_t i45 = 0;
  while ((i45 < (s17.len))) {
    (acc26 = t267(acc26, ((s17.seq)[i45])));
    (i45 = (i45 + 1));
  }
  return acc26;
}
__host__ __device__ float nnAccuracyProportion(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, char printStatus2, Rec2 network5, Seq2 batches1) {
  int64_t correct_guesses1;
  (correct_guesses1 = nnAccuracyDiscrete(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, printStatus2, network5, batches1));
  int64_t datalength;
  (datalength = foldl7(0, batches1));
  float t272;
  (t272 = (( float ) correct_guesses1));
  float t273;
  (t273 = (( float ) datalength));
  {
    float t274;
    (t274 = (t272 / t273));
    return t274;
  }
}
__host__ __device__ void wrappedPrint(char (*s18)) {
  printf("%s", s18);
}
__host__ __device__ void t275(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, float alpha1, float lambda1, int64_t batch_idx) {
  char (*t276) = "/";
  char (*t277) = "\rround ";
  char t278;
  char X78 = (params.printStatus);
  (t278 = X78);
  if ((t278 == 1)) {
    wrappedPrint(t277);
    int64_t t279;
    (t279 = (batch_idx + 1));
    float t280;
    (t280 = (( float ) t279));
    printf("%f", t280);
    wrappedPrint(t276);
    float t281;
    (t281 = (( float ) rounds));
    printf("%f", t281);
  } else {
    ;
  }
  Rec3 t282;
  (t282 = ((training_batches.seq)[batch_idx]));
  nnGradientDescentExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network6, alpha1, lambda1, t282);
}
__host__ __device__ Rec5 t283(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, Seq2 validation_batches, Rec5 acc27, int64_t epoch_idx) {
  Rec5 alloc;
  char (*t284) = "%\n";
  char (*t285) = "Computed accuracy: ";
  char (*t286) = "evalating performance...\n";
  char (*t287) = "\n";
  char (*t288) = "]\n";
  char (*t289) = "[lambda = ";
  char (*t290) = "]\n";
  char (*t291) = "[alpha = ";
  char (*t292) = "]\n";
  char (*t293) = "/";
  char (*t294) = "[Iteration ";
  int64_t epoch;
  (epoch = (epoch_idx + 1));
  float alpha1;
  float X79 = (acc27._0);
  (alpha1 = X79);
  float lambda1;
  float X80 = (acc27._1);
  (lambda1 = X80);
  char t295;
  char X81 = (params.printStatus);
  (t295 = X81);
  if ((t295 == 1)) {
    wrappedPrint(t294);
    float t296;
    (t296 = (( float ) epoch));
    printf("%f", t296);
    wrappedPrint(t293);
    int64_t t297;
    int64_t X82 = (params.epochs);
    (t297 = X82);
    float t298;
    (t298 = (( float ) t297));
    printf("%f", t298);
    printf("%s", t292);
    wrappedPrint(t291);
    printf("%f", alpha1);
    wrappedPrint(t290);
    wrappedPrint(t289);
    printf("%f", lambda1);
    wrappedPrint(t288);
  } else {
    ;
  }
  int64_t t299;
  (t299 = (training_batches.len));
  {
    int64_t i46 = 0;
    while ((i46 < t299)) {
      t275(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, params, network6, rounds, training_batches, alpha1, lambda1, i46);
      (i46 = (i46 + 1));
    }
  }
  wrappedPrint(t287);
  char t300;
  char X83 = (params.evaluateBetweenEpochs);
  (t300 = X83);
  if ((t300 == 1)) {
    char t301;
    char X84 = (params.printStatus);
    (t301 = X84);
    if ((t301 == 1)) {
      wrappedPrint(t286);
    } else {
      ;
    }
    char t302;
    char X85 = (params.printStatus);
    (t302 = X85);
    float accuracy;
    (accuracy = nnAccuracyProportion(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, t302, network6, validation_batches));
    char t303;
    char X86 = (params.printStatus);
    (t303 = X86);
    if ((t303 == 1)) {
      wrappedPrint(t285);
      float t304;
      (t304 = (accuracy * 100.));
      printf("%f", t304);
      wrappedPrint(t284);
    } else {
      ;
    }
  } else {
    ;
  }
  float t305;
  float X87 = (params.decay_alpha);
  (t305 = X87);
  float t306;
  (t306 = (1. - t305));
  float decayed_alpha;
  (decayed_alpha = (alpha1 * t306));
  float t307;
  float X88 = (params.decay_lambda);
  (t307 = X88);
  float t308;
  (t308 = (1. - t307));
  float decayed_lambda;
  (decayed_lambda = (lambda1 * t308));
  ((alloc._0) = decayed_alpha);
  ((alloc._1) = decayed_lambda);
  {
    Rec5 t309;
    (t309 = alloc);
    return t309;
  }
}
void cuda_wrap(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, Seq2 validation_batches) {
  Rec5 t310;
  char (*t311) = "%\n";
  char (*t312) = "Computed accuracy: ";
  char (*t313) = "evalating performance...\n";
  char t314;
  char X89 = (params.evaluateBeforeFirstEpoch);
  (t314 = X89);
  if ((t314 == 1)) {
    char t315;
    char X90 = (params.printStatus);
    (t315 = X90);
    if ((t315 == 1)) {
      wrappedPrint(t313);
    } else {
      ;
    }
    char t316;
    char X91 = (params.printStatus);
    (t316 = X91);
    float accuracy1;
    (accuracy1 = nnAccuracyProportion(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, t316, network6, validation_batches));
    char t317;
    char X92 = (params.printStatus);
    (t317 = X92);
    if ((t317 == 1)) {
      wrappedPrint(t312);
      float t318;
      (t318 = (accuracy1 * 100.));
      printf("%f", t318);
      wrappedPrint(t311);
    } else {
      ;
    }
  } else {
    ;
  }
  float t319;
  float X93 = (params.init_lambda);
  (t319 = X93);
  float t320;
  float X94 = (params.init_alpha);
  (t320 = X94);
  ((t310._0) = t320);
  ((t310._1) = t319);
  int64_t t321;
  int64_t X95 = (params.epochs);
  (t321 = X95);
  Rec5 _8;
  {
    int64_t i47 = 0;
    Rec5 acc28 = t310;
    while ((i47 < t321)) {
      (acc28 = t283(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, params, network6, rounds, training_batches, validation_batches, acc28, i47));
      (i47 = (i47 + 1));
    }
    (_8 = acc28);
  }
  ;
}
extern "C" void vjivEXkNw6z(value nnCompType_FullyConnected, value nnCompType_ReLU, value nnCompType_SoftMax, value nnLossfnType_CrossEntropyLoss, value nnLossfnType_SoftMaxCrossEntropyLoss, value params, value network6, value rounds, value training_batches, value validation_batches) {
  CAMLparam5(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss);
  CAMLxparam5(params, network6, rounds, training_batches, validation_batches);
  int64_t tensor_count = 0;
  int64_t gpu_tmp = Long_val(nnCompType_FullyConnected);
  int64_t gpu_tmp1 = Long_val(nnCompType_ReLU);
  int64_t gpu_tmp2 = Long_val(nnCompType_SoftMax);
  int64_t gpu_tmp3 = Long_val(nnLossfnType_CrossEntropyLoss);
  int64_t gpu_tmp4 = Long_val(nnLossfnType_SoftMaxCrossEntropyLoss);
  Rec4 gpu_tmp5;
  int64_t cuda_rec_field = Long_val(Field(params, 0));
  ((gpu_tmp5.epochs) = cuda_rec_field);
  int64_t cuda_rec_field1 = Long_val(Field(params, 1));
  ((gpu_tmp5.batchsize) = cuda_rec_field1);
  float cuda_rec_field2 = Double_val(Field(params, 2));
  ((gpu_tmp5.init_alpha) = cuda_rec_field2);
  float cuda_rec_field3 = Double_val(Field(params, 3));
  ((gpu_tmp5.decay_alpha) = cuda_rec_field3);
  float cuda_rec_field4 = Double_val(Field(params, 4));
  ((gpu_tmp5.init_lambda) = cuda_rec_field4);
  char cuda_rec_field5 = Int_val(Field(params, 5));
  ((gpu_tmp5.printStatus) = cuda_rec_field5);
  float cuda_rec_field6 = Double_val(Field(params, 6));
  ((gpu_tmp5.decay_lambda) = cuda_rec_field6);
  char cuda_rec_field7 = Int_val(Field(params, 7));
  ((gpu_tmp5.evaluateBetweenEpochs) = cuda_rec_field7);
  char cuda_rec_field8 = Int_val(Field(params, 8));
  ((gpu_tmp5.evaluateBeforeFirstEpoch) = cuda_rec_field8);
  Rec2 gpu_tmp6;
  Rec1 cuda_rec_field9;
  int64_t cuda_rec_field10 = Long_val(Field(Field(network6, 0), 0));
  ((cuda_rec_field9.ty) = cuda_rec_field10);
  Tensor cuda_rec_field11;
  ((cuda_rec_field11.rank) = (Caml_ba_array_val(Field(Field(network6, 0), 1))->num_dims));
  if (((cuda_rec_field11.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field11.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field11.offset) = 0);
  ((cuda_rec_field11.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i48 = 0;
  int64_t n20 = (sizeof(float));
  while ((i48 < (cuda_rec_field11.rank))) {
    (((cuda_rec_field11.dims)[i48]) = ((Caml_ba_array_val(Field(Field(network6, 0), 1))->dim)[i48]));
    (n20 = (n20 * ((Caml_ba_array_val(Field(Field(network6, 0), 1))->dim)[i48])));
    (i48 = (i48 + 1));
  }
  ((cuda_rec_field11.size) = n20);
  float (*t322);
  hipMallocManaged((&t322), n20);
  GPU_UTILS_CHECK_CUDA_ERROR();
  double (*t_ocaml) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 1)));
  int64_t i49 = 0;
  int64_t elems = (n20 / (sizeof(float)));
  while ((i49 < elems)) {
    ((t322[i49]) = (( float ) (t_ocaml[i49])));
    (i49 = (i49 + 1));
  }
  ((cuda_rec_field11.data) = t322);
  ((cuda_rec_field9.in_grads) = cuda_rec_field11);
  Tensor cuda_rec_field12;
  ((cuda_rec_field12.rank) = (Caml_ba_array_val(Field(Field(network6, 0), 2))->num_dims));
  if (((cuda_rec_field12.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field12.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field12.offset) = 0);
  ((cuda_rec_field12.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i50 = 0;
  int64_t n21 = (sizeof(float));
  while ((i50 < (cuda_rec_field12.rank))) {
    (((cuda_rec_field12.dims)[i50]) = ((Caml_ba_array_val(Field(Field(network6, 0), 2))->dim)[i50]));
    (n21 = (n21 * ((Caml_ba_array_val(Field(Field(network6, 0), 2))->dim)[i50])));
    (i50 = (i50 + 1));
  }
  ((cuda_rec_field12.size) = n21);
  float (*t323);
  hipMallocManaged((&t323), n21);
  GPU_UTILS_CHECK_CUDA_ERROR();
  double (*t_ocaml1) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 2)));
  int64_t i51 = 0;
  int64_t elems1 = (n21 / (sizeof(float)));
  while ((i51 < elems1)) {
    ((t323[i51]) = (( float ) (t_ocaml1[i51])));
    (i51 = (i51 + 1));
  }
  ((cuda_rec_field12.data) = t323);
  ((cuda_rec_field9.out_bufs) = cuda_rec_field12);
  Tensor cuda_rec_field13;
  ((cuda_rec_field13.rank) = (Caml_ba_array_val(Field(Field(network6, 0), 3))->num_dims));
  if (((cuda_rec_field13.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field13.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field13.offset) = 0);
  ((cuda_rec_field13.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i52 = 0;
  int64_t n22 = (sizeof(float));
  while ((i52 < (cuda_rec_field13.rank))) {
    (((cuda_rec_field13.dims)[i52]) = ((Caml_ba_array_val(Field(Field(network6, 0), 3))->dim)[i52]));
    (n22 = (n22 * ((Caml_ba_array_val(Field(Field(network6, 0), 3))->dim)[i52])));
    (i52 = (i52 + 1));
  }
  ((cuda_rec_field13.size) = n22);
  float (*t324);
  hipMallocManaged((&t324), n22);
  GPU_UTILS_CHECK_CUDA_ERROR();
  double (*t_ocaml2) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 3)));
  int64_t i53 = 0;
  int64_t elems2 = (n22 / (sizeof(float)));
  while ((i53 < elems2)) {
    ((t324[i53]) = (( float ) (t_ocaml2[i53])));
    (i53 = (i53 + 1));
  }
  ((cuda_rec_field13.data) = t324);
  ((cuda_rec_field9.softmax_bufs) = cuda_rec_field13);
  ((gpu_tmp6.lossfn) = cuda_rec_field9);
  Seq1 cuda_rec_field14;
  ((cuda_rec_field14.len) = Wosize_val(Field(network6, 1)));
  hipMallocManaged((&(cuda_rec_field14.seq)), (Wosize_val(Field(network6, 1)) * (sizeof(Rec))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i54 = 0;
  while ((i54 < (cuda_rec_field14.len))) {
    Rec cuda_seq_temp;
    Tensor cuda_rec_field15;
    ((cuda_rec_field15.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 0))->num_dims));
    if (((cuda_rec_field15.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field15.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field15.offset) = 0);
    ((cuda_rec_field15.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i55 = 0;
    int64_t n23 = (sizeof(float));
    while ((i55 < (cuda_rec_field15.rank))) {
      (((cuda_rec_field15.dims)[i55]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 0))->dim)[i55]));
      (n23 = (n23 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 0))->dim)[i55])));
      (i55 = (i55 + 1));
    }
    ((cuda_rec_field15.size) = n23);
    float (*t325);
    hipMallocManaged((&t325), n23);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml3) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 0)));
    int64_t i56 = 0;
    int64_t elems3 = (n23 / (sizeof(float)));
    while ((i56 < elems3)) {
      ((t325[i56]) = (( float ) (t_ocaml3[i56])));
      (i56 = (i56 + 1));
    }
    ((cuda_rec_field15.data) = t325);
    ((cuda_seq_temp.b) = cuda_rec_field15);
    Tensor cuda_rec_field16;
    ((cuda_rec_field16.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 1))->num_dims));
    if (((cuda_rec_field16.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field16.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field16.offset) = 0);
    ((cuda_rec_field16.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i57 = 0;
    int64_t n24 = (sizeof(float));
    while ((i57 < (cuda_rec_field16.rank))) {
      (((cuda_rec_field16.dims)[i57]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 1))->dim)[i57]));
      (n24 = (n24 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 1))->dim)[i57])));
      (i57 = (i57 + 1));
    }
    ((cuda_rec_field16.size) = n24);
    float (*t326);
    hipMallocManaged((&t326), n24);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml4) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 1)));
    int64_t i58 = 0;
    int64_t elems4 = (n24 / (sizeof(float)));
    while ((i58 < elems4)) {
      ((t326[i58]) = (( float ) (t_ocaml4[i58])));
      (i58 = (i58 + 1));
    }
    ((cuda_rec_field16.data) = t326);
    ((cuda_seq_temp.w) = cuda_rec_field16);
    int64_t cuda_rec_field17 = Long_val(Field(Field(Field(network6, 1), i54), 2));
    ((cuda_seq_temp.ty) = cuda_rec_field17);
    Tensor cuda_rec_field18;
    ((cuda_rec_field18.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 3))->num_dims));
    if (((cuda_rec_field18.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field18.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field18.offset) = 0);
    ((cuda_rec_field18.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i59 = 0;
    int64_t n25 = (sizeof(float));
    while ((i59 < (cuda_rec_field18.rank))) {
      (((cuda_rec_field18.dims)[i59]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 3))->dim)[i59]));
      (n25 = (n25 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 3))->dim)[i59])));
      (i59 = (i59 + 1));
    }
    ((cuda_rec_field18.size) = n25);
    float (*t327);
    hipMallocManaged((&t327), n25);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml5) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 3)));
    int64_t i60 = 0;
    int64_t elems5 = (n25 / (sizeof(float)));
    while ((i60 < elems5)) {
      ((t327[i60]) = (( float ) (t_ocaml5[i60])));
      (i60 = (i60 + 1));
    }
    ((cuda_rec_field18.data) = t327);
    ((cuda_seq_temp.b_grads) = cuda_rec_field18);
    Tensor cuda_rec_field19;
    ((cuda_rec_field19.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 4))->num_dims));
    if (((cuda_rec_field19.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field19.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field19.offset) = 0);
    ((cuda_rec_field19.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i61 = 0;
    int64_t n26 = (sizeof(float));
    while ((i61 < (cuda_rec_field19.rank))) {
      (((cuda_rec_field19.dims)[i61]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 4))->dim)[i61]));
      (n26 = (n26 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 4))->dim)[i61])));
      (i61 = (i61 + 1));
    }
    ((cuda_rec_field19.size) = n26);
    float (*t328);
    hipMallocManaged((&t328), n26);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml6) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 4)));
    int64_t i62 = 0;
    int64_t elems6 = (n26 / (sizeof(float)));
    while ((i62 < elems6)) {
      ((t328[i62]) = (( float ) (t_ocaml6[i62])));
      (i62 = (i62 + 1));
    }
    ((cuda_rec_field19.data) = t328);
    ((cuda_seq_temp.w_grads) = cuda_rec_field19);
    Tensor cuda_rec_field20;
    ((cuda_rec_field20.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 5))->num_dims));
    if (((cuda_rec_field20.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field20.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field20.offset) = 0);
    ((cuda_rec_field20.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i63 = 0;
    int64_t n27 = (sizeof(float));
    while ((i63 < (cuda_rec_field20.rank))) {
      (((cuda_rec_field20.dims)[i63]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 5))->dim)[i63]));
      (n27 = (n27 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 5))->dim)[i63])));
      (i63 = (i63 + 1));
    }
    ((cuda_rec_field20.size) = n27);
    float (*t329);
    hipMallocManaged((&t329), n27);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml7) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 5)));
    int64_t i64 = 0;
    int64_t elems7 = (n27 / (sizeof(float)));
    while ((i64 < elems7)) {
      ((t329[i64]) = (( float ) (t_ocaml7[i64])));
      (i64 = (i64 + 1));
    }
    ((cuda_rec_field20.data) = t329);
    ((cuda_seq_temp.in_grads) = cuda_rec_field20);
    Tensor cuda_rec_field21;
    ((cuda_rec_field21.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 6))->num_dims));
    if (((cuda_rec_field21.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field21.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field21.offset) = 0);
    ((cuda_rec_field21.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i65 = 0;
    int64_t n28 = (sizeof(float));
    while ((i65 < (cuda_rec_field21.rank))) {
      (((cuda_rec_field21.dims)[i65]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 6))->dim)[i65]));
      (n28 = (n28 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 6))->dim)[i65])));
      (i65 = (i65 + 1));
    }
    ((cuda_rec_field21.size) = n28);
    float (*t330);
    hipMallocManaged((&t330), n28);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml8) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 6)));
    int64_t i66 = 0;
    int64_t elems8 = (n28 / (sizeof(float)));
    while ((i66 < elems8)) {
      ((t330[i66]) = (( float ) (t_ocaml8[i66])));
      (i66 = (i66 + 1));
    }
    ((cuda_rec_field21.data) = t330);
    ((cuda_seq_temp.out_bufs) = cuda_rec_field21);
    Tensor cuda_rec_field22;
    ((cuda_rec_field22.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 7))->num_dims));
    if (((cuda_rec_field22.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field22.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field22.offset) = 0);
    ((cuda_rec_field22.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i67 = 0;
    int64_t n29 = (sizeof(float));
    while ((i67 < (cuda_rec_field22.rank))) {
      (((cuda_rec_field22.dims)[i67]) = ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 7))->dim)[i67]));
      (n29 = (n29 * ((Caml_ba_array_val(Field(Field(Field(network6, 1), i54), 7))->dim)[i67])));
      (i67 = (i67 + 1));
    }
    ((cuda_rec_field22.size) = n29);
    float (*t331);
    hipMallocManaged((&t331), n29);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml9) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i54), 7)));
    int64_t i68 = 0;
    int64_t elems9 = (n29 / (sizeof(float)));
    while ((i68 < elems9)) {
      ((t331[i68]) = (( float ) (t_ocaml9[i68])));
      (i68 = (i68 + 1));
    }
    ((cuda_rec_field22.data) = t331);
    ((cuda_seq_temp.softmax_bufs) = cuda_rec_field22);
    (((cuda_rec_field14.seq)[i54]) = cuda_seq_temp);
    (i54 = (i54 + 1));
  }
  ((gpu_tmp6.components) = cuda_rec_field14);
  int64_t gpu_tmp7 = Long_val(rounds);
  Seq2 gpu_tmp8;
  ((gpu_tmp8.len) = Wosize_val(training_batches));
  hipMallocManaged((&(gpu_tmp8.seq)), (Wosize_val(training_batches) * (sizeof(Rec3))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i69 = 0;
  while ((i69 < (gpu_tmp8.len))) {
    Rec3 cuda_seq_temp1;
    Tensor cuda_rec_field23;
    ((cuda_rec_field23.rank) = (Caml_ba_array_val(Field(Field(training_batches, i69), 0))->num_dims));
    if (((cuda_rec_field23.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field23.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field23.offset) = 0);
    ((cuda_rec_field23.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i70 = 0;
    int64_t n30 = (sizeof(float));
    while ((i70 < (cuda_rec_field23.rank))) {
      (((cuda_rec_field23.dims)[i70]) = ((Caml_ba_array_val(Field(Field(training_batches, i69), 0))->dim)[i70]));
      (n30 = (n30 * ((Caml_ba_array_val(Field(Field(training_batches, i69), 0))->dim)[i70])));
      (i70 = (i70 + 1));
    }
    ((cuda_rec_field23.size) = n30);
    float (*t332);
    hipMallocManaged((&t332), n30);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml10) = (( double (*) ) Caml_ba_data_val(Field(Field(training_batches, i69), 0)));
    int64_t i71 = 0;
    int64_t elems10 = (n30 / (sizeof(float)));
    while ((i71 < elems10)) {
      ((t332[i71]) = (( float ) (t_ocaml10[i71])));
      (i71 = (i71 + 1));
    }
    ((cuda_rec_field23.data) = t332);
    ((cuda_seq_temp1.inputs) = cuda_rec_field23);
    Tensor1 cuda_rec_field24;
    ((cuda_rec_field24.rank) = (Caml_ba_array_val(Field(Field(training_batches, i69), 1))->num_dims));
    if (((cuda_rec_field24.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field24.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field24.offset) = 0);
    ((cuda_rec_field24.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i72 = 0;
    int64_t n31 = (sizeof(int64_t));
    while ((i72 < (cuda_rec_field24.rank))) {
      (((cuda_rec_field24.dims)[i72]) = ((Caml_ba_array_val(Field(Field(training_batches, i69), 1))->dim)[i72]));
      (n31 = (n31 * ((Caml_ba_array_val(Field(Field(training_batches, i69), 1))->dim)[i72])));
      (i72 = (i72 + 1));
    }
    ((cuda_rec_field24.size) = n31);
    int64_t (*t333);
    hipMallocManaged((&t333), n31);
    GPU_UTILS_CHECK_CUDA_ERROR();
    int64_t (*t_ocaml11) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(training_batches, i69), 1)));
    int64_t i73 = 0;
    int64_t elems11 = (n31 / (sizeof(int64_t)));
    while ((i73 < elems11)) {
      ((t333[i73]) = (( int64_t ) (t_ocaml11[i73])));
      (i73 = (i73 + 1));
    }
    ((cuda_rec_field24.data) = t333);
    ((cuda_seq_temp1.correct_linear_outidxs) = cuda_rec_field24);
    (((gpu_tmp8.seq)[i69]) = cuda_seq_temp1);
    (i69 = (i69 + 1));
  }
  Seq2 gpu_tmp9;
  ((gpu_tmp9.len) = Wosize_val(validation_batches));
  hipMallocManaged((&(gpu_tmp9.seq)), (Wosize_val(validation_batches) * (sizeof(Rec3))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i74 = 0;
  while ((i74 < (gpu_tmp9.len))) {
    Rec3 cuda_seq_temp2;
    Tensor cuda_rec_field25;
    ((cuda_rec_field25.rank) = (Caml_ba_array_val(Field(Field(validation_batches, i74), 0))->num_dims));
    if (((cuda_rec_field25.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field25.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field25.offset) = 0);
    ((cuda_rec_field25.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i75 = 0;
    int64_t n32 = (sizeof(float));
    while ((i75 < (cuda_rec_field25.rank))) {
      (((cuda_rec_field25.dims)[i75]) = ((Caml_ba_array_val(Field(Field(validation_batches, i74), 0))->dim)[i75]));
      (n32 = (n32 * ((Caml_ba_array_val(Field(Field(validation_batches, i74), 0))->dim)[i75])));
      (i75 = (i75 + 1));
    }
    ((cuda_rec_field25.size) = n32);
    float (*t334);
    hipMallocManaged((&t334), n32);
    GPU_UTILS_CHECK_CUDA_ERROR();
    double (*t_ocaml12) = (( double (*) ) Caml_ba_data_val(Field(Field(validation_batches, i74), 0)));
    int64_t i76 = 0;
    int64_t elems12 = (n32 / (sizeof(float)));
    while ((i76 < elems12)) {
      ((t334[i76]) = (( float ) (t_ocaml12[i76])));
      (i76 = (i76 + 1));
    }
    ((cuda_rec_field25.data) = t334);
    ((cuda_seq_temp2.inputs) = cuda_rec_field25);
    Tensor1 cuda_rec_field26;
    ((cuda_rec_field26.rank) = (Caml_ba_array_val(Field(Field(validation_batches, i74), 1))->num_dims));
    if (((cuda_rec_field26.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field26.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field26.offset) = 0);
    ((cuda_rec_field26.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i77 = 0;
    int64_t n33 = (sizeof(int64_t));
    while ((i77 < (cuda_rec_field26.rank))) {
      (((cuda_rec_field26.dims)[i77]) = ((Caml_ba_array_val(Field(Field(validation_batches, i74), 1))->dim)[i77]));
      (n33 = (n33 * ((Caml_ba_array_val(Field(Field(validation_batches, i74), 1))->dim)[i77])));
      (i77 = (i77 + 1));
    }
    ((cuda_rec_field26.size) = n33);
    int64_t (*t335);
    hipMallocManaged((&t335), n33);
    GPU_UTILS_CHECK_CUDA_ERROR();
    int64_t (*t_ocaml13) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(validation_batches, i74), 1)));
    int64_t i78 = 0;
    int64_t elems13 = (n33 / (sizeof(int64_t)));
    while ((i78 < elems13)) {
      ((t335[i78]) = (( int64_t ) (t_ocaml13[i78])));
      (i78 = (i78 + 1));
    }
    ((cuda_rec_field26.data) = t335);
    ((cuda_seq_temp2.correct_linear_outidxs) = cuda_rec_field26);
    (((gpu_tmp9.seq)[i74]) = cuda_seq_temp2);
    (i74 = (i74 + 1));
  }
  hipMallocManaged((&t_state), (tensor_count * (sizeof(enum tensor_state))));
  GPU_UTILS_CHECK_CUDA_ERROR();
  ((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) = STATE_OK);
  ((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) = STATE_OK);
  ((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) = STATE_OK);
  int64_t i79 = 0;
  while ((i79 < ((gpu_tmp6.components).len))) {
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).b).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).w).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).b_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).w_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).in_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).out_bufs).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i79]).softmax_bufs).id)]) = STATE_OK);
    (i79 = (i79 + 1));
  }
  int64_t i80 = 0;
  while ((i80 < (gpu_tmp8.len))) {
    ((t_state[((((gpu_tmp8.seq)[i80]).inputs).id)]) = STATE_OK);
    ((t_state[((((gpu_tmp8.seq)[i80]).correct_linear_outidxs).id)]) = STATE_OK);
    (i80 = (i80 + 1));
  }
  int64_t i81 = 0;
  while ((i81 < (gpu_tmp9.len))) {
    ((t_state[((((gpu_tmp9.seq)[i81]).inputs).id)]) = STATE_OK);
    ((t_state[((((gpu_tmp9.seq)[i81]).correct_linear_outidxs).id)]) = STATE_OK);
    (i81 = (i81 + 1));
  }
  cuda_wrap(gpu_tmp, gpu_tmp1, gpu_tmp2, gpu_tmp3, gpu_tmp4, gpu_tmp5, gpu_tmp6, gpu_tmp7, gpu_tmp8, gpu_tmp9);
  if (((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) != STATE_OK)) {
    double (*t_ocaml14) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 1)));
    int64_t i82 = 0;
    int64_t elems14 = ((((gpu_tmp6.lossfn).in_grads).size) / (sizeof(float)));
    while ((i82 < elems14)) {
      ((t_ocaml14[i82]) = (( float ) ((((gpu_tmp6.lossfn).in_grads).data)[i82])));
      (i82 = (i82 + 1));
    }
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).in_grads).data));
    GPU_UTILS_CHECK_CUDA_ERROR();
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) != STATE_OK)) {
    double (*t_ocaml15) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 2)));
    int64_t i83 = 0;
    int64_t elems15 = ((((gpu_tmp6.lossfn).out_bufs).size) / (sizeof(float)));
    while ((i83 < elems15)) {
      ((t_ocaml15[i83]) = (( float ) ((((gpu_tmp6.lossfn).out_bufs).data)[i83])));
      (i83 = (i83 + 1));
    }
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).out_bufs).data));
    GPU_UTILS_CHECK_CUDA_ERROR();
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) != STATE_OK)) {
    double (*t_ocaml16) = (( double (*) ) Caml_ba_data_val(Field(Field(network6, 0), 3)));
    int64_t i84 = 0;
    int64_t elems16 = ((((gpu_tmp6.lossfn).softmax_bufs).size) / (sizeof(float)));
    while ((i84 < elems16)) {
      ((t_ocaml16[i84]) = (( float ) ((((gpu_tmp6.lossfn).softmax_bufs).data)[i84])));
      (i84 = (i84 + 1));
    }
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).softmax_bufs).data));
    GPU_UTILS_CHECK_CUDA_ERROR();
  } else {
    
  }
  int64_t i85 = 0;
  while ((i85 < ((gpu_tmp6.components).len))) {
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b).id)]) != STATE_OK)) {
      double (*t_ocaml17) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 0)));
      int64_t i86 = 0;
      int64_t elems17 = ((((((gpu_tmp6.components).seq)[i85]).b).size) / (sizeof(float)));
      while ((i86 < elems17)) {
        ((t_ocaml17[i86]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).b).data)[i86])));
        (i86 = (i86 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).b).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w).id)]) != STATE_OK)) {
      double (*t_ocaml18) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 1)));
      int64_t i87 = 0;
      int64_t elems18 = ((((((gpu_tmp6.components).seq)[i85]).w).size) / (sizeof(float)));
      while ((i87 < elems18)) {
        ((t_ocaml18[i87]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).w).data)[i87])));
        (i87 = (i87 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).w).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b_grads).id)]) != STATE_OK)) {
      double (*t_ocaml19) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 3)));
      int64_t i88 = 0;
      int64_t elems19 = ((((((gpu_tmp6.components).seq)[i85]).b_grads).size) / (sizeof(float)));
      while ((i88 < elems19)) {
        ((t_ocaml19[i88]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).b_grads).data)[i88])));
        (i88 = (i88 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).b_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).b_grads).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w_grads).id)]) != STATE_OK)) {
      double (*t_ocaml20) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 4)));
      int64_t i89 = 0;
      int64_t elems20 = ((((((gpu_tmp6.components).seq)[i85]).w_grads).size) / (sizeof(float)));
      while ((i89 < elems20)) {
        ((t_ocaml20[i89]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).w_grads).data)[i89])));
        (i89 = (i89 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).w_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).w_grads).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).in_grads).id)]) != STATE_OK)) {
      double (*t_ocaml21) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 5)));
      int64_t i90 = 0;
      int64_t elems21 = ((((((gpu_tmp6.components).seq)[i85]).in_grads).size) / (sizeof(float)));
      while ((i90 < elems21)) {
        ((t_ocaml21[i90]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).in_grads).data)[i90])));
        (i90 = (i90 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).in_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).in_grads).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).out_bufs).id)]) != STATE_OK)) {
      double (*t_ocaml22) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 6)));
      int64_t i91 = 0;
      int64_t elems22 = ((((((gpu_tmp6.components).seq)[i85]).out_bufs).size) / (sizeof(float)));
      while ((i91 < elems22)) {
        ((t_ocaml22[i91]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).out_bufs).data)[i91])));
        (i91 = (i91 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).out_bufs).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).out_bufs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).softmax_bufs).id)]) != STATE_OK)) {
      double (*t_ocaml23) = (( double (*) ) Caml_ba_data_val(Field(Field(Field(network6, 1), i85), 7)));
      int64_t i92 = 0;
      int64_t elems23 = ((((((gpu_tmp6.components).seq)[i85]).softmax_bufs).size) / (sizeof(float)));
      while ((i92 < elems23)) {
        ((t_ocaml23[i92]) = (( float ) ((((((gpu_tmp6.components).seq)[i85]).softmax_bufs).data)[i92])));
        (i92 = (i92 + 1));
      }
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i85]).softmax_bufs).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i85]).softmax_bufs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    (i85 = (i85 + 1));
  }
  hipFree(((gpu_tmp6.components).seq));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i93 = 0;
  while ((i93 < (gpu_tmp8.len))) {
    if (((t_state[((((gpu_tmp8.seq)[i93]).inputs).id)]) != STATE_OK)) {
      double (*t_ocaml24) = (( double (*) ) Caml_ba_data_val(Field(Field(training_batches, i93), 0)));
      int64_t i94 = 0;
      int64_t elems24 = (((((gpu_tmp8.seq)[i93]).inputs).size) / (sizeof(float)));
      while ((i94 < elems24)) {
        ((t_ocaml24[i94]) = (( float ) (((((gpu_tmp8.seq)[i93]).inputs).data)[i94])));
        (i94 = (i94 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i93]).inputs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp8.seq)[i93]).inputs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).id)]) != STATE_OK)) {
      int64_t (*t_ocaml25) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(training_batches, i93), 1)));
      int64_t i95 = 0;
      int64_t elems25 = (((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).size) / (sizeof(int64_t)));
      while ((i95 < elems25)) {
        ((t_ocaml25[i95]) = (( int64_t ) (((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).data)[i95])));
        (i95 = (i95 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp8.seq)[i93]).correct_linear_outidxs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    (i93 = (i93 + 1));
  }
  hipFree((gpu_tmp8.seq));
  GPU_UTILS_CHECK_CUDA_ERROR();
  int64_t i96 = 0;
  while ((i96 < (gpu_tmp9.len))) {
    if (((t_state[((((gpu_tmp9.seq)[i96]).inputs).id)]) != STATE_OK)) {
      double (*t_ocaml26) = (( double (*) ) Caml_ba_data_val(Field(Field(validation_batches, i96), 0)));
      int64_t i97 = 0;
      int64_t elems26 = (((((gpu_tmp9.seq)[i96]).inputs).size) / (sizeof(float)));
      while ((i97 < elems26)) {
        ((t_ocaml26[i97]) = (( float ) (((((gpu_tmp9.seq)[i96]).inputs).data)[i97])));
        (i97 = (i97 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i96]).inputs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp9.seq)[i96]).inputs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).id)]) != STATE_OK)) {
      int64_t (*t_ocaml27) = (( int64_t (*) ) Caml_ba_data_val(Field(Field(validation_batches, i96), 1)));
      int64_t i98 = 0;
      int64_t elems27 = (((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).size) / (sizeof(int64_t)));
      while ((i98 < elems27)) {
        ((t_ocaml27[i98]) = (( int64_t ) (((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).data)[i98])));
        (i98 = (i98 + 1));
      }
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp9.seq)[i96]).correct_linear_outidxs).data));
      GPU_UTILS_CHECK_CUDA_ERROR();
    } else {
      
    }
    (i96 = (i96 + 1));
  }
  hipFree((gpu_tmp9.seq));
  GPU_UTILS_CHECK_CUDA_ERROR();
  hipFree(t_state);
  GPU_UTILS_CHECK_CUDA_ERROR();
  CAMLreturn0;
}
extern "C" void vmcWpSQoKpn(value (*args), int argc) {
  return vjivEXkNw6z((args[0]), (args[1]), (args[2]), (args[3]), (args[4]), (args[5]), (args[6]), (args[7]), (args[8]), (args[9]));
}