#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <math.h>
#include <stddef.h>
#include <stdlib.h>
#include <stdio.h>
#include "caml/alloc.h"
#include "caml/bigarray.h"
#include "caml/memory.h"
#include "caml/mlvalues.h"
enum tensor_state {STATE_OK, STATE_CPU_INVALID, STATE_GPU_INVALID, STATE_RETURNED};
__managed__ enum tensor_state (*t_state);
typedef struct Tensor {int64_t id; double (*data); int64_t dims[3]; int64_t rank; int64_t offset; int64_t size;} Tensor;
typedef struct Seq {int64_t (*seq); int64_t len;} Seq;
typedef struct Tensor1 {int64_t id; int64_t (*data); int64_t dims[3]; int64_t rank; int64_t offset; int64_t size;} Tensor1;
typedef struct Rec {int64_t ty; Tensor w; Tensor b; Tensor w_grads; Tensor b_grads; Tensor out_bufs; Tensor in_grads; Tensor softmax_bufs;} Rec;
typedef struct Rec1 {int64_t ty; Tensor out_bufs; Tensor in_grads; Tensor softmax_bufs;} Rec1;
typedef struct Seq1 {Rec (*seq); int64_t len;} Seq1;
typedef struct Rec2 {Seq1 components; Rec1 lossfn;} Rec2;
typedef struct Rec3 {Tensor inputs; Tensor1 correct_linear_outidxs;} Rec3;
typedef struct Seq2 {Rec3 (*seq); int64_t len;} Seq2;
typedef struct Rec4 {double init_alpha; double init_lambda; double decay_alpha; double decay_lambda; int64_t batchsize; int64_t epochs; char printStatus; char evaluateBetweenEpochs; char evaluateBeforeFirstEpoch;} Rec4;
typedef struct Rec5 {double _0; double _1;} Rec5;
__host__ __device__ int64_t cartesian_to_linear_index0(int64_t dims1[3], int64_t rank1) {
  {
    int64_t t;
    (t = 0);
    return t;
  }
}
__host__ __device__ int64_t cartesian_to_linear_index1(int64_t dims1[3], int64_t rank1, int64_t i) {
  if ((rank1 == 3)) {
    {
      int64_t t1;
      (t1 = (((dims1[2]) * (dims1[1])) * i));
      return t1;
    }
  } else {
    if ((rank1 == 2)) {
      {
        int64_t t2;
        (t2 = ((dims1[1]) * i));
        return t2;
      }
    } else {
      {
        int64_t t3;
        (t3 = i);
        return t3;
      }
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index2(int64_t dims1[3], int64_t rank1, int64_t i1, int64_t i2) {
  if ((rank1 == 3)) {
    {
      int64_t t4;
      (t4 = ((((dims1[2]) * (dims1[1])) * i1) + ((dims1[2]) * i2)));
      return t4;
    }
  } else {
    if ((rank1 == 2)) {
      {
        int64_t t5;
        (t5 = (((dims1[1]) * i1) + i2));
        return t5;
      }
    } else {
      printf("Accessed tensor of rank %ld using 2 indices\n", rank1);
      {
        int64_t t6;
        (t6 = -1);
        return t6;
      }
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index3(int64_t dims1[3], int64_t rank1, int64_t i3, int64_t i4, int64_t i5) {
  if ((rank1 == 3)) {
    {
      int64_t t7;
      (t7 = (((((dims1[2]) * (dims1[1])) * i3) + ((dims1[2]) * i4)) + i5));
      return t7;
    }
  } else {
    printf("Accessed tensor of rank %ld using 3 indices\n", rank1);
    {
      int64_t t8;
      (t8 = -1);
      return t8;
    }
  }
}
__host__ __device__ int64_t cartesian_to_linear_index(int64_t dims1[3], int64_t rank1, Seq cartesian_idx) {
  if (((cartesian_idx.len) == 1)) {
    {
      int64_t t9;
      (t9 = cartesian_to_linear_index1(dims1, rank1, ((cartesian_idx.seq)[0])));
      return t9;
    }
  } else {
    if (((cartesian_idx.len) == 2)) {
      {
        int64_t t10;
        (t10 = cartesian_to_linear_index2(dims1, rank1, ((cartesian_idx.seq)[0]), ((cartesian_idx.seq)[1])));
        return t10;
      }
    } else {
      if (((cartesian_idx.len) == 3)) {
        {
          int64_t t11;
          (t11 = cartesian_to_linear_index3(dims1, rank1, ((cartesian_idx.seq)[0]), ((cartesian_idx.seq)[1]), ((cartesian_idx.seq)[2])));
          return t11;
        }
      } else {
        {
          int64_t t12;
          (t12 = cartesian_to_linear_index0(dims1, rank1));
          return t12;
        }
      }
    }
  }
}
__host__ __device__ Seq tensor_shape(int64_t dims2[3], int64_t rank2) {
  Seq s;
  ((s.seq) = dims2);
  ((s.len) = rank2);
  {
    Seq t13;
    (t13 = s);
    return t13;
  }
}
extern double exp(double);
__host__ __device__ double exp1(double x) {
  {
    double t14;
    (t14 = exp(x));
    return t14;
  }
}
__host__ __device__ int64_t t15(int64_t acc, int64_t e) {
  {
    int64_t t16;
    (t16 = (acc * e));
    return t16;
  }
}
__host__ __device__ int64_t foldl(int64_t acc_init, Seq s1) {
  int64_t acc1 = acc_init;
  int64_t i6 = 0;
  while ((i6 < (s1.len))) {
    (acc1 = t15(acc1, ((s1.seq)[i6])));
    (i6 = (i6 + 1));
  }
  return acc1;
}
__host__ __device__ int64_t tensorSize(Tensor t17) {
  Seq t18;
  (t18 = tensor_shape((t17.dims), (t17.rank)));
  {
    int64_t t19;
    (t19 = foldl(1, t18));
    return t19;
  }
}
__host__ __device__ double t20(Tensor w1, Tensor x1, int64_t n, int64_t x_offset, int64_t i7, double acc2, int64_t j) {
  int64_t t21;
  (t21 = (n * i7));
  int64_t t22;
  (t22 = (t21 + j));
  double t23;
  (t23 = ((w1.data)[t22]));
  int64_t t24;
  (t24 = (x_offset + j));
  double t25;
  (t25 = ((x1.data)[t24]));
  double t26;
  (t26 = (t23 * t25));
  {
    double t27;
    (t27 = (acc2 + t26));
    return t27;
  }
}
__host__ __device__ void iterfun(Tensor w1, Tensor x1, Tensor b1, Tensor z, int64_t m, int64_t n, int64_t i8) {
  int64_t s_idx;
  (s_idx = (i8 / m));
  int64_t x_offset;
  (x_offset = (s_idx * n));
  int64_t z_idx;
  (z_idx = i8);
  int64_t i7;
  (i7 = (i8 % m));
  double acc_init1;
  (acc_init1 = ((b1.data)[i7]));
  double v;
  {
    int64_t i9 = 0;
    double acc3 = acc_init1;
    while ((i9 < n)) {
      (acc3 = t20(w1, x1, n, x_offset, i7, acc3, i9));
      (i9 = (i9 + 1));
    }
    (v = acc3);
  }
  (((z.data)[z_idx]) = v);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel(int64_t n1, Tensor w1, Tensor x1, Tensor b1, Tensor z, int64_t m, int64_t n) {
  int64_t idx = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride = (gridDim.x * blockDim.x);
  while ((idx < n1)) {
    iterfun(w1, x1, b1, z, m, n, idx);
    (idx = (idx + stride));
  }
}
__host__ void tensorOpExn__z___Wx_B(int64_t s_max, Tensor w1, Tensor x1, Tensor b1, Tensor z) {
  Seq w_shape;
  (w_shape = tensor_shape((w1.dims), (w1.rank)));
  int64_t m;
  (m = ((w_shape.seq)[0]));
  int64_t n;
  (n = ((w_shape.seq)[1]));
  int64_t t28;
  (t28 = (s_max * m));
  {
    int64_t niterations = t28;
    int64_t tpb = 256;
    int64_t nblocks = (((niterations + (tpb * 10)) - 1) / (tpb * 10));
    loopKernel<<<nblocks, tpb>>>(t28, w1, x1, b1, z, m, n);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun1(Tensor x2, Tensor y, Tensor z1, int64_t m1, int64_t n2, int64_t m_x_n, int64_t i10) {
  int64_t s_idx1;
  (s_idx1 = (i10 / m_x_n));
  int64_t z_idx1;
  (z_idx1 = i10);
  int64_t i11;
  (i11 = (i10 % m_x_n));
  int64_t row;
  (row = (i11 / n2));
  int64_t col;
  (col = (i11 % n2));
  int64_t t29;
  (t29 = (s_idx1 * m1));
  int64_t x_idx;
  (x_idx = (row + t29));
  int64_t t30;
  (t30 = (s_idx1 * n2));
  int64_t y_idx;
  (y_idx = (col + t30));
  double t31;
  (t31 = ((x2.data)[x_idx]));
  double t32;
  (t32 = ((y.data)[y_idx]));
  double t33;
  (t33 = (t31 * t32));
  (((z1.data)[z_idx1]) = t33);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z1.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z1.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel1(int64_t n3, Tensor x2, Tensor y, Tensor z1, int64_t m1, int64_t n2, int64_t m_x_n) {
  int64_t idx1 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride1 = (gridDim.x * blockDim.x);
  while ((idx1 < n3)) {
    iterfun1(x2, y, z1, m1, n2, m_x_n, idx1);
    (idx1 = (idx1 + stride1));
  }
}
__host__ void tensorOpExn__z___x___y_T(int64_t s_max1, Tensor x2, Tensor y, Tensor z1) {
  Seq z_shape;
  (z_shape = tensor_shape((z1.dims), (z1.rank)));
  int64_t m1;
  (m1 = ((z_shape.seq)[1]));
  int64_t n2;
  (n2 = ((z_shape.seq)[2]));
  int64_t m_x_n;
  (m_x_n = (m1 * n2));
  int64_t t34;
  (t34 = (s_max1 * m_x_n));
  {
    int64_t niterations1 = t34;
    int64_t tpb1 = 256;
    int64_t nblocks1 = (((niterations1 + (tpb1 * 10)) - 1) / (tpb1 * 10));
    loopKernel1<<<nblocks1, tpb1>>>(t34, x2, y, z1, m1, n2, m_x_n);
    hipDeviceSynchronize();
  }
}
__host__ __device__ double t35(Tensor x3, Tensor w2, int64_t n4, int64_t x_offset1, int64_t j1, double acc4, int64_t i12) {
  int64_t t36;
  (t36 = (n4 * i12));
  int64_t t37;
  (t37 = (t36 + j1));
  double t38;
  (t38 = ((w2.data)[t37]));
  int64_t t39;
  (t39 = (x_offset1 + i12));
  double t40;
  (t40 = ((x3.data)[t39]));
  double t41;
  (t41 = (t38 * t40));
  {
    double t42;
    (t42 = (acc4 + t41));
    return t42;
  }
}
__host__ __device__ void iterfun2(Tensor x3, Tensor w2, Tensor z2, int64_t m2, int64_t n4, int64_t j2) {
  int64_t s_idx2;
  (s_idx2 = (j2 / n4));
  int64_t n_idx;
  (n_idx = (j2 % n4));
  int64_t z_idx2;
  (z_idx2 = j2);
  int64_t x_offset1;
  (x_offset1 = (s_idx2 * m2));
  int64_t j1;
  (j1 = n_idx);
  double v1;
  {
    int64_t i13 = 0;
    double acc5 = 0.;
    while ((i13 < m2)) {
      (acc5 = t35(x3, w2, n4, x_offset1, j1, acc5, i13));
      (i13 = (i13 + 1));
    }
    (v1 = acc5);
  }
  (((z2.data)[z_idx2]) = v1);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z2.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z2.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel2(int64_t n5, Tensor x3, Tensor w2, Tensor z2, int64_t m2, int64_t n4) {
  int64_t idx2 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride2 = (gridDim.x * blockDim.x);
  while ((idx2 < n5)) {
    iterfun2(x3, w2, z2, m2, n4, idx2);
    (idx2 = (idx2 + stride2));
  }
}
__host__ void tensorOpExn__z____x_T___W__T(int64_t s_max2, Tensor x3, Tensor w2, Tensor z2) {
  Seq w_shape1;
  (w_shape1 = tensor_shape((w2.dims), (w2.rank)));
  int64_t m2;
  (m2 = ((w_shape1.seq)[0]));
  int64_t n4;
  (n4 = ((w_shape1.seq)[1]));
  int64_t t43;
  (t43 = (s_max2 * n4));
  {
    int64_t niterations2 = t43;
    int64_t tpb2 = 256;
    int64_t nblocks2 = (((niterations2 + (tpb2 * 10)) - 1) / (tpb2 * 10));
    loopKernel2<<<nblocks2, tpb2>>>(t43, x3, w2, z2, m2, n4);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun3(Tensor x4, Tensor z3, int64_t i14) {
  double x_i;
  (x_i = ((x4.data)[i14]));
  char t44;
  (t44 = (x_i > 0.));
  double t45;
  if ((t44 == 1)) {
    (t45 = x_i);
  } else {
    (t45 = 0.);
  }
  (((z3.data)[i14]) = t45);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z3.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z3.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel3(int64_t n6, Tensor x4, Tensor z3) {
  int64_t idx3 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride3 = (gridDim.x * blockDim.x);
  while ((idx3 < n6)) {
    iterfun3(x4, z3, idx3);
    (idx3 = (idx3 + stride3));
  }
}
__host__ void tensorOpExn__z___ReLU_x_(int64_t s_max3, Tensor x4, Tensor z3) {
  Seq t46;
  (t46 = tensor_shape((x4.dims), (x4.rank)));
  int64_t s2;
  (s2 = ((t46.seq)[0]));
  int64_t t47;
  (t47 = tensorSize(x4));
  int64_t m3;
  (m3 = (t47 / s2));
  int64_t t48;
  (t48 = (s_max3 * m3));
  {
    int64_t niterations3 = t48;
    int64_t tpb3 = 256;
    int64_t nblocks3 = (((niterations3 + (tpb3 * 10)) - 1) / (tpb3 * 10));
    loopKernel3<<<nblocks3, tpb3>>>(t48, x4, z3);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun4(Tensor x5, Tensor z4, int64_t i15) {
  double x_i1;
  (x_i1 = ((x5.data)[i15]));
  double t49;
  (t49 = exp1(x_i1));
  (((z4.data)[i15]) = t49);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z4.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z4.id)]) = STATE_GPU_INVALID);
  #endif
}
__host__ __device__ double t50(Tensor z4, int64_t offset1, double acc6, int64_t i16) {
  int64_t t51;
  (t51 = (offset1 + i16));
  double t52;
  (t52 = ((z4.data)[t51]));
  {
    double t53;
    (t53 = (acc6 + t52));
    return t53;
  }
}
__host__ __device__ void iterfunSummarize(Tensor expsumbuf, Tensor z4, int64_t m4, int64_t s_idx3) {
  int64_t offset1;
  (offset1 = (s_idx3 * m4));
  double expsum;
  {
    int64_t i17 = 0;
    double acc7 = 0.;
    while ((i17 < m4)) {
      (acc7 = t50(z4, offset1, acc7, i17));
      (i17 = (i17 + 1));
    }
    (expsum = acc7);
  }
  (((expsumbuf.data)[s_idx3]) = expsum);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(expsumbuf.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(expsumbuf.id)]) = STATE_GPU_INVALID);
  #endif
}
__host__ __device__ void iterfunRegularize(Tensor expsumbuf, Tensor z4, int64_t m4, int64_t i18) {
  int64_t s_idx4;
  (s_idx4 = (i18 / m4));
  double expsum1;
  (expsum1 = ((expsumbuf.data)[s_idx4]));
  double z_i;
  (z_i = ((z4.data)[i18]));
  double t54;
  (t54 = (z_i / expsum1));
  (((z4.data)[i18]) = t54);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z4.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z4.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel4(int64_t n7, Tensor expsumbuf, Tensor z4, int64_t m4) {
  int64_t idx4 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride4 = (gridDim.x * blockDim.x);
  while ((idx4 < n7)) {
    iterfunRegularize(expsumbuf, z4, m4, idx4);
    (idx4 = (idx4 + stride4));
  }
}
__global__ void loopKernel5(int64_t n8, Tensor expsumbuf, Tensor z4, int64_t m4) {
  int64_t idx5 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride5 = (gridDim.x * blockDim.x);
  while ((idx5 < n8)) {
    iterfunSummarize(expsumbuf, z4, m4, idx5);
    (idx5 = (idx5 + stride5));
  }
}
__global__ void loopKernel6(int64_t n9, Tensor x5, Tensor z4) {
  int64_t idx6 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride6 = (gridDim.x * blockDim.x);
  while ((idx6 < n9)) {
    iterfun4(x5, z4, idx6);
    (idx6 = (idx6 + stride6));
  }
}
__host__ void tensorOpExn__z___SoftMax_x_(int64_t s_max4, Tensor x5, Tensor expsumbuf, Tensor z4) {
  Seq t55;
  (t55 = tensor_shape((x5.dims), (x5.rank)));
  int64_t s3;
  (s3 = ((t55.seq)[0]));
  int64_t t56;
  (t56 = tensorSize(x5));
  int64_t m4;
  (m4 = (t56 / s3));
  int64_t t57;
  (t57 = (s_max4 * m4));
  {
    int64_t niterations4 = t57;
    int64_t tpb4 = 256;
    int64_t nblocks4 = (((niterations4 + (tpb4 * 10)) - 1) / (tpb4 * 10));
    loopKernel6<<<nblocks4, tpb4>>>(t57, x5, z4);
    hipDeviceSynchronize();
  }
  {
    int64_t niterations5 = s_max4;
    int64_t tpb5 = 256;
    int64_t nblocks5 = (((niterations5 + (tpb5 * 10)) - 1) / (tpb5 * 10));
    loopKernel5<<<nblocks5, tpb5>>>(s_max4, expsumbuf, z4, m4);
    hipDeviceSynchronize();
  }
  int64_t t58;
  (t58 = (s_max4 * m4));
  {
    int64_t niterations6 = t58;
    int64_t tpb6 = 256;
    int64_t nblocks6 = (((niterations6 + (tpb6 * 10)) - 1) / (tpb6 * 10));
    loopKernel4<<<nblocks6, tpb6>>>(t58, expsumbuf, z4, m4);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun5(Tensor h, Tensor dldh, Tensor z5, int64_t i19) {
  double t59;
  (t59 = ((h.data)[i19]));
  char t60;
  (t60 = (t59 > 0.));
  double dhds_ii;
  if ((t60 == 1)) {
    (dhds_ii = 1.);
  } else {
    (dhds_ii = 0.);
  }
  double dldh_i;
  (dldh_i = ((dldh.data)[i19]));
  double t61;
  (t61 = (dhds_ii * dldh_i));
  (((z5.data)[i19]) = t61);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z5.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z5.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel7(int64_t n10, Tensor h, Tensor dldh, Tensor z5) {
  int64_t idx7 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride7 = (gridDim.x * blockDim.x);
  while ((idx7 < n10)) {
    iterfun5(h, dldh, z5, idx7);
    (idx7 = (idx7 + stride7));
  }
}
__host__ void tensorOpExn__z___d_dx_l_ReLU_x__(int64_t s_max5, Tensor h, Tensor dldh, Tensor z5) {
  Seq t62;
  (t62 = tensor_shape((h.dims), (h.rank)));
  int64_t s4;
  (s4 = ((t62.seq)[0]));
  int64_t t63;
  (t63 = tensorSize(h));
  int64_t m5;
  (m5 = (t63 / s4));
  int64_t t64;
  (t64 = (s_max5 * m5));
  {
    int64_t niterations7 = t64;
    int64_t tpb7 = 256;
    int64_t nblocks7 = (((niterations7 + (tpb7 * 10)) - 1) / (tpb7 * 10));
    loopKernel7<<<nblocks7, tpb7>>>(t64, h, dldh, z5);
    hipDeviceSynchronize();
  }
}
__host__ __device__ double t65(Tensor p, Tensor dldp, int64_t s_offset, int64_t i20, double p_i, double acc8, int64_t j3) {
  char t66;
  (t66 = (j3 == i20));
  double s_ij;
  if ((t66 == 1)) {
    double t67;
    (t67 = (p_i * p_i));
    (s_ij = (p_i - t67));
  } else {
    int64_t t68;
    (t68 = (s_offset + j3));
    double p_j;
    (p_j = ((p.data)[t68]));
    double t69;
    (t69 = (p_i * p_j));
    (s_ij = (-t69));
  }
  int64_t t70;
  (t70 = (s_offset + j3));
  double dldp_j;
  (dldp_j = ((dldp.data)[t70]));
  double t71;
  (t71 = (dldp_j * s_ij));
  {
    double t72;
    (t72 = (acc8 + t71));
    return t72;
  }
}
__host__ __device__ void iterfun6(Tensor p, Tensor dldp, Tensor z6, int64_t m6, int64_t i21) {
  int64_t s_idx5;
  (s_idx5 = (i21 / m6));
  int64_t s_offset;
  (s_offset = (s_idx5 * m6));
  int64_t i20;
  (i20 = (i21 % m6));
  int64_t t73;
  (t73 = (s_offset + i20));
  double p_i;
  (p_i = ((p.data)[t73]));
  double v2;
  {
    int64_t i22 = 0;
    double acc9 = 0.;
    while ((i22 < m6)) {
      (acc9 = t65(p, dldp, s_offset, i20, p_i, acc9, i22));
      (i22 = (i22 + 1));
    }
    (v2 = acc9);
  }
  int64_t t74;
  (t74 = (s_offset + i20));
  (((z6.data)[t74]) = v2);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z6.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z6.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel8(int64_t n11, Tensor p, Tensor dldp, Tensor z6, int64_t m6) {
  int64_t idx8 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride8 = (gridDim.x * blockDim.x);
  while ((idx8 < n11)) {
    iterfun6(p, dldp, z6, m6, idx8);
    (idx8 = (idx8 + stride8));
  }
}
__host__ void tensorOpExn__z___d_dx_l_SoftMax_x___(int64_t s_max6, Tensor p, Tensor dldp, Tensor z6) {
  Seq t75;
  (t75 = tensor_shape((p.dims), (p.rank)));
  int64_t s5;
  (s5 = ((t75.seq)[0]));
  int64_t t76;
  (t76 = tensorSize(p));
  int64_t m6;
  (m6 = (t76 / s5));
  int64_t t77;
  (t77 = (s_max6 * m6));
  {
    int64_t niterations8 = t77;
    int64_t tpb8 = 256;
    int64_t nblocks8 = (((niterations8 + (tpb8 * 10)) - 1) / (tpb8 * 10));
    loopKernel8<<<nblocks8, tpb8>>>(t77, p, dldp, z6, m6);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun7(Tensor x6, Tensor z7, int64_t i23) {
  double t78;
  (t78 = ((x6.data)[i23]));
  (((z7.data)[i23]) = t78);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z7.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z7.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel9(int64_t n12, Tensor x6, Tensor z7) {
  int64_t idx9 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride9 = (gridDim.x * blockDim.x);
  while ((idx9 < n12)) {
    iterfun7(x6, z7, idx9);
    (idx9 = (idx9 + stride9));
  }
}
__host__ void tensorOpExn__z___x(int64_t s_max7, Tensor x6, Tensor z7) {
  Seq t79;
  (t79 = tensor_shape((x6.dims), (x6.rank)));
  int64_t s6;
  (s6 = ((t79.seq)[0]));
  int64_t t80;
  (t80 = tensorSize(x6));
  int64_t m7;
  (m7 = (t80 / s6));
  int64_t t81;
  (t81 = (s_max7 * m7));
  {
    int64_t niterations9 = t81;
    int64_t tpb9 = 256;
    int64_t nblocks9 = (((niterations9 + (tpb9 * 10)) - 1) / (tpb9 * 10));
    loopKernel9<<<nblocks9, tpb9>>>(t81, x6, z7);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun8(double c, Tensor z8, int64_t i24) {
  double t82;
  (t82 = ((z8.data)[i24]));
  double t83;
  (t83 = (t82 * c));
  (((z8.data)[i24]) = t83);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z8.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z8.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel10(int64_t n13, double c, Tensor z8) {
  int64_t idx10 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride10 = (gridDim.x * blockDim.x);
  while ((idx10 < n13)) {
    iterfun8(c, z8, idx10);
    (idx10 = (idx10 + stride10));
  }
}
__host__ void tensorOpExn__z____scalar_c_(int64_t s_max8, double c, Tensor z8) {
  Seq t84;
  (t84 = tensor_shape((z8.dims), (z8.rank)));
  int64_t s7;
  (s7 = ((t84.seq)[0]));
  int64_t t85;
  (t85 = tensorSize(z8));
  int64_t m8;
  (m8 = (t85 / s7));
  int64_t t86;
  (t86 = (s_max8 * m8));
  {
    int64_t niterations10 = t86;
    int64_t tpb10 = 256;
    int64_t nblocks10 = (((niterations10 + (tpb10 * 10)) - 1) / (tpb10 * 10));
    loopKernel10<<<nblocks10, tpb10>>>(t86, c, z8);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun9(double c1, Tensor z9, int64_t i25) {
  (((z9.data)[i25]) = c1);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z9.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z9.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel11(int64_t n14, double c1, Tensor z9) {
  int64_t idx11 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride11 = (gridDim.x * blockDim.x);
  while ((idx11 < n14)) {
    iterfun9(c1, z9, idx11);
    (idx11 = (idx11 + stride11));
  }
}
__host__ void tensorOpExn__Z___scalar_c_(double c1, Tensor z9) {
  int64_t m9;
  (m9 = tensorSize(z9));
  {
    int64_t niterations11 = m9;
    int64_t tpb11 = 256;
    int64_t nblocks11 = (((niterations11 + (tpb11 * 10)) - 1) / (tpb11 * 10));
    loopKernel11<<<nblocks11, tpb11>>>(m9, c1, z9);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun10(Tensor x7, double c2, Tensor z10, int64_t x_offset2, int64_t i26) {
  double t87;
  (t87 = ((z10.data)[i26]));
  int64_t t88;
  (t88 = (i26 + x_offset2));
  double t89;
  (t89 = ((x7.data)[t88]));
  double t90;
  (t90 = (t89 * c2));
  double t91;
  (t91 = (t87 + t90));
  (((z10.data)[i26]) = t91);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z10.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z10.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel12(int64_t n15, Tensor x7, double c2, Tensor z10, int64_t x_offset2) {
  int64_t idx12 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride12 = (gridDim.x * blockDim.x);
  while ((idx12 < n15)) {
    iterfun10(x7, c2, z10, x_offset2, idx12);
    (idx12 = (idx12 + stride12));
  }
}
__host__ void tensorOpExn__Z____x___scalar_c_(int64_t s_idx6, Tensor x7, double c2, Tensor z10) {
  int64_t m10;
  (m10 = tensorSize(z10));
  int64_t x_offset2;
  (x_offset2 = (s_idx6 * m10));
  {
    int64_t niterations12 = m10;
    int64_t tpb12 = 256;
    int64_t nblocks12 = (((niterations12 + (tpb12 * 10)) - 1) / (tpb12 * 10));
    loopKernel12<<<nblocks12, tpb12>>>(m10, x7, c2, z10, x_offset2);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun11(Tensor x8, double c3, Tensor z11, int64_t z_offset, int64_t i27) {
  int64_t z_idx3;
  (z_idx3 = (i27 + z_offset));
  double t92;
  (t92 = ((z11.data)[z_idx3]));
  double t93;
  (t93 = ((x8.data)[i27]));
  double t94;
  (t94 = (t93 * c3));
  double t95;
  (t95 = (t92 + t94));
  (((z11.data)[z_idx3]) = t95);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z11.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z11.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel13(int64_t n16, Tensor x8, double c3, Tensor z11, int64_t z_offset) {
  int64_t idx13 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride13 = (gridDim.x * blockDim.x);
  while ((idx13 < n16)) {
    iterfun11(x8, c3, z11, z_offset, idx13);
    (idx13 = (idx13 + stride13));
  }
}
__host__ void tensorOpExn__z____X___scalar_c_(int64_t s_idx7, Tensor x8, double c3, Tensor z11) {
  int64_t m11;
  (m11 = tensorSize(x8));
  int64_t z_offset;
  (z_offset = (s_idx7 * m11));
  {
    int64_t niterations13 = m11;
    int64_t tpb13 = 256;
    int64_t nblocks13 = (((niterations13 + (tpb13 * 10)) - 1) / (tpb13 * 10));
    loopKernel13<<<nblocks13, tpb13>>>(m11, x8, c3, z11, z_offset);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void iterfun12(Tensor1 y1, double c4, Tensor z12, int64_t m12, int64_t i28) {
  int64_t idx14;
  (idx14 = ((y1.data)[i28]));
  int64_t offset2;
  (offset2 = (i28 * m12));
  int64_t z_idx4;
  (z_idx4 = (idx14 + offset2));
  double t96;
  (t96 = ((z12.data)[z_idx4]));
  double t97;
  (t97 = (t96 + c4));
  (((z12.data)[z_idx4]) = t97);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z12.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z12.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel14(int64_t n17, Tensor1 y1, double c4, Tensor z12, int64_t m12) {
  int64_t idx15 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride14 = (gridDim.x * blockDim.x);
  while ((idx15 < n17)) {
    iterfun12(y1, c4, z12, m12, idx15);
    (idx15 = (idx15 + stride14));
  }
}
__host__ void tensorOpExp__z____1_Hot_y____scalar_c_(int64_t s_max9, Tensor1 y1, double c4, Tensor z12) {
  Seq t98;
  (t98 = tensor_shape((z12.dims), (z12.rank)));
  int64_t m12;
  (m12 = ((t98.seq)[1]));
  {
    int64_t niterations14 = s_max9;
    int64_t tpb14 = 256;
    int64_t nblocks14 = (((niterations14 + (tpb14 * 10)) - 1) / (tpb14 * 10));
    loopKernel14<<<nblocks14, tpb14>>>(s_max9, y1, c4, z12, m12);
    hipDeviceSynchronize();
  }
}
__host__ __device__ void t99(Tensor z13, int64_t offset3, int64_t idx16, double v3, int64_t j4) {
  char t100;
  (t100 = (j4 == idx16));
  if ((t100 == 1)) {
    int64_t t101;
    (t101 = (j4 + offset3));
    (((z13.data)[t101]) = v3);
  } else {
    int64_t t102;
    (t102 = (j4 + offset3));
    (((z13.data)[t102]) = 0.);
  }
}
__host__ __device__ void iterfun13(Tensor1 y2, Tensor x9, Tensor z13, int64_t m13, int64_t i29) {
  int64_t offset3;
  (offset3 = (i29 * m13));
  int64_t idx16;
  (idx16 = ((y2.data)[i29]));
  int64_t t103;
  (t103 = (idx16 + offset3));
  double t104;
  (t104 = ((x9.data)[t103]));
  double t105;
  (t105 = (1. / t104));
  double v3;
  (v3 = (-t105));
  {
    int64_t i30 = 0;
    while ((i30 < m13)) {
      t99(z13, offset3, idx16, v3, i30);
      (i30 = (i30 + 1));
    }
  }
}
__global__ void loopKernel15(int64_t n18, Tensor1 y2, Tensor x9, Tensor z13, int64_t m13) {
  int64_t idx17 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride15 = (gridDim.x * blockDim.x);
  while ((idx17 < n18)) {
    iterfun13(y2, x9, z13, m13, idx17);
    (idx17 = (idx17 + stride15));
  }
}
__host__ void tensorOpExn__z___1_Hot_y____scalar__1__x_T___1_Hot_y___(int64_t s_max10, Tensor1 y2, Tensor x9, Tensor z13) {
  Seq t106;
  (t106 = tensor_shape((x9.dims), (x9.rank)));
  int64_t s8;
  (s8 = ((t106.seq)[0]));
  Seq t107;
  (t107 = tensor_shape((x9.dims), (x9.rank)));
  int64_t m13;
  (m13 = ((t107.seq)[1]));
  {
    int64_t niterations15 = s_max10;
    int64_t tpb15 = 256;
    int64_t nblocks15 = (((niterations15 + (tpb15 * 10)) - 1) / (tpb15 * 10));
    loopKernel15<<<nblocks15, tpb15>>>(s_max10, y2, x9, z13, m13);
    hipDeviceSynchronize();
  }
}
__host__ __device__ double t108(Tensor z14, int64_t m14, int64_t i31, double acc10, int64_t j5) {
  int64_t s_idx8;
  (s_idx8 = (j5 + 1));
  int64_t s_offset1;
  (s_offset1 = (s_idx8 * m14));
  int64_t t109;
  (t109 = (s_offset1 + i31));
  double t110;
  (t110 = ((z14.data)[t109]));
  {
    double t111;
    (t111 = (acc10 + t110));
    return t111;
  }
}
__host__ __device__ void iterfun14(Tensor z14, int64_t s9, int64_t m14, int64_t i31) {
  int64_t t112;
  (t112 = (s9 - 1));
  double v4;
  {
    int64_t i32 = 0;
    double acc11 = ((z14.data)[i31]);
    while ((i32 < t112)) {
      (acc11 = t108(z14, m14, i31, acc11, i32));
      (i32 = (i32 + 1));
    }
    (v4 = acc11);
  }
  (((z14.data)[i31]) = v4);
  #if ((defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
  ((t_state[(z14.id)]) = STATE_CPU_INVALID);
  #else
  ((t_state[(z14.id)]) = STATE_GPU_INVALID);
  #endif
}
__global__ void loopKernel16(int64_t n19, Tensor z14, int64_t s9, int64_t m14) {
  int64_t idx18 = ((blockDim.x * blockIdx.x) + threadIdx.x);
  int64_t stride16 = (gridDim.x * blockDim.x);
  while ((idx18 < n19)) {
    iterfun14(z14, s9, m14, idx18);
    (idx18 = (idx18 + stride16));
  }
}
__host__ void tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(Tensor z14) {
  Seq t113;
  (t113 = tensor_shape((z14.dims), (z14.rank)));
  int64_t s9;
  (s9 = ((t113.seq)[0]));
  int64_t t114;
  (t114 = tensorSize(z14));
  int64_t m14;
  (m14 = (t114 / s9));
  {
    int64_t niterations16 = m14;
    int64_t tpb16 = 256;
    int64_t nblocks16 = (((niterations16 + (tpb16 * 10)) - 1) / (tpb16 * 10));
    loopKernel16<<<nblocks16, tpb16>>>(m14, z14, s9, m14);
    hipDeviceSynchronize();
  }
}
__host__ __device__ Tensor nnComponentOutBufs(Rec comp) {
  Tensor X = (comp.out_bufs);
  {
    Tensor t115;
    (t115 = X);
    return t115;
  }
}
__host__ __device__ Tensor nnComponentApplyExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max11, Tensor inputs1, Rec comp1) {
  int64_t ty1;
  int64_t X1 = (comp1.ty);
  (ty1 = X1);
  char t116;
  (t116 = (ty1 == nnCompType_FullyConnected));
  if ((t116 == 1)) {
    Tensor t117;
    Tensor X2 = (comp1.w);
    (t117 = X2);
    Tensor t118;
    Tensor X3 = (comp1.b);
    (t118 = X3);
    Tensor t119;
    Tensor X4 = (comp1.out_bufs);
    (t119 = X4);
    tensorOpExn__z___Wx_B(s_max11, t117, inputs1, t118, t119);
    Tensor X5 = (comp1.out_bufs);
    {
      Tensor t120;
      (t120 = X5);
      return t120;
    }
  } else {
    char t121;
    (t121 = (ty1 == nnCompType_ReLU));
    if ((t121 == 1)) {
      Tensor t122;
      Tensor X6 = (comp1.out_bufs);
      (t122 = X6);
      tensorOpExn__z___ReLU_x_(s_max11, inputs1, t122);
      Tensor X7 = (comp1.out_bufs);
      {
        Tensor t123;
        (t123 = X7);
        return t123;
      }
    } else {
      char t124;
      (t124 = (ty1 == nnCompType_SoftMax));
      if ((t124 == 1)) {
        Tensor t125;
        Tensor X8 = (comp1.softmax_bufs);
        (t125 = X8);
        Tensor t126;
        Tensor X9 = (comp1.out_bufs);
        (t126 = X9);
        tensorOpExn__z___SoftMax_x_(s_max11, inputs1, t125, t126);
        Tensor X10 = (comp1.out_bufs);
        {
          Tensor t127;
          (t127 = X10);
          return t127;
        }
      } else {
        Tensor X11 = (comp1.out_bufs);
        {
          Tensor t128;
          (t128 = X11);
          return t128;
        }
      }
    }
  }
}
__host__ __device__ Tensor nnComponentBackpropExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max12, Tensor comp_inputs, Tensor output_grads, Rec comp2) {
  int64_t ty2;
  int64_t X12 = (comp2.ty);
  (ty2 = X12);
  char t129;
  (t129 = (ty2 == nnCompType_FullyConnected));
  if ((t129 == 1)) {
    Tensor t130;
    Tensor X13 = (comp2.b_grads);
    (t130 = X13);
    tensorOpExn__z___x(s_max12, output_grads, t130);
    Tensor t131;
    Tensor X14 = (comp2.w_grads);
    (t131 = X14);
    tensorOpExn__z___x___y_T(s_max12, output_grads, comp_inputs, t131);
    Tensor t132;
    Tensor X15 = (comp2.w);
    (t132 = X15);
    Tensor t133;
    Tensor X16 = (comp2.in_grads);
    (t133 = X16);
    tensorOpExn__z____x_T___W__T(s_max12, output_grads, t132, t133);
    Tensor X17 = (comp2.in_grads);
    {
      Tensor t134;
      (t134 = X17);
      return t134;
    }
  } else {
    char t135;
    (t135 = (ty2 == nnCompType_ReLU));
    if ((t135 == 1)) {
      Tensor t136;
      Tensor X18 = (comp2.out_bufs);
      (t136 = X18);
      Tensor t137;
      Tensor X19 = (comp2.in_grads);
      (t137 = X19);
      tensorOpExn__z___d_dx_l_ReLU_x__(s_max12, t136, output_grads, t137);
      Tensor X20 = (comp2.in_grads);
      {
        Tensor t138;
        (t138 = X20);
        return t138;
      }
    } else {
      char t139;
      (t139 = (ty2 == nnCompType_SoftMax));
      if ((t139 == 1)) {
        Tensor t140;
        Tensor X21 = (comp2.out_bufs);
        (t140 = X21);
        Tensor t141;
        Tensor X22 = (comp2.in_grads);
        (t141 = X22);
        tensorOpExn__z___d_dx_l_SoftMax_x___(s_max12, t140, output_grads, t141);
        Tensor X23 = (comp2.in_grads);
        {
          Tensor t142;
          (t142 = X23);
          return t142;
        }
      } else {
        Tensor X24 = (comp2.in_grads);
        {
          Tensor t143;
          (t143 = X24);
          return t143;
        }
      }
    }
  }
}
__host__ __device__ void nnComponent_TEMP_SetGradients(int64_t nnCompType_FullyConnected, double scalar, Rec comp3) {
  int64_t ty3;
  int64_t X25 = (comp3.ty);
  (ty3 = X25);
  char t144;
  (t144 = (ty3 == nnCompType_FullyConnected));
  if ((t144 == 1)) {
    Tensor t145;
    Tensor X26 = (comp3.w_grads);
    (t145 = X26);
    tensorOpExn__Z___scalar_c_(scalar, t145);
    Tensor t146;
    Tensor X27 = (comp3.b_grads);
    (t146 = X27);
    tensorOpExn__Z___scalar_c_(scalar, t146);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ReduceGradients(int64_t nnCompType_FullyConnected, Rec comp4) {
  int64_t ty4;
  int64_t X28 = (comp4.ty);
  (ty4 = X28);
  char t147;
  (t147 = (ty4 == nnCompType_FullyConnected));
  if ((t147 == 1)) {
    Tensor t148;
    Tensor X29 = (comp4.w_grads);
    (t148 = X29);
    tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(t148);
    Tensor t149;
    Tensor X30 = (comp4.b_grads);
    (t149 = X30);
    tensorOpExn__Dim1Reduce_z__dst___z_0__op_____(t149);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ScaleGradients(int64_t nnCompType_FullyConnected, double scalar1, Rec comp5) {
  int64_t ty5;
  int64_t X31 = (comp5.ty);
  (ty5 = X31);
  char t150;
  (t150 = (ty5 == nnCompType_FullyConnected));
  if ((t150 == 1)) {
    Tensor t151;
    Tensor X32 = (comp5.w_grads);
    (t151 = X32);
    tensorOpExn__z____scalar_c_(1, scalar1, t151);
    Tensor t152;
    Tensor X33 = (comp5.b_grads);
    (t152 = X33);
    tensorOpExn__z____scalar_c_(1, scalar1, t152);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_ApplyGradients(int64_t nnCompType_FullyConnected, double scalar2, Rec comp6) {
  int64_t ty6;
  int64_t X34 = (comp6.ty);
  (ty6 = X34);
  char t153;
  (t153 = (ty6 == nnCompType_FullyConnected));
  if ((t153 == 1)) {
    Tensor t154;
    Tensor X35 = (comp6.w_grads);
    (t154 = X35);
    Tensor t155;
    Tensor X36 = (comp6.w);
    (t155 = X36);
    tensorOpExn__Z____x___scalar_c_(0, t154, scalar2, t155);
    Tensor t156;
    Tensor X37 = (comp6.b_grads);
    (t156 = X37);
    Tensor t157;
    Tensor X38 = (comp6.b);
    (t157 = X38);
    tensorOpExn__Z____x___scalar_c_(0, t156, scalar2, t157);
  } else {
    ;
  }
}
__host__ __device__ void nnComponent_TEMP_L2Regularize(int64_t nnCompType_FullyConnected, double scalar3, Rec comp7) {
  int64_t ty7;
  int64_t X39 = (comp7.ty);
  (ty7 = X39);
  char t158;
  (t158 = (ty7 == nnCompType_FullyConnected));
  if ((t158 == 1)) {
    Tensor t159;
    Tensor X40 = (comp7.w);
    (t159 = X40);
    Tensor t160;
    Tensor X41 = (comp7.w_grads);
    (t160 = X41);
    tensorOpExn__z____X___scalar_c_(0, t159, scalar3, t160);
    Tensor t161;
    Tensor X42 = (comp7.b);
    (t161 = X42);
    Tensor t162;
    Tensor X43 = (comp7.b_grads);
    (t162 = X43);
    tensorOpExn__z____X___scalar_c_(0, t161, scalar3, t162);
  } else {
    ;
  }
}
__host__ __device__ void nnComponentZeroGrad(int64_t nnCompType_FullyConnected, Rec comp8) {
  nnComponent_TEMP_SetGradients(nnCompType_FullyConnected, 0., comp8);
}
__host__ __device__ Tensor nnLossFunctionBackpropExn(int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, int64_t s_max13, Tensor inputs2, Tensor1 expecteds, Rec1 lossfn1) {
  int64_t ty8;
  int64_t X44 = (lossfn1.ty);
  (ty8 = X44);
  char t163;
  (t163 = (ty8 == nnLossfnType_CrossEntropyLoss));
  if ((t163 == 1)) {
    Tensor t164;
    Tensor X45 = (lossfn1.in_grads);
    (t164 = X45);
    tensorOpExn__z___1_Hot_y____scalar__1__x_T___1_Hot_y___(s_max13, expecteds, inputs2, t164);
    Tensor X46 = (lossfn1.in_grads);
    {
      Tensor t165;
      (t165 = X46);
      return t165;
    }
  } else {
    char t166;
    (t166 = (ty8 == nnLossfnType_SoftMaxCrossEntropyLoss));
    if ((t166 == 1)) {
      Tensor t167;
      Tensor X47 = (lossfn1.softmax_bufs);
      (t167 = X47);
      Tensor t168;
      Tensor X48 = (lossfn1.in_grads);
      (t168 = X48);
      tensorOpExn__z___SoftMax_x_(s_max13, inputs2, t167, t168);
      double t169;
      (t169 = (-1.));
      Tensor t170;
      Tensor X49 = (lossfn1.in_grads);
      (t170 = X49);
      tensorOpExp__z____1_Hot_y____scalar_c_(s_max13, expecteds, t169, t170);
      Tensor X50 = (lossfn1.in_grads);
      {
        Tensor t171;
        (t171 = X50);
        return t171;
      }
    } else {
      Tensor X51 = (lossfn1.in_grads);
      {
        Tensor t172;
        (t172 = X51);
        return t172;
      }
    }
  }
}
__host__ __device__ int64_t t173(int64_t nnCompType_FullyConnected, int64_t x10, Rec comp9) {
  nnComponentZeroGrad(nnCompType_FullyConnected, comp9);
  {
    int64_t t174;
    (t174 = 0);
    return t174;
  }
}
__host__ __device__ int64_t foldl1(int64_t acc_init2, Seq1 s10, int64_t nnCompType_FullyConnected) {
  int64_t acc12 = acc_init2;
  int64_t i33 = 0;
  while ((i33 < (s10.len))) {
    (acc12 = t173(nnCompType_FullyConnected, acc12, ((s10.seq)[i33])));
    (i33 = (i33 + 1));
  }
  return acc12;
}
__host__ __device__ void nnZeroGrad(int64_t nnCompType_FullyConnected, Rec2 network) {
  Seq1 t175;
  Seq1 X52 = (network.components);
  (t175 = X52);
  int64_t _;
  (_ = foldl1(0, t175, nnCompType_FullyConnected));
  ;
}
__host__ __device__ Tensor t176(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max14, Tensor prevouts, Rec comp10) {
  {
    Tensor t177;
    (t177 = nnComponentApplyExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max14, prevouts, comp10));
    return t177;
  }
}
__host__ __device__ Tensor foldl2(Tensor acc_init3, Seq1 s11, int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t s_max14) {
  Tensor acc13 = acc_init3;
  int64_t i34 = 0;
  while ((i34 < (s11.len))) {
    (acc13 = t176(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max14, acc13, ((s11.seq)[i34])));
    (i34 = (i34 + 1));
  }
  return acc13;
}
__host__ __device__ Tensor nnEvalExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, Rec2 network1, Tensor inputs3) {
  Seq t178;
  (t178 = tensor_shape((inputs3.dims), (inputs3.rank)));
  int64_t s_max14;
  (s_max14 = ((t178.seq)[0]));
  Seq1 t179;
  Seq1 X53 = (network1.components);
  (t179 = X53);
  {
    Tensor t180;
    (t180 = foldl2(inputs3, t179, nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max14));
    return t180;
  }
}
__host__ __device__ Tensor t181(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, Rec2 network2, int64_t s_max15, int64_t n_components, Tensor out_grads, int64_t i35) {
  int64_t t182;
  (t182 = (i35 + 2));
  int64_t idx19;
  (idx19 = (n_components - t182));
  int64_t previdx;
  (previdx = (idx19 - 1));
  Seq1 t183;
  Seq1 X54 = (network2.components);
  (t183 = X54);
  Rec comp11;
  (comp11 = ((t183.seq)[idx19]));
  Seq1 t184;
  Seq1 X55 = (network2.components);
  (t184 = X55);
  Rec t185;
  (t185 = ((t184.seq)[previdx]));
  Tensor in_bufs;
  (in_bufs = nnComponentOutBufs(t185));
  {
    Tensor t186;
    (t186 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, in_bufs, out_grads, comp11));
    return t186;
  }
}
__host__ __device__ void nnBackpropExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network2, Rec3 batch) {
  Tensor t187;
  Tensor X56 = (batch.inputs);
  (t187 = X56);
  Seq t188;
  (t188 = tensor_shape((t187.dims), (t187.rank)));
  int64_t s_max15;
  (s_max15 = ((t188.seq)[0]));
  Tensor t189;
  Tensor X57 = (batch.inputs);
  (t189 = X57);
  Tensor outputs;
  (outputs = nnEvalExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, network2, t189));
  Tensor1 t190;
  Tensor1 X58 = (batch.correct_linear_outidxs);
  (t190 = X58);
  Rec1 t191;
  Rec1 X59 = (network2.lossfn);
  (t191 = X59);
  Tensor lossgrads;
  (lossgrads = nnLossFunctionBackpropExn(nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, s_max15, outputs, t190, t191));
  Seq1 t192;
  Seq1 X60 = (network2.components);
  (t192 = X60);
  int64_t n_components;
  (n_components = (t192.len));
  char t193;
  (t193 = (n_components == 0));
  if ((t193 == 1)) {
    ;
  } else {
    char t194;
    (t194 = (n_components == 1));
    if ((t194 == 1)) {
      Seq1 t195;
      Seq1 X61 = (network2.components);
      (t195 = X61);
      Rec lastcomp;
      (lastcomp = ((t195.seq)[0]));
      Tensor t196;
      Tensor X62 = (batch.inputs);
      (t196 = X62);
      Tensor _2;
      (_2 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, t196, lossgrads, lastcomp));
      ;
    } else {
      Seq1 t197;
      Seq1 X63 = (network2.components);
      (t197 = X63);
      int64_t t198;
      (t198 = (n_components - 1));
      Rec lastcomp1;
      (lastcomp1 = ((t197.seq)[t198]));
      Seq1 t199;
      Seq1 X64 = (network2.components);
      (t199 = X64);
      int64_t t200;
      (t200 = (n_components - 2));
      Rec t201;
      (t201 = ((t199.seq)[t200]));
      Tensor lastcomp_in_bufs;
      (lastcomp_in_bufs = nnComponentOutBufs(t201));
      Tensor lastcomp_in_grads;
      (lastcomp_in_grads = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, lastcomp_in_bufs, lossgrads, lastcomp1));
      int64_t t202;
      (t202 = (n_components - 2));
      Tensor firstcomp_out_grads;
      {
        int64_t i36 = 0;
        Tensor acc14 = lastcomp_in_grads;
        while ((i36 < t202)) {
          (acc14 = t181(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, network2, s_max15, n_components, acc14, i36));
          (i36 = (i36 + 1));
        }
        (firstcomp_out_grads = acc14);
      }
      Seq1 t203;
      Seq1 X65 = (network2.components);
      (t203 = X65);
      Rec firstcomp;
      (firstcomp = ((t203.seq)[0]));
      Tensor firstcomp_in_bufs;
      Tensor X66 = (batch.inputs);
      (firstcomp_in_bufs = X66);
      Tensor _3;
      (_3 = nnComponentBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, s_max15, firstcomp_in_bufs, firstcomp_out_grads, firstcomp));
      ;
    }
  }
}
__host__ __device__ int64_t t204(int64_t nnCompType_FullyConnected, double batchsize_regularizer, int64_t x11, Rec comp12) {
  nnComponent_TEMP_ReduceGradients(nnCompType_FullyConnected, comp12);
  nnComponent_TEMP_ScaleGradients(nnCompType_FullyConnected, batchsize_regularizer, comp12);
  {
    int64_t t205;
    (t205 = 0);
    return t205;
  }
}
__host__ __device__ int64_t t206(int64_t nnCompType_FullyConnected, double lambda, int64_t x12, Rec comp13) {
  double t207;
  (t207 = (2. * lambda));
  nnComponent_TEMP_L2Regularize(nnCompType_FullyConnected, t207, comp13);
  {
    int64_t t208;
    (t208 = 0);
    return t208;
  }
}
__host__ __device__ int64_t t209(int64_t nnCompType_FullyConnected, double alpha, int64_t x13, Rec comp14) {
  double t210;
  (t210 = (-alpha));
  nnComponent_TEMP_ApplyGradients(nnCompType_FullyConnected, t210, comp14);
  {
    int64_t t211;
    (t211 = 0);
    return t211;
  }
}
__host__ __device__ int64_t foldl3(int64_t acc_init4, Seq1 s12, int64_t nnCompType_FullyConnected, double alpha) {
  int64_t acc15 = acc_init4;
  int64_t i37 = 0;
  while ((i37 < (s12.len))) {
    (acc15 = t209(nnCompType_FullyConnected, alpha, acc15, ((s12.seq)[i37])));
    (i37 = (i37 + 1));
  }
  return acc15;
}
__host__ __device__ int64_t foldl4(int64_t acc_init5, Seq1 s13, int64_t nnCompType_FullyConnected, double lambda) {
  int64_t acc16 = acc_init5;
  int64_t i38 = 0;
  while ((i38 < (s13.len))) {
    (acc16 = t206(nnCompType_FullyConnected, lambda, acc16, ((s13.seq)[i38])));
    (i38 = (i38 + 1));
  }
  return acc16;
}
__host__ __device__ int64_t foldl5(int64_t acc_init6, Seq1 s14, int64_t nnCompType_FullyConnected, double batchsize_regularizer) {
  int64_t acc17 = acc_init6;
  int64_t i39 = 0;
  while ((i39 < (s14.len))) {
    (acc17 = t204(nnCompType_FullyConnected, batchsize_regularizer, acc17, ((s14.seq)[i39])));
    (i39 = (i39 + 1));
  }
  return acc17;
}
__host__ __device__ void nnGradientDescentExn(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec2 network3, double alpha, double lambda, Rec3 batch1) {
  nnZeroGrad(nnCompType_FullyConnected, network3);
  Tensor t212;
  Tensor X67 = (batch1.inputs);
  (t212 = X67);
  Seq t213;
  (t213 = tensor_shape((t212.dims), (t212.rank)));
  int64_t batchsize1;
  (batchsize1 = ((t213.seq)[0]));
  nnBackpropExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network3, batch1);
  double t214;
  (t214 = (( double ) batchsize1));
  double batchsize_regularizer;
  (batchsize_regularizer = (1. / t214));
  Seq1 t215;
  Seq1 X68 = (network3.components);
  (t215 = X68);
  int64_t _4;
  (_4 = foldl5(0, t215, nnCompType_FullyConnected, batchsize_regularizer));
  char t216;
  (t216 = (lambda == 0.));
  if ((t216 == 1)) {
    ;
  } else {
    Seq1 t217;
    Seq1 X69 = (network3.components);
    (t217 = X69);
    int64_t _5;
    (_5 = foldl4(0, t217, nnCompType_FullyConnected, lambda));
    ;
  }
  Seq1 t218;
  Seq1 X70 = (network3.components);
  (t218 = X70);
  int64_t _6;
  (_6 = foldl3(0, t218, nnCompType_FullyConnected, alpha));
  ;
}
__host__ __device__ int64_t t219(int64_t acc18, int64_t e1) {
  {
    int64_t t220;
    (t220 = (acc18 * e1));
    return t220;
  }
}
__host__ __device__ int64_t foldl6(int64_t acc_init7, Seq s15) {
  int64_t acc19 = acc_init7;
  int64_t i40 = 0;
  while ((i40 < (s15.len))) {
    (acc19 = t219(acc19, ((s15.seq)[i40])));
    (i40 = (i40 + 1));
  }
  return acc19;
}
__host__ __device__ int64_t tensorSize1(Tensor t221) {
  Seq t222;
  (t222 = tensor_shape((t221.dims), (t221.rank)));
  {
    int64_t t223;
    (t223 = foldl6(1, t222));
    return t223;
  }
}
__host__ __device__ int64_t t224(Tensor t225, int64_t s_offset2, int64_t cand_idx, int64_t idx20) {
  int64_t t226;
  (t226 = (idx20 + s_offset2));
  double t227;
  (t227 = ((t225.data)[t226]));
  int64_t t228;
  (t228 = (cand_idx + s_offset2));
  double t229;
  (t229 = ((t225.data)[t228]));
  char t230;
  (t230 = (t227 > t229));
  if ((t230 == 1)) {
    {
      int64_t t231;
      (t231 = idx20);
      return t231;
    }
  } else {
    {
      int64_t t232;
      (t232 = cand_idx);
      return t232;
    }
  }
}
__host__ __device__ int64_t tensorLinearMaxIdx(int64_t s_idx9, Tensor t225) {
  Seq t233;
  (t233 = tensor_shape((t225.dims), (t225.rank)));
  int64_t s16;
  (s16 = ((t233.seq)[0]));
  int64_t t234;
  (t234 = tensorSize1(t225));
  int64_t size1;
  (size1 = (t234 / s16));
  int64_t s_offset2;
  (s_offset2 = (s_idx9 * size1));
  int64_t t235;
  (t235 = (size1 - 1));
  {
    int64_t t236;
    {
      int64_t i41 = 0;
      int64_t acc20 = (size1 - 1);
      while ((i41 < t235)) {
        (acc20 = t224(t225, s_offset2, acc20, i41));
        (i41 = (i41 + 1));
      }
      (t236 = acc20);
    }
    return t236;
  }
}
__host__ __device__ int64_t t237(Rec3 batch2, Tensor outputs1, int64_t b_acc, int64_t b_idx) {
  int64_t seqAlloc[1];
  Seq t238;
  int64_t t239;
  (t239 = tensorLinearMaxIdx(b_idx, outputs1));
  Tensor1 t240;
  Tensor1 X71 = (batch2.correct_linear_outidxs);
  (t240 = X71);
  ((seqAlloc[0]) = b_idx);
  ((t238.seq) = seqAlloc);
  ((t238.len) = 1);
  int64_t t241;
  (t241 = ((t240.data)[(cartesian_to_linear_index((t240.dims), (t240.rank), t238) + (t240.offset))]));
  char t242;
  (t242 = (t239 == t241));
  if ((t242 == 1)) {
    {
      int64_t t243;
      (t243 = (b_acc + 1));
      return t243;
    }
  } else {
    {
      int64_t t244;
      (t244 = b_acc);
      return t244;
    }
  }
}
__host__ __device__ int64_t t245(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, char printStatus1, Rec2 network4, Seq2 batches, int64_t acc21, int64_t i42) {
  char (*t246) = "/";
  char (*t247) = "\r";
  if ((printStatus1 == 1)) {
    printf("%s", t247);
    int64_t t248;
    (t248 = (i42 + 1));
    double t249;
    (t249 = (( double ) t248));
    printf("%f", t249);
    printf("%s", t246);
    int64_t t250;
    (t250 = (batches.len));
    double t251;
    (t251 = (( double ) t250));
    printf("%f", t251);
  } else {
    ;
  }
  Rec3 batch2;
  (batch2 = ((batches.seq)[i42]));
  Tensor t252;
  Tensor X72 = (batch2.inputs);
  (t252 = X72);
  Seq t253;
  (t253 = tensor_shape((t252.dims), (t252.rank)));
  int64_t batchsize2;
  (batchsize2 = ((t253.seq)[0]));
  Tensor t254;
  Tensor X73 = (batch2.inputs);
  (t254 = X73);
  Tensor outputs1;
  (outputs1 = nnEvalExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, network4, t254));
  int64_t batch_accuracy;
  {
    int64_t i43 = 0;
    int64_t acc22 = 0;
    while ((i43 < batchsize2)) {
      (acc22 = t237(batch2, outputs1, acc22, i43));
      (i43 = (i43 + 1));
    }
    (batch_accuracy = acc22);
  }
  {
    int64_t t255;
    (t255 = (acc21 + batch_accuracy));
    return t255;
  }
}
__host__ __device__ int64_t nnAccuracyDiscrete(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, char printStatus1, Rec2 network4, Seq2 batches) {
  char (*t256) = "\n";
  int64_t t257;
  (t257 = (batches.len));
  int64_t correct_guesses;
  {
    int64_t i44 = 0;
    int64_t acc23 = 0;
    while ((i44 < t257)) {
      (acc23 = t245(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, printStatus1, network4, batches, acc23, i44));
      (i44 = (i44 + 1));
    }
    (correct_guesses = acc23);
  }
  if ((printStatus1 == 1)) {
    printf("%s", t256);
  } else {
    ;
  }
  {
    int64_t t258;
    (t258 = correct_guesses);
    return t258;
  }
}
__host__ __device__ int64_t t259(int64_t acc24, Rec3 batch3) {
  Tensor t260;
  Tensor X74 = (batch3.inputs);
  (t260 = X74);
  Seq t261;
  (t261 = tensor_shape((t260.dims), (t260.rank)));
  int64_t t262;
  (t262 = ((t261.seq)[0]));
  {
    int64_t t263;
    (t263 = (acc24 + t262));
    return t263;
  }
}
__host__ __device__ int64_t foldl7(int64_t acc_init8, Seq2 s17) {
  int64_t acc25 = acc_init8;
  int64_t i45 = 0;
  while ((i45 < (s17.len))) {
    (acc25 = t259(acc25, ((s17.seq)[i45])));
    (i45 = (i45 + 1));
  }
  return acc25;
}
__host__ __device__ double nnAccuracyProportion(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, char printStatus2, Rec2 network5, Seq2 batches1) {
  int64_t correct_guesses1;
  (correct_guesses1 = nnAccuracyDiscrete(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, printStatus2, network5, batches1));
  int64_t datalength;
  (datalength = foldl7(0, batches1));
  double t264;
  (t264 = (( double ) correct_guesses1));
  double t265;
  (t265 = (( double ) datalength));
  {
    double t266;
    (t266 = (t264 / t265));
    return t266;
  }
}
__host__ __device__ void wrappedPrint(char (*s18)) {
  printf("%s", s18);
}
__host__ __device__ void t267(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, double alpha1, double lambda1, int64_t batch_idx) {
  char (*t268) = "/";
  char (*t269) = "\rround ";
  char t270;
  char X75 = (params.printStatus);
  (t270 = X75);
  if ((t270 == 1)) {
    wrappedPrint(t269);
    int64_t t271;
    (t271 = (batch_idx + 1));
    double t272;
    (t272 = (( double ) t271));
    printf("%f", t272);
    wrappedPrint(t268);
    double t273;
    (t273 = (( double ) rounds));
    printf("%f", t273);
  } else {
    ;
  }
  Rec3 t274;
  (t274 = ((training_batches.seq)[batch_idx]));
  nnGradientDescentExn(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, network6, alpha1, lambda1, t274);
}
__host__ __device__ Rec5 t275(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, Seq2 validation_batches, Rec5 acc26, int64_t epoch_idx) {
  Rec5 alloc;
  char (*t276) = "%\n";
  char (*t277) = "Computed accuracy: ";
  char (*t278) = "evalating performance...\n";
  char (*t279) = "\n";
  char (*t280) = "]\n";
  char (*t281) = "[lambda = ";
  char (*t282) = "]\n";
  char (*t283) = "[alpha = ";
  char (*t284) = "]\n";
  char (*t285) = "/";
  char (*t286) = "[Iteration ";
  int64_t epoch;
  (epoch = (epoch_idx + 1));
  double alpha1;
  double X76 = (acc26._0);
  (alpha1 = X76);
  double lambda1;
  double X77 = (acc26._1);
  (lambda1 = X77);
  char t287;
  char X78 = (params.printStatus);
  (t287 = X78);
  if ((t287 == 1)) {
    wrappedPrint(t286);
    double t288;
    (t288 = (( double ) epoch));
    printf("%f", t288);
    wrappedPrint(t285);
    int64_t t289;
    int64_t X79 = (params.epochs);
    (t289 = X79);
    double t290;
    (t290 = (( double ) t289));
    printf("%f", t290);
    printf("%s", t284);
    wrappedPrint(t283);
    printf("%f", alpha1);
    wrappedPrint(t282);
    wrappedPrint(t281);
    printf("%f", lambda1);
    wrappedPrint(t280);
  } else {
    ;
  }
  int64_t t291;
  (t291 = (training_batches.len));
  {
    int64_t i46 = 0;
    while ((i46 < t291)) {
      t267(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, params, network6, rounds, training_batches, alpha1, lambda1, i46);
      (i46 = (i46 + 1));
    }
  }
  wrappedPrint(t279);
  char t292;
  char X80 = (params.evaluateBetweenEpochs);
  (t292 = X80);
  if ((t292 == 1)) {
    char t293;
    char X81 = (params.printStatus);
    (t293 = X81);
    if ((t293 == 1)) {
      wrappedPrint(t278);
    } else {
      ;
    }
    char t294;
    char X82 = (params.printStatus);
    (t294 = X82);
    double accuracy;
    (accuracy = nnAccuracyProportion(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, t294, network6, validation_batches));
    char t295;
    char X83 = (params.printStatus);
    (t295 = X83);
    if ((t295 == 1)) {
      wrappedPrint(t277);
      double t296;
      (t296 = (accuracy * 100.));
      printf("%f", t296);
      wrappedPrint(t276);
    } else {
      ;
    }
  } else {
    ;
  }
  double t297;
  double X84 = (params.decay_alpha);
  (t297 = X84);
  double t298;
  (t298 = (1. - t297));
  double decayed_alpha;
  (decayed_alpha = (alpha1 * t298));
  double t299;
  double X85 = (params.decay_lambda);
  (t299 = X85);
  double t300;
  (t300 = (1. - t299));
  double decayed_lambda;
  (decayed_lambda = (lambda1 * t300));
  ((alloc._0) = decayed_alpha);
  ((alloc._1) = decayed_lambda);
  {
    Rec5 t301;
    (t301 = alloc);
    return t301;
  }
}
void cuda_wrap(int64_t nnCompType_FullyConnected, int64_t nnCompType_ReLU, int64_t nnCompType_SoftMax, int64_t nnLossfnType_CrossEntropyLoss, int64_t nnLossfnType_SoftMaxCrossEntropyLoss, Rec4 params, Rec2 network6, int64_t rounds, Seq2 training_batches, Seq2 validation_batches) {
  Rec5 t302;
  char (*t303) = "%\n";
  char (*t304) = "Computed accuracy: ";
  char (*t305) = "evalating performance...\n";
  char t306;
  char X86 = (params.evaluateBeforeFirstEpoch);
  (t306 = X86);
  if ((t306 == 1)) {
    char t307;
    char X87 = (params.printStatus);
    (t307 = X87);
    if ((t307 == 1)) {
      wrappedPrint(t305);
    } else {
      ;
    }
    char t308;
    char X88 = (params.printStatus);
    (t308 = X88);
    double accuracy1;
    (accuracy1 = nnAccuracyProportion(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, t308, network6, validation_batches));
    char t309;
    char X89 = (params.printStatus);
    (t309 = X89);
    if ((t309 == 1)) {
      wrappedPrint(t304);
      double t310;
      (t310 = (accuracy1 * 100.));
      printf("%f", t310);
      wrappedPrint(t303);
    } else {
      ;
    }
  } else {
    ;
  }
  double t311;
  double X90 = (params.init_lambda);
  (t311 = X90);
  double t312;
  double X91 = (params.init_alpha);
  (t312 = X91);
  ((t302._0) = t312);
  ((t302._1) = t311);
  int64_t t313;
  int64_t X92 = (params.epochs);
  (t313 = X92);
  Rec5 _7;
  {
    int64_t i47 = 0;
    Rec5 acc27 = t302;
    while ((i47 < t313)) {
      (acc27 = t275(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss, params, network6, rounds, training_batches, validation_batches, acc27, i47));
      (i47 = (i47 + 1));
    }
    (_7 = acc27);
  }
  ;
}
extern "C" void vBkMGL47RfZ(value nnCompType_FullyConnected, value nnCompType_ReLU, value nnCompType_SoftMax, value nnLossfnType_CrossEntropyLoss, value nnLossfnType_SoftMaxCrossEntropyLoss, value params, value network6, value rounds, value training_batches, value validation_batches) {
  CAMLparam5(nnCompType_FullyConnected, nnCompType_ReLU, nnCompType_SoftMax, nnLossfnType_CrossEntropyLoss, nnLossfnType_SoftMaxCrossEntropyLoss);
  CAMLxparam5(params, network6, rounds, training_batches, validation_batches);
  int64_t tensor_count = (tensor_count = 0);
  int64_t gpu_tmp = Long_val(nnCompType_FullyConnected);
  int64_t gpu_tmp1 = Long_val(nnCompType_ReLU);
  int64_t gpu_tmp2 = Long_val(nnCompType_SoftMax);
  int64_t gpu_tmp3 = Long_val(nnLossfnType_CrossEntropyLoss);
  int64_t gpu_tmp4 = Long_val(nnLossfnType_SoftMaxCrossEntropyLoss);
  Rec4 gpu_tmp5;
  double cuda_rec_field = Double_val(Field(params, 0));
  ((gpu_tmp5.init_alpha) = cuda_rec_field);
  double cuda_rec_field1 = Double_val(Field(params, 1));
  ((gpu_tmp5.init_lambda) = cuda_rec_field1);
  double cuda_rec_field2 = Double_val(Field(params, 2));
  ((gpu_tmp5.decay_alpha) = cuda_rec_field2);
  double cuda_rec_field3 = Double_val(Field(params, 3));
  ((gpu_tmp5.decay_lambda) = cuda_rec_field3);
  int64_t cuda_rec_field4 = Long_val(Field(params, 4));
  ((gpu_tmp5.batchsize) = cuda_rec_field4);
  int64_t cuda_rec_field5 = Long_val(Field(params, 5));
  ((gpu_tmp5.epochs) = cuda_rec_field5);
  char cuda_rec_field6 = Int_val(Field(params, 6));
  ((gpu_tmp5.printStatus) = cuda_rec_field6);
  char cuda_rec_field7 = Int_val(Field(params, 7));
  ((gpu_tmp5.evaluateBetweenEpochs) = cuda_rec_field7);
  char cuda_rec_field8 = Int_val(Field(params, 8));
  ((gpu_tmp5.evaluateBeforeFirstEpoch) = cuda_rec_field8);
  Rec2 gpu_tmp6;
  Seq1 cuda_rec_field9;
  ((cuda_rec_field9.len) = Wosize_val(Field(network6, 0)));
  hipMallocManaged((&(cuda_rec_field9.seq)), (Wosize_val(Field(network6, 0)) * (sizeof(Rec))));
  int64_t i48 = 0;
  while ((i48 < (cuda_rec_field9.len))) {
    Rec cuda_seq_temp;
    int64_t cuda_rec_field10 = Long_val(Field(Field(Field(network6, 0), i48), 0));
    ((cuda_seq_temp.ty) = cuda_rec_field10);
    Tensor cuda_rec_field11;
    ((cuda_rec_field11.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 1))->num_dims));
    if (((cuda_rec_field11.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field11.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field11.offset) = 0);
    ((cuda_rec_field11.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i49 = 0;
    int64_t n20 = (sizeof(double));
    while ((i49 < (cuda_rec_field11.rank))) {
      (((cuda_rec_field11.dims)[i49]) = ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 1))->dim)[i49]));
      (n20 = (n20 * ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 1))->dim)[i49])));
      (i49 = (i49 + 1));
    }
    ((cuda_rec_field11.size) = n20);
    double (*t314);
    hipMallocManaged((&t314), n20);
    hipMemcpy(t314, Caml_ba_data_val(Field(Field(Field(network6, 0), i48), 1)), n20, hipMemcpyHostToDevice);
    ((cuda_rec_field11.data) = t314);
    ((cuda_seq_temp.w) = cuda_rec_field11);
    Tensor cuda_rec_field12;
    ((cuda_rec_field12.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 2))->num_dims));
    if (((cuda_rec_field12.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field12.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field12.offset) = 0);
    ((cuda_rec_field12.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i50 = 0;
    int64_t n21 = (sizeof(double));
    while ((i50 < (cuda_rec_field12.rank))) {
      (((cuda_rec_field12.dims)[i50]) = ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 2))->dim)[i50]));
      (n21 = (n21 * ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 2))->dim)[i50])));
      (i50 = (i50 + 1));
    }
    ((cuda_rec_field12.size) = n21);
    double (*t315);
    hipMallocManaged((&t315), n21);
    hipMemcpy(t315, Caml_ba_data_val(Field(Field(Field(network6, 0), i48), 2)), n21, hipMemcpyHostToDevice);
    ((cuda_rec_field12.data) = t315);
    ((cuda_seq_temp.b) = cuda_rec_field12);
    Tensor cuda_rec_field13;
    ((cuda_rec_field13.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 3))->num_dims));
    if (((cuda_rec_field13.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field13.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field13.offset) = 0);
    ((cuda_rec_field13.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i51 = 0;
    int64_t n22 = (sizeof(double));
    while ((i51 < (cuda_rec_field13.rank))) {
      (((cuda_rec_field13.dims)[i51]) = ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 3))->dim)[i51]));
      (n22 = (n22 * ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 3))->dim)[i51])));
      (i51 = (i51 + 1));
    }
    ((cuda_rec_field13.size) = n22);
    double (*t316);
    hipMallocManaged((&t316), n22);
    hipMemcpy(t316, Caml_ba_data_val(Field(Field(Field(network6, 0), i48), 3)), n22, hipMemcpyHostToDevice);
    ((cuda_rec_field13.data) = t316);
    ((cuda_seq_temp.w_grads) = cuda_rec_field13);
    Tensor cuda_rec_field14;
    ((cuda_rec_field14.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 4))->num_dims));
    if (((cuda_rec_field14.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field14.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field14.offset) = 0);
    ((cuda_rec_field14.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i52 = 0;
    int64_t n23 = (sizeof(double));
    while ((i52 < (cuda_rec_field14.rank))) {
      (((cuda_rec_field14.dims)[i52]) = ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 4))->dim)[i52]));
      (n23 = (n23 * ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 4))->dim)[i52])));
      (i52 = (i52 + 1));
    }
    ((cuda_rec_field14.size) = n23);
    double (*t317);
    hipMallocManaged((&t317), n23);
    hipMemcpy(t317, Caml_ba_data_val(Field(Field(Field(network6, 0), i48), 4)), n23, hipMemcpyHostToDevice);
    ((cuda_rec_field14.data) = t317);
    ((cuda_seq_temp.b_grads) = cuda_rec_field14);
    Tensor cuda_rec_field15;
    ((cuda_rec_field15.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 5))->num_dims));
    if (((cuda_rec_field15.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field15.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field15.offset) = 0);
    ((cuda_rec_field15.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i53 = 0;
    int64_t n24 = (sizeof(double));
    while ((i53 < (cuda_rec_field15.rank))) {
      (((cuda_rec_field15.dims)[i53]) = ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 5))->dim)[i53]));
      (n24 = (n24 * ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 5))->dim)[i53])));
      (i53 = (i53 + 1));
    }
    ((cuda_rec_field15.size) = n24);
    double (*t318);
    hipMallocManaged((&t318), n24);
    hipMemcpy(t318, Caml_ba_data_val(Field(Field(Field(network6, 0), i48), 5)), n24, hipMemcpyHostToDevice);
    ((cuda_rec_field15.data) = t318);
    ((cuda_seq_temp.out_bufs) = cuda_rec_field15);
    Tensor cuda_rec_field16;
    ((cuda_rec_field16.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 6))->num_dims));
    if (((cuda_rec_field16.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field16.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field16.offset) = 0);
    ((cuda_rec_field16.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i54 = 0;
    int64_t n25 = (sizeof(double));
    while ((i54 < (cuda_rec_field16.rank))) {
      (((cuda_rec_field16.dims)[i54]) = ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 6))->dim)[i54]));
      (n25 = (n25 * ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 6))->dim)[i54])));
      (i54 = (i54 + 1));
    }
    ((cuda_rec_field16.size) = n25);
    double (*t319);
    hipMallocManaged((&t319), n25);
    hipMemcpy(t319, Caml_ba_data_val(Field(Field(Field(network6, 0), i48), 6)), n25, hipMemcpyHostToDevice);
    ((cuda_rec_field16.data) = t319);
    ((cuda_seq_temp.in_grads) = cuda_rec_field16);
    Tensor cuda_rec_field17;
    ((cuda_rec_field17.rank) = (Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 7))->num_dims));
    if (((cuda_rec_field17.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field17.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field17.offset) = 0);
    ((cuda_rec_field17.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i55 = 0;
    int64_t n26 = (sizeof(double));
    while ((i55 < (cuda_rec_field17.rank))) {
      (((cuda_rec_field17.dims)[i55]) = ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 7))->dim)[i55]));
      (n26 = (n26 * ((Caml_ba_array_val(Field(Field(Field(network6, 0), i48), 7))->dim)[i55])));
      (i55 = (i55 + 1));
    }
    ((cuda_rec_field17.size) = n26);
    double (*t320);
    hipMallocManaged((&t320), n26);
    hipMemcpy(t320, Caml_ba_data_val(Field(Field(Field(network6, 0), i48), 7)), n26, hipMemcpyHostToDevice);
    ((cuda_rec_field17.data) = t320);
    ((cuda_seq_temp.softmax_bufs) = cuda_rec_field17);
    (((cuda_rec_field9.seq)[i48]) = cuda_seq_temp);
    (i48 = (i48 + 1));
  }
  ((gpu_tmp6.components) = cuda_rec_field9);
  Rec1 cuda_rec_field18;
  int64_t cuda_rec_field19 = Long_val(Field(Field(network6, 1), 0));
  ((cuda_rec_field18.ty) = cuda_rec_field19);
  Tensor cuda_rec_field20;
  ((cuda_rec_field20.rank) = (Caml_ba_array_val(Field(Field(network6, 1), 1))->num_dims));
  if (((cuda_rec_field20.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field20.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field20.offset) = 0);
  ((cuda_rec_field20.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i56 = 0;
  int64_t n27 = (sizeof(double));
  while ((i56 < (cuda_rec_field20.rank))) {
    (((cuda_rec_field20.dims)[i56]) = ((Caml_ba_array_val(Field(Field(network6, 1), 1))->dim)[i56]));
    (n27 = (n27 * ((Caml_ba_array_val(Field(Field(network6, 1), 1))->dim)[i56])));
    (i56 = (i56 + 1));
  }
  ((cuda_rec_field20.size) = n27);
  double (*t321);
  hipMallocManaged((&t321), n27);
  hipMemcpy(t321, Caml_ba_data_val(Field(Field(network6, 1), 1)), n27, hipMemcpyHostToDevice);
  ((cuda_rec_field20.data) = t321);
  ((cuda_rec_field18.in_grads) = cuda_rec_field20);
  Tensor cuda_rec_field21;
  ((cuda_rec_field21.rank) = (Caml_ba_array_val(Field(Field(network6, 1), 2))->num_dims));
  if (((cuda_rec_field21.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field21.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field21.offset) = 0);
  ((cuda_rec_field21.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i57 = 0;
  int64_t n28 = (sizeof(double));
  while ((i57 < (cuda_rec_field21.rank))) {
    (((cuda_rec_field21.dims)[i57]) = ((Caml_ba_array_val(Field(Field(network6, 1), 2))->dim)[i57]));
    (n28 = (n28 * ((Caml_ba_array_val(Field(Field(network6, 1), 2))->dim)[i57])));
    (i57 = (i57 + 1));
  }
  ((cuda_rec_field21.size) = n28);
  double (*t322);
  hipMallocManaged((&t322), n28);
  hipMemcpy(t322, Caml_ba_data_val(Field(Field(network6, 1), 2)), n28, hipMemcpyHostToDevice);
  ((cuda_rec_field21.data) = t322);
  ((cuda_rec_field18.out_bufs) = cuda_rec_field21);
  Tensor cuda_rec_field22;
  ((cuda_rec_field22.rank) = (Caml_ba_array_val(Field(Field(network6, 1), 3))->num_dims));
  if (((cuda_rec_field22.rank) > 3)) {
    printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field22.rank));
    exit(1);
  } else {
    
  }
  ((cuda_rec_field22.offset) = 0);
  ((cuda_rec_field22.id) = tensor_count);
  (tensor_count = (tensor_count + 1));
  int64_t i58 = 0;
  int64_t n29 = (sizeof(double));
  while ((i58 < (cuda_rec_field22.rank))) {
    (((cuda_rec_field22.dims)[i58]) = ((Caml_ba_array_val(Field(Field(network6, 1), 3))->dim)[i58]));
    (n29 = (n29 * ((Caml_ba_array_val(Field(Field(network6, 1), 3))->dim)[i58])));
    (i58 = (i58 + 1));
  }
  ((cuda_rec_field22.size) = n29);
  double (*t323);
  hipMallocManaged((&t323), n29);
  hipMemcpy(t323, Caml_ba_data_val(Field(Field(network6, 1), 3)), n29, hipMemcpyHostToDevice);
  ((cuda_rec_field22.data) = t323);
  ((cuda_rec_field18.softmax_bufs) = cuda_rec_field22);
  ((gpu_tmp6.lossfn) = cuda_rec_field18);
  int64_t gpu_tmp7 = Long_val(rounds);
  Seq2 gpu_tmp8;
  ((gpu_tmp8.len) = Wosize_val(training_batches));
  hipMallocManaged((&(gpu_tmp8.seq)), (Wosize_val(training_batches) * (sizeof(Rec3))));
  int64_t i59 = 0;
  while ((i59 < (gpu_tmp8.len))) {
    Rec3 cuda_seq_temp1;
    Tensor cuda_rec_field23;
    ((cuda_rec_field23.rank) = (Caml_ba_array_val(Field(Field(training_batches, i59), 0))->num_dims));
    if (((cuda_rec_field23.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field23.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field23.offset) = 0);
    ((cuda_rec_field23.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i60 = 0;
    int64_t n30 = (sizeof(double));
    while ((i60 < (cuda_rec_field23.rank))) {
      (((cuda_rec_field23.dims)[i60]) = ((Caml_ba_array_val(Field(Field(training_batches, i59), 0))->dim)[i60]));
      (n30 = (n30 * ((Caml_ba_array_val(Field(Field(training_batches, i59), 0))->dim)[i60])));
      (i60 = (i60 + 1));
    }
    ((cuda_rec_field23.size) = n30);
    double (*t324);
    hipMallocManaged((&t324), n30);
    hipMemcpy(t324, Caml_ba_data_val(Field(Field(training_batches, i59), 0)), n30, hipMemcpyHostToDevice);
    ((cuda_rec_field23.data) = t324);
    ((cuda_seq_temp1.inputs) = cuda_rec_field23);
    Tensor1 cuda_rec_field24;
    ((cuda_rec_field24.rank) = (Caml_ba_array_val(Field(Field(training_batches, i59), 1))->num_dims));
    if (((cuda_rec_field24.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field24.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field24.offset) = 0);
    ((cuda_rec_field24.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i61 = 0;
    int64_t n31 = (sizeof(int64_t));
    while ((i61 < (cuda_rec_field24.rank))) {
      (((cuda_rec_field24.dims)[i61]) = ((Caml_ba_array_val(Field(Field(training_batches, i59), 1))->dim)[i61]));
      (n31 = (n31 * ((Caml_ba_array_val(Field(Field(training_batches, i59), 1))->dim)[i61])));
      (i61 = (i61 + 1));
    }
    ((cuda_rec_field24.size) = n31);
    int64_t (*t325);
    hipMallocManaged((&t325), n31);
    hipMemcpy(t325, Caml_ba_data_val(Field(Field(training_batches, i59), 1)), n31, hipMemcpyHostToDevice);
    ((cuda_rec_field24.data) = t325);
    ((cuda_seq_temp1.correct_linear_outidxs) = cuda_rec_field24);
    (((gpu_tmp8.seq)[i59]) = cuda_seq_temp1);
    (i59 = (i59 + 1));
  }
  Seq2 gpu_tmp9;
  ((gpu_tmp9.len) = Wosize_val(validation_batches));
  hipMallocManaged((&(gpu_tmp9.seq)), (Wosize_val(validation_batches) * (sizeof(Rec3))));
  int64_t i62 = 0;
  while ((i62 < (gpu_tmp9.len))) {
    Rec3 cuda_seq_temp2;
    Tensor cuda_rec_field25;
    ((cuda_rec_field25.rank) = (Caml_ba_array_val(Field(Field(validation_batches, i62), 0))->num_dims));
    if (((cuda_rec_field25.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field25.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field25.offset) = 0);
    ((cuda_rec_field25.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i63 = 0;
    int64_t n32 = (sizeof(double));
    while ((i63 < (cuda_rec_field25.rank))) {
      (((cuda_rec_field25.dims)[i63]) = ((Caml_ba_array_val(Field(Field(validation_batches, i62), 0))->dim)[i63]));
      (n32 = (n32 * ((Caml_ba_array_val(Field(Field(validation_batches, i62), 0))->dim)[i63])));
      (i63 = (i63 + 1));
    }
    ((cuda_rec_field25.size) = n32);
    double (*t326);
    hipMallocManaged((&t326), n32);
    hipMemcpy(t326, Caml_ba_data_val(Field(Field(validation_batches, i62), 0)), n32, hipMemcpyHostToDevice);
    ((cuda_rec_field25.data) = t326);
    ((cuda_seq_temp2.inputs) = cuda_rec_field25);
    Tensor1 cuda_rec_field26;
    ((cuda_rec_field26.rank) = (Caml_ba_array_val(Field(Field(validation_batches, i62), 1))->num_dims));
    if (((cuda_rec_field26.rank) > 3)) {
      printf("Tensors with rank at most 3 are supported, found rank %ld\n", (cuda_rec_field26.rank));
      exit(1);
    } else {
      
    }
    ((cuda_rec_field26.offset) = 0);
    ((cuda_rec_field26.id) = tensor_count);
    (tensor_count = (tensor_count + 1));
    int64_t i64 = 0;
    int64_t n33 = (sizeof(int64_t));
    while ((i64 < (cuda_rec_field26.rank))) {
      (((cuda_rec_field26.dims)[i64]) = ((Caml_ba_array_val(Field(Field(validation_batches, i62), 1))->dim)[i64]));
      (n33 = (n33 * ((Caml_ba_array_val(Field(Field(validation_batches, i62), 1))->dim)[i64])));
      (i64 = (i64 + 1));
    }
    ((cuda_rec_field26.size) = n33);
    int64_t (*t327);
    hipMallocManaged((&t327), n33);
    hipMemcpy(t327, Caml_ba_data_val(Field(Field(validation_batches, i62), 1)), n33, hipMemcpyHostToDevice);
    ((cuda_rec_field26.data) = t327);
    ((cuda_seq_temp2.correct_linear_outidxs) = cuda_rec_field26);
    (((gpu_tmp9.seq)[i62]) = cuda_seq_temp2);
    (i62 = (i62 + 1));
  }
  hipMallocManaged((&t_state), (tensor_count * (sizeof(enum tensor_state))));
  int64_t i65 = 0;
  while ((i65 < ((gpu_tmp6.components).len))) {
    ((t_state[(((((gpu_tmp6.components).seq)[i65]).w).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i65]).b).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i65]).w_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i65]).b_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i65]).out_bufs).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i65]).in_grads).id)]) = STATE_OK);
    ((t_state[(((((gpu_tmp6.components).seq)[i65]).softmax_bufs).id)]) = STATE_OK);
    (i65 = (i65 + 1));
  }
  ((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) = STATE_OK);
  ((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) = STATE_OK);
  ((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) = STATE_OK);
  int64_t i66 = 0;
  while ((i66 < (gpu_tmp8.len))) {
    ((t_state[((((gpu_tmp8.seq)[i66]).inputs).id)]) = STATE_OK);
    ((t_state[((((gpu_tmp8.seq)[i66]).correct_linear_outidxs).id)]) = STATE_OK);
    (i66 = (i66 + 1));
  }
  int64_t i67 = 0;
  while ((i67 < (gpu_tmp9.len))) {
    ((t_state[((((gpu_tmp9.seq)[i67]).inputs).id)]) = STATE_OK);
    ((t_state[((((gpu_tmp9.seq)[i67]).correct_linear_outidxs).id)]) = STATE_OK);
    (i67 = (i67 + 1));
  }
  cuda_wrap(gpu_tmp, gpu_tmp1, gpu_tmp2, gpu_tmp3, gpu_tmp4, gpu_tmp5, gpu_tmp6, gpu_tmp7, gpu_tmp8, gpu_tmp9);
  int64_t i68 = 0;
  while ((i68 < ((gpu_tmp6.components).len))) {
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).w).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(Field(network6, 0), i68), 1)), (((((gpu_tmp6.components).seq)[i68]).w).data), (((((gpu_tmp6.components).seq)[i68]).w).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).w).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i68]).w).data));
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).b).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(Field(network6, 0), i68), 2)), (((((gpu_tmp6.components).seq)[i68]).b).data), (((((gpu_tmp6.components).seq)[i68]).b).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).b).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i68]).b).data));
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).w_grads).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(Field(network6, 0), i68), 3)), (((((gpu_tmp6.components).seq)[i68]).w_grads).data), (((((gpu_tmp6.components).seq)[i68]).w_grads).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).w_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i68]).w_grads).data));
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).b_grads).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(Field(network6, 0), i68), 4)), (((((gpu_tmp6.components).seq)[i68]).b_grads).data), (((((gpu_tmp6.components).seq)[i68]).b_grads).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).b_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i68]).b_grads).data));
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).out_bufs).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(Field(network6, 0), i68), 5)), (((((gpu_tmp6.components).seq)[i68]).out_bufs).data), (((((gpu_tmp6.components).seq)[i68]).out_bufs).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).out_bufs).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i68]).out_bufs).data));
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).in_grads).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(Field(network6, 0), i68), 6)), (((((gpu_tmp6.components).seq)[i68]).in_grads).data), (((((gpu_tmp6.components).seq)[i68]).in_grads).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).in_grads).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i68]).in_grads).data));
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).softmax_bufs).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(Field(network6, 0), i68), 7)), (((((gpu_tmp6.components).seq)[i68]).softmax_bufs).data), (((((gpu_tmp6.components).seq)[i68]).softmax_bufs).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[(((((gpu_tmp6.components).seq)[i68]).softmax_bufs).id)]) != STATE_RETURNED)) {
      hipFree((((((gpu_tmp6.components).seq)[i68]).softmax_bufs).data));
    } else {
      
    }
    (i68 = (i68 + 1));
  }
  hipFree(((gpu_tmp6.components).seq));
  if (((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) != STATE_OK)) {
    hipMemcpy(Caml_ba_data_val(Field(Field(network6, 1), 1)), (((gpu_tmp6.lossfn).in_grads).data), (((gpu_tmp6.lossfn).in_grads).size), hipMemcpyDeviceToHost);
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).in_grads).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).in_grads).data));
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) != STATE_OK)) {
    hipMemcpy(Caml_ba_data_val(Field(Field(network6, 1), 2)), (((gpu_tmp6.lossfn).out_bufs).data), (((gpu_tmp6.lossfn).out_bufs).size), hipMemcpyDeviceToHost);
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).out_bufs).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).out_bufs).data));
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) != STATE_OK)) {
    hipMemcpy(Caml_ba_data_val(Field(Field(network6, 1), 3)), (((gpu_tmp6.lossfn).softmax_bufs).data), (((gpu_tmp6.lossfn).softmax_bufs).size), hipMemcpyDeviceToHost);
  } else {
    
  }
  if (((t_state[(((gpu_tmp6.lossfn).softmax_bufs).id)]) != STATE_RETURNED)) {
    hipFree((((gpu_tmp6.lossfn).softmax_bufs).data));
  } else {
    
  }
  int64_t i69 = 0;
  while ((i69 < (gpu_tmp8.len))) {
    if (((t_state[((((gpu_tmp8.seq)[i69]).inputs).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(training_batches, i69), 0)), ((((gpu_tmp8.seq)[i69]).inputs).data), ((((gpu_tmp8.seq)[i69]).inputs).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i69]).inputs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp8.seq)[i69]).inputs).data));
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i69]).correct_linear_outidxs).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(training_batches, i69), 1)), ((((gpu_tmp8.seq)[i69]).correct_linear_outidxs).data), ((((gpu_tmp8.seq)[i69]).correct_linear_outidxs).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[((((gpu_tmp8.seq)[i69]).correct_linear_outidxs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp8.seq)[i69]).correct_linear_outidxs).data));
    } else {
      
    }
    (i69 = (i69 + 1));
  }
  hipFree((gpu_tmp8.seq));
  int64_t i70 = 0;
  while ((i70 < (gpu_tmp9.len))) {
    if (((t_state[((((gpu_tmp9.seq)[i70]).inputs).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(validation_batches, i70), 0)), ((((gpu_tmp9.seq)[i70]).inputs).data), ((((gpu_tmp9.seq)[i70]).inputs).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i70]).inputs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp9.seq)[i70]).inputs).data));
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i70]).correct_linear_outidxs).id)]) != STATE_OK)) {
      hipMemcpy(Caml_ba_data_val(Field(Field(validation_batches, i70), 1)), ((((gpu_tmp9.seq)[i70]).correct_linear_outidxs).data), ((((gpu_tmp9.seq)[i70]).correct_linear_outidxs).size), hipMemcpyDeviceToHost);
    } else {
      
    }
    if (((t_state[((((gpu_tmp9.seq)[i70]).correct_linear_outidxs).id)]) != STATE_RETURNED)) {
      hipFree(((((gpu_tmp9.seq)[i70]).correct_linear_outidxs).data));
    } else {
      
    }
    (i70 = (i70 + 1));
  }
  hipFree((gpu_tmp9.seq));
  hipFree(t_state);
  CAMLreturn0;
}
extern "C" void vvdRMkFMy5E(value (*args), int argc) {
  return vBkMGL47RfZ((args[0]), (args[1]), (args[2]), (args[3]), (args[4]), (args[5]), (args[6]), (args[7]), (args[8]), (args[9]));
}